#include "hip/hip_runtime.h"
#if defined(NAMD_CUDA)
#if __CUDACC_VER_MAJOR__ >= 11
#include <hipcub/hipcub.hpp>
#else
#include <namd_cub/cub.cuh>
#endif
#include <hip/hip_runtime.h>
#endif // NAMD_CUDA

#include "CudaComputeNonbondedKernel.h"
#include "CudaTileListKernel.h"
#include "DeviceCUDA.h"
#include "CudaComputeNonbondedInteractions.h"

#if defined(NAMD_CUDA)

#ifdef WIN32
#define __thread __declspec(thread)
#endif
extern __thread DeviceCUDA *deviceCUDA;

#define OVERALLOC 1.2f

void NAMD_die(const char *);
void NAMD_bug(const char *);

#define MAX_CONST_EXCLUSIONS 2048  // cache size is 8k
__constant__ unsigned int constExclusions[MAX_CONST_EXCLUSIONS];

//FEP parameters
__constant__ AlchData alchflags;
#define NONBONDKERNEL_NUM_WARP 4

template<typename T>
__device__ __forceinline__
T make_zero();

template<>
__device__ __forceinline__
float3 make_zero<float3>() {
  return make_float3(0.0f, 0.0f, 0.0f);
}

template<>
__device__ __forceinline__
float4 make_zero<float4>() {
  return make_float4(0.0f, 0.0f, 0.0f, 0.0f);
}

template<bool doEnergy, bool doSlow, typename jForceType>
__device__ __forceinline__
void calcForceEnergyMath(const float r2, const float qi, const float qj,
  const float dx, const float dy, const float dz,
  const int vdwtypei, const int vdwtypej, const float2* __restrict__ vdwCoefTable,
  hipTextureObject_t vdwCoefTableTex, 
  hipTextureObject_t forceTableTex, hipTextureObject_t energyTableTex,
  float3& iforce, float3& iforceSlow, jForceType& jforce, jForceType& jforceSlow,
  float& energyVdw, float& energyElec, float& energySlow,
  const CudaNBConstants nbConstants) {

  int vdwIndex = vdwtypej + vdwtypei;
#if __CUDA_ARCH__ >= 350
  float2 ljab = __ldg(&vdwCoefTable[vdwIndex]);
#else
  float2 ljab = tex1Dfetch<float2>(vdwCoefTableTex, vdwIndex);
#endif

  float rinv = rsqrtf(r2);
  float f, fSlow;
  float charge = qi * qj;

  cudaNBForceMagCalc_VdwEnergySwitch_PMEC1<doEnergy, doSlow>(
    r2, rinv, charge, ljab, nbConstants, 
    f, fSlow, energyVdw, energyElec, energySlow);  

  float fx = dx * f;
  float fy = dy * f;
  float fz = dz * f;
  iforce.x += fx;
  iforce.y += fy;
  iforce.z += fz;
  jforce.x -= fx;
  jforce.y -= fy;
  jforce.z -= fz;
  if (doSlow) {
    float fxSlow = dx * fSlow;
    float fySlow = dy * fSlow;
    float fzSlow = dz * fSlow;
    iforceSlow.x += fxSlow;
    iforceSlow.y += fySlow;
    iforceSlow.z += fzSlow;
    jforceSlow.x -= fxSlow;
    jforceSlow.y -= fySlow;
    jforceSlow.z -= fzSlow;
  }
}


template<bool doEnergy, bool doSlow, typename jForceType>
__device__ __forceinline__
void calcForceEnergy(const float r2, const float qi, const float qj,
  const float dx, const float dy, const float dz,
  const int vdwtypei, const int vdwtypej, const float2* __restrict__ vdwCoefTable,
  hipTextureObject_t vdwCoefTableTex, 
  hipTextureObject_t forceTableTex, hipTextureObject_t energyTableTex,
  float3& iforce, float3& iforceSlow, jForceType& jforce, jForceType& jforceSlow,
  float& energyVdw, float& energyElec, float& energySlow) {

  int vdwIndex = vdwtypej + vdwtypei;
#if __CUDA_ARCH__ >= 350
  float2 ljab = __ldg(&vdwCoefTable[vdwIndex]);
#else
  float2 ljab = tex1Dfetch<float2>(vdwCoefTableTex, vdwIndex);
#endif

  float rinv = rsqrtf(r2);
  float4 ei, fi;
  float f, fSlow;
 
  fi = tex1D<float4>(forceTableTex, rinv);
  if (doEnergy) ei = tex1D<float4>(energyTableTex, rinv);

  fSlow = qi * qj;
  f = ljab.x * fi.z + ljab.y * fi.y + fSlow * fi.x;

  if (doEnergy) {
    energyVdw    += ljab.x * ei.z + ljab.y * ei.y;
    energyElec   += fSlow * ei.x;

    if (doSlow) {
      energySlow   += fSlow * ei.w;
    }
  }
  if (doSlow) fSlow *= fi.w;
  
  float fx = dx * f;
  float fy = dy * f;
  float fz = dz * f;
  iforce.x += fx;
  iforce.y += fy;
  iforce.z += fz;
  jforce.x -= fx;
  jforce.y -= fy;
  jforce.z -= fz;

  if (doSlow) {
    float fxSlow = dx * fSlow;
    float fySlow = dy * fSlow;
    float fzSlow = dz * fSlow;
    iforceSlow.x += fxSlow;
    iforceSlow.y += fySlow;
    iforceSlow.z += fzSlow;
    jforceSlow.x -= fxSlow;
    jforceSlow.y -= fySlow;
    jforceSlow.z -= fzSlow;
  }
}


/* JM: Special __device__ function to compute VDW forces for alchemy. 
 * Partially swiped from ComputeNonbondedFEP.C
 */
template<bool doEnergy, bool doSlow, bool shift, bool vdwForceSwitch, typename jForceType>
__device__ __forceinline__
void calcForceEnergyFEP(const float r2, const float qi, const float qj,
  const float dx, const float dy, const float dz,
  const int vdwtypei, const int vdwtypej, 
  char p1, char p2,
  /*const AlchData &alchflags, */
  const float2* __restrict__ vdwCoefTable,
  hipTextureObject_t vdwCoefTableTex, 
  hipTextureObject_t forceTableTex, hipTextureObject_t energyTableTex,
  float3& iforce, float3& iforceSlow, jForceType& jforce, jForceType& jforceSlow,
  float& energyVdw, float &energyVdw_s, float& energyElec, float& energySlow, 
  float& energyElec_s, float& energySlow_s) {
  

 int vdwIndex = vdwtypej + vdwtypei;
#if __CUDA_ARCH__ >= 350
  float2 ljab = __ldg(&vdwCoefTable[vdwIndex]);
#else
  float2 ljab = tex1D<float2>(vdwCoefTableTex, vdwIndex); //ljab.x is A and ljab.y is B
#endif
  
  float myVdwLambda = 0.0f;
  float myVdwLambda2 = 0.0f;
  float myElecLambda = 0.0f;
  float myElecLambda2 = 0.0f;
  float rinv = rsqrtf(r2);
  float f;
  float alch_vdw_energy = 0.0f;
  float alch_vdw_energy_2 = 0.0f;
  float alch_vdw_force = 0.0f;
  float fSlow = qi * qj;
  float4 ei;
  float4 fi = tex1D<float4>(forceTableTex, rinv);
  if (doEnergy) ei = tex1D<float4>(energyTableTex, rinv);
  
  //John said that there is a better way to avoid divergences here
  //alch: true if => 1-0, 1-1, 2-0, 2-2
  //dec:  true if => 1-1, 2-2 && decouple
  //up: true if => 1-0 && 1,1
  //down: true if => 2-0, && 2,2
  int ref  = (p1 == 0 && p2 == 0);
  int alch = (!ref && !(p1 == 1  && p2 ==2) && !(p1 == 2 && p2 == 1));
  int dec  = (alch &&  (p1 == p2) && alchflags.alchDecouple);
  int up   = (alch &&  (p1 == 1 || p2 == 1) && !dec);
  int down = (alch &&  (p1 == 2 || p2 == 2) && !dec);

  float r2_1, r2_2;
  f = (fSlow * fi.x);

/*---------------   VDW SPECIAL ALCH FORCES (Swiped from ComputeNonbondedFEP.C)  ---------------*/

  myVdwLambda   = alchflags.vdwLambdaUp*(up)   + alchflags.vdwLambdaDown*(down)   + 1.f*(ref || dec);
  myVdwLambda2  = alchflags.vdwLambda2Up*(up)  + alchflags.vdwLambda2Down*(down)  + 1.f*(ref || dec);
  myElecLambda  = alchflags.elecLambdaUp*(up)  + alchflags.elecLambdaDown*(down)  + 1.f*(ref || dec);
  myElecLambda2 = alchflags.elecLambda2Up*(up) + alchflags.elecLambda2Down*(down) + 1.f*(ref || dec);

  if (alch) {
    if (vdwForceSwitch) {
      // force switching
      float switchdist6_1, switchdist6_2;
      const float cutoff6 = alchflags.cutoff2 * alchflags.cutoff2 * alchflags.cutoff2;
      // const float
      //Templated parameter. No control divergence here
      if (shift) {
        const float myVdwShift = alchflags.vdwShiftUp*up + alchflags.vdwShiftDown*(!up);
        const float myVdwShift2 = alchflags.vdwShift2Up*up + alchflags.vdwShift2Down*(!up);
        r2_1 = __fdividef(1.f,(r2 + myVdwShift));
        r2_2 = __fdividef(1.f,(r2 + myVdwShift2));
        switchdist6_1 = alchflags.switchdist2 + myVdwShift;
        switchdist6_1 = switchdist6_1 * switchdist6_1 * switchdist6_1;
        switchdist6_2 = alchflags.switchdist2 + myVdwShift2;
        switchdist6_2 = switchdist6_2 * switchdist6_2 * switchdist6_2;
      } else {
        r2_1 = rinv*rinv;
        r2_2 = rinv*rinv;
        switchdist6_1 = alchflags.switchdist2 * alchflags.switchdist2 * alchflags.switchdist2;
        switchdist6_2 = switchdist6_1;
      }
      const float r6_1 = r2_1*r2_1*r2_1;
      const float r6_2 = r2_2*r2_2*r2_2;
      if (r2 <= alchflags.switchdist2) {
        const float U1 = ljab.x*r6_1*r6_1 - ljab.y*r6_1; // NB: unscaled, shorthand only!
        const float U2 = ljab.x*r6_2*r6_2 - ljab.y*r6_2;
        // A == ljab.x, B == ljab.y
        const float dU_1 = -ljab.x / (cutoff6 * switchdist6_1) - (-ljab.y * rsqrtf(cutoff6 * switchdist6_1));
        const float dU_2 = -ljab.x / (cutoff6 * switchdist6_2) - (-ljab.y * rsqrtf(cutoff6 * switchdist6_2));
        alch_vdw_energy   = myVdwLambda  * (U1 + dU_1);
        alch_vdw_energy_2 = myVdwLambda2 * (U2 + dU_2);

        //Multiplied by -1.0 to match CPU values
        alch_vdw_force    =-1.f*myVdwLambda*((12.f*U1 + 6.f*ljab.y*r6_1)*r2_1);
      } else {
        const float r3_1 = sqrtf(r6_1);
        const float r3_2 = sqrtf(r6_2);
        const float inv_cutoff6 = 1.0f / cutoff6;
        const float inv_cutoff3 = rsqrtf(cutoff6);
        // A == ljab.x, B == ljab.y
        const float k_vdwa_1 = ljab.x / (1.0f - switchdist6_1 * inv_cutoff6);
        const float k_vdwb_1 = ljab.y / (1.0f - sqrtf(switchdist6_1 * inv_cutoff6));
        const float k_vdwa_2 = ljab.x / (1.0f - switchdist6_2 * inv_cutoff6);
        const float k_vdwb_2 = ljab.y / (1.0f - sqrtf(switchdist6_2 * inv_cutoff6));
        const float tmpa_1 = r6_1 - inv_cutoff6;
        const float tmpb_1 = r3_1 - inv_cutoff3;
        const float tmpa_2 = r6_2 - inv_cutoff6;
        const float tmpb_2 = r3_2 - inv_cutoff3;
        alch_vdw_energy   = myVdwLambda  * (k_vdwa_1 * tmpa_1 * tmpa_1 - k_vdwb_1 * tmpb_1 * tmpb_1);
        alch_vdw_energy_2 = myVdwLambda2 * (k_vdwa_2 * tmpa_2 * tmpa_2 - k_vdwb_2 * tmpb_2 * tmpb_2);
        //Multiplied by -1.0 to match CPU values
        alch_vdw_force = -1.0f * myVdwLambda * (6.0f * r2_1 *  (2.0f * k_vdwa_1 * tmpa_1 * r6_1 - k_vdwb_1 * tmpb_1 * r3_1));
      } // r2 <= alchflags.switchdist2
    } else {
      // potential switching
      const float diff = alchflags.cutoff2 - r2;

      const float switchmul  = (alchflags.switchfactor*(diff)*(diff)*(alchflags.cutoff2 - 3.f*alchflags.switchdist2 + 2.f*r2))*(r2 > alchflags.switchdist2)  + (1.f)*(r2 <= alchflags.switchdist2);
      const float switchmul2 = (12.f*alchflags.switchfactor*(diff)*(r2 - alchflags.switchdist2))*(r2 > alchflags.switchdist2) +  (0.f) * (r2 <= alchflags.switchdist2);

      //Templated parameter. No control divergence here
      if(shift){
        //This templated parameter lets me get away with not making 2 divisions. But for myVdwShift != 0, how do I do this?
        const float myVdwShift = alchflags.vdwShiftUp*up + alchflags.vdwShiftDown*(!up);
        const float myVdwShift2 = alchflags.vdwShift2Up*up + alchflags.vdwShift2Down*(!up);
        //r2_1 = 1.0/(r2 + myVdwShift);
        //r2_2 = 1.0/(r2 + myVdwShift2);
        r2_1 = __fdividef(1.f,(r2 + myVdwShift));
        r2_2 = __fdividef(1.f,(r2 + myVdwShift2));
      }else{
        r2_1 = rinv*rinv;
        r2_2 = rinv*rinv;
      }

      const float r6_1 = r2_1*r2_1*r2_1;
      const float r6_2 = r2_2*r2_2*r2_2;
      const float U1 = ljab.x*r6_1*r6_1 - ljab.y*r6_1; // NB: unscaled, shorthand only!
      const float U2 = ljab.x*r6_2*r6_2 - ljab.y*r6_2;
      alch_vdw_energy   = myVdwLambda*switchmul*U1;
      alch_vdw_energy_2 = myVdwLambda2*switchmul*U2;

      //Multiplied by -1.0 to match CPU values
      alch_vdw_force    =-1.f*myVdwLambda*((switchmul*(12.f*U1 + 6.f*ljab.y*r6_1)*r2_1+ switchmul2*U1));
    } // vdwForceSwitch
  }

/*-----------------------------------------------------------*/

  if (doEnergy){
    //All energies should be scaled by the corresponding lambda
    energyVdw    += (ljab.x * ei.z + ljab.y * ei.y)*(ref || dec) + alch_vdw_energy*(alch && !dec);
    energyElec   += (fSlow * ei.x)*myElecLambda;
    energyVdw_s  += (ljab.x * ei.z + ljab.y * ei.y)*(ref || dec) + alch_vdw_energy_2*(alch && !dec);
    energyElec_s += (fSlow * ei.x)*myElecLambda2;
    if (doSlow){
      energySlow   += (fSlow * ei.w)*myElecLambda;
      energySlow_s += (fSlow * ei.w)*myElecLambda2;
    }
  }

  if (doSlow) fSlow *= fi.w;

  //We should include the regular VDW forces if not dealing with alch pairs
  f = (f + ((ljab.x * fi.z + ljab.y * fi.y)*(!alch || dec)))*myElecLambda 
  + alch_vdw_force*(alch && !dec); 

  float fx = dx * f;
  float fy = dy * f;
  float fz = dz * f;

  iforce.x += fx;
  iforce.y += fy;
  iforce.z += fz;
  jforce.x -= fx;
  jforce.y -= fy;
  jforce.z -= fz;
   
  if (doSlow) {
    /*There's stuff that needs to be added here, when FAST AND NOSHORT macros are on*/
    fSlow = myElecLambda*fSlow; 
    float fxSlow = dx * fSlow;
    float fySlow = dy * fSlow;
    float fzSlow = dz * fSlow;
    iforceSlow.x += fxSlow;
    iforceSlow.y += fySlow;
    iforceSlow.z += fzSlow;
    jforceSlow.x -= fxSlow;
    jforceSlow.y -= fySlow;
    jforceSlow.z -= fzSlow;
  }
}

/* JM: Special __device__ function to compute VDW forces for TI. 
 */

template<bool doEnergy, bool doSlow, bool shift, bool vdwForceSwitch, typename jForceType>
__device__ __forceinline__
void calcForceEnergyTI(const float r2, const float qi, const float qj,
  const float dx, const float dy, const float dz,
  const int vdwtypei, const int vdwtypej, 
  char p1, char p2,
  const float2* __restrict__ vdwCoefTable,
  hipTextureObject_t vdwCoefTableTex, 
  hipTextureObject_t forceTableTex, hipTextureObject_t energyTableTex,
  float3& iforce, float3& iforceSlow, jForceType& jforce, jForceType& jforceSlow,
  float& energyVdw,  float& energyVdw_ti_1,  float& energyVdw_ti_2,  
  float& energyElec, float& energyElec_ti_1, float& energyElec_ti_2,
  float& energySlow, float& energySlow_ti_1, float& energySlow_ti_2) {
  
 int vdwIndex = vdwtypej + vdwtypei;
#if __CUDA_ARCH__ >= 350
  float2 ljab = __ldg(&vdwCoefTable[vdwIndex]);
#else
  float2 ljab = tex1D<float2>(vdwCoefTableTex, vdwIndex); //ljab.x is A and ljab.y is B
#endif

  /*  JM: For TI, we have to deal ALCH1 OR ALCH2 during ComputeNonbondedBase2
   *  ALCH1 for appearing terms;
   *  ALCH2 for dissapearing terms;
   *  Instead of the _s energy terms, we need the to calculate:
   * 
   *  vdwEnergy_ti_1 and _2 for VDW energies. For those we need to add the special terms calculated on
   *  ComputeNonbondedTI.C
   * 
   * elecEnergy_ti_1 and _2 for electrostatic energy. No correction needed here though.
   * 
   */
  
  float myVdwLambda = 0.0f;
  float myElecLambda = 0.0f;
  float rinv = rsqrtf(r2);
  float f;
  float alch_vdw_energy = 0.0f;
  float alch_vdw_force  = 0.0f;
  float alch_vdw_dUdl   = 0.0f;
  float fSlow = qi * qj;
  float4 ei;
  float4 fi = tex1D<float4>(forceTableTex, rinv);
  if (doEnergy) ei = tex1D<float4>(energyTableTex, rinv);
  
  //John said that there is a better way to avoid divergences here
  //alch: true if => 1-0, 1-1, 2-0, 2-2
  //dec:  true if => 1-1, 2-2 && decouple
  //up: true if => 1-0 && 1,1
  //down: true if => 2-0, && 2,2
  int ref  = (p1 == 0 && p2 == 0);
  int alch = (!ref && !(p1 == 1  && p2 ==2) && !(p1 == 2 && p2 == 1));
  int dec  = (alch &&  (p1 == p2) && alchflags.alchDecouple);
  int up   = (alch &&  (p1 == 1 || p2 == 1) && !dec);
  int down = (alch &&  (p1 == 2 || p2 == 2) && !dec);
  
  float r2_1;
  f = (fSlow * fi.x);

/*---------------   VDW SPECIAL ALCH STUFF (Swiped from ComputeNonbondedTI.C)  ---------------*/
  myVdwLambda   = alchflags.vdwLambdaUp*(up)   + alchflags.vdwLambdaDown*(down)   + 1.f*(ref || dec);
  myElecLambda  = alchflags.elecLambdaUp*(up)  + alchflags.elecLambdaDown*(down)  + 1.f*(ref || dec);
  if(alch){
    if (vdwForceSwitch) {
      const float cutoff6 = alchflags.cutoff2 * alchflags.cutoff2 * alchflags.cutoff2;
      float switchdist6;
      if (shift) {
        const float myVdwShift = alchflags.vdwShiftUp*up + alchflags.vdwShiftDown*(!up);
        r2_1 = __fdividef(1.f,(r2 + myVdwShift));
        switchdist6 = alchflags.switchdist2 + myVdwShift;
        switchdist6 = switchdist6 * switchdist6 * switchdist6;
      } else {
        r2_1 = rinv*rinv;
        switchdist6 = alchflags.switchdist2 * alchflags.switchdist2 * alchflags.switchdist2;
      }
      const float r6_1 = r2_1*r2_1*r2_1;
      if (r2 <= alchflags.switchdist2) {
        const float U    = ljab.x*r6_1*r6_1 - ljab.y*r6_1;
        const float dU =  -ljab.x / (cutoff6 * switchdist6) - (-ljab.y * rsqrtf(cutoff6 * switchdist6));
        alch_vdw_force = -1.f*(myVdwLambda*((12.f*U + 6.f*ljab.y*r6_1)*r2_1));
        alch_vdw_energy = myVdwLambda * (U + dU);
        alch_vdw_dUdl = U + myVdwLambda * alchflags.alchVdwShiftCoeff * (6.f*U + 3.f*ljab.y*r6_1)*r2_1 + dU;
      } else {
        const float r3_1 = sqrtf(r6_1);
        const float inv_cutoff6 = 1.0f / cutoff6;
        const float inv_cutoff3 = sqrtf(inv_cutoff6);
        const float k_vdwa_1 = ljab.x / (1.0f - switchdist6 * inv_cutoff6);
        const float k_vdwb_1 = ljab.y / (1.0f - sqrtf(switchdist6 * inv_cutoff6));
        const float tmpa_1 = r6_1 - inv_cutoff6;
        const float tmpb_1 = r3_1 - inv_cutoff3;
        const float U = k_vdwa_1 * tmpa_1 * tmpa_1 - k_vdwb_1 * tmpb_1 * tmpb_1;
        alch_vdw_force = -1.0f * myVdwLambda * (6.0f * r2_1 * (2.0f * k_vdwa_1 * tmpa_1 * r6_1 - k_vdwb_1 * tmpb_1 * r3_1));
        alch_vdw_energy = myVdwLambda * U;
        alch_vdw_dUdl = U + myVdwLambda * alchflags.alchVdwShiftCoeff * (3.0f * r2_1 * (2.0f * k_vdwa_1 * tmpa_1 * r6_1 - k_vdwb_1 * tmpb_1 * r3_1));
      } // r2 <= alchflags.switchdist2
    } else {
      const float diff = alchflags.cutoff2 - r2;
      const float switchmul  = (r2 > alchflags.switchdist2 ? alchflags.switchfactor*(diff)*(diff) \
            *(alchflags.cutoff2 - 3.f*alchflags.switchdist2 + 2.f*r2) : 1.f);

      const float switchmul2 = (r2 > alchflags.switchdist2 ?          \
                                12.f*alchflags.switchfactor*(diff)       \
                                *(r2 - alchflags.switchdist2) : 0.f);
      //Templated parameter. No control divergence here
      if(shift){
        const float myVdwShift = alchflags.vdwShiftUp*up + alchflags.vdwShiftDown*(!up);
        r2_1 = __fdividef(1.f,(r2 + myVdwShift));
      }else r2_1 = rinv*rinv;

      const float r6_1 = r2_1*r2_1*r2_1;
      const float U    = ljab.x*r6_1*r6_1 - ljab.y*r6_1; // NB: unscaled! for shorthand only!
      alch_vdw_energy = myVdwLambda*switchmul*U;
      //Multiplied by -1.0 to match CPU values
      alch_vdw_force = -1.f*(myVdwLambda*(switchmul*(12.f*U + 6.f*ljab.y*r6_1)*r2_1 \
                                    + switchmul2*U));
      alch_vdw_dUdl = (switchmul*(U + myVdwLambda*alchflags.alchVdwShiftCoeff \
                                    *(6.f*U + 3.f*ljab.y*r6_1)*r2_1));
    } // vdwForceSwitch
  }
  /*-------------------------------------------------------------------------*/

  if (doEnergy){
    //All energies should be scaled by the corresponding lambda
    energyVdw       += (ljab.x * ei.z + ljab.y * ei.y)*(ref || dec) + alch_vdw_energy*(alch && !dec);
    energyElec      += (fSlow * ei.x)*myElecLambda;
    if(alch){
      energyVdw_ti_1  += alch_vdw_dUdl*up;
      energyVdw_ti_2  += alch_vdw_dUdl*down;
      energyElec_ti_1 += (fSlow * ei.x)*up;
      energyElec_ti_2 += (fSlow * ei.x)*down;
    }
    if (doSlow){
      energySlow      += (fSlow * ei.w)*myElecLambda;
      if(alch){
        energySlow_ti_1 += (fSlow * ei.w)*up;
        energySlow_ti_2 += (fSlow * ei.w)*down;
      }
    }
  }

  if (doSlow) fSlow *= fi.w;
  //We should include the regular VDW forces if not dealing with alch pairs
  f = (f + ((ljab.x * fi.z + ljab.y * fi.y)*(ref || dec)))*myElecLambda 
  + alch_vdw_force*(alch && !dec);

  float fx = dx * f;
  float fy = dy * f;
  float fz = dz * f;

  iforce.x += fx;
  iforce.y += fy;
  iforce.z += fz;
  jforce.x -= fx;
  jforce.y -= fy;
  jforce.z -= fz;
   
  if (doSlow) {
    /*There's stuff that needs to be added here, when FAST AND NOSHORT macros are on*/
    fSlow = myElecLambda*fSlow;  /* FAST(NOSHORT(+alch_vdw_force))*/ //Those should also be zeroed 
    float fxSlow = dx * fSlow;
    float fySlow = dy * fSlow;
    float fzSlow = dz * fSlow;
    iforceSlow.x += fxSlow;
    iforceSlow.y += fySlow;
    iforceSlow.z += fzSlow;
    jforceSlow.x -= fxSlow;
    jforceSlow.y -= fySlow;
    jforceSlow.z -= fzSlow;
  }
}
  

template<bool doSlow, typename jForceType>
__device__ __forceinline__
void storeForces(const int pos, const jForceType force, const jForceType forceSlow,
  float4* __restrict__ devForces, float4* __restrict__ devForcesSlow) {
  atomicAdd(&devForces[pos].x, force.x);
  atomicAdd(&devForces[pos].y, force.y);
  atomicAdd(&devForces[pos].z, force.z);
  if (doSlow) {
    atomicAdd(&devForcesSlow[pos].x, forceSlow.x);
    atomicAdd(&devForcesSlow[pos].y, forceSlow.y);
    atomicAdd(&devForcesSlow[pos].z, forceSlow.z);
  }
}

template<bool doSlow, typename jForceType>
__device__ __forceinline__
void storeForces(const int pos, const jForceType force, const jForceType forceSlow,
                 float* __restrict__ devForces_x, 
                 float* __restrict__ devForces_y, 
                 float* __restrict__ devForces_z,
                 float* __restrict__ devForcesSlow_x, 
                 float* __restrict__ devForcesSlow_y, 
                 float* __restrict__ devForcesSlow_z)
{
  atomicAdd(&devForces_x[pos], force.x);
  atomicAdd(&devForces_y[pos], force.y);
  atomicAdd(&devForces_z[pos], force.z);
  if (doSlow) {
    atomicAdd(&devForcesSlow_x[pos], forceSlow.x);
    atomicAdd(&devForcesSlow_y[pos], forceSlow.y);
    atomicAdd(&devForcesSlow_z[pos], forceSlow.z);
  }
}

template<bool doSlow, typename jForceType>
__device__ __forceinline__
void storeForces(const int pos, const jForceType force, const jForceType forceSlow,
  float3* __restrict__ forces, float3* __restrict__ forcesSlow) {
  atomicAdd(&forces[pos].x, force.x);
  atomicAdd(&forces[pos].y, force.y);
  atomicAdd(&forces[pos].z, force.z);
  if (doSlow) {
    atomicAdd(&forcesSlow[pos].x, forceSlow.x);
    atomicAdd(&forcesSlow[pos].y, forceSlow.y);
    atomicAdd(&forcesSlow[pos].z, forceSlow.z);
  }
}

template<bool doPairlist>
__device__ __forceinline__
void shuffleNext(float& xyzq_j_w, int& vdwtypej, int& jatomIndex, int& jexclMaxdiff, int& jexclIndex) {
  xyzq_j_w = WARP_SHUFFLE(WARP_FULL_MASK, xyzq_j_w, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);
  vdwtypej = WARP_SHUFFLE(WARP_FULL_MASK, vdwtypej, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);
  if (doPairlist) {
    jatomIndex   = WARP_SHUFFLE(WARP_FULL_MASK, jatomIndex, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);    
    jexclIndex   = WARP_SHUFFLE(WARP_FULL_MASK, jexclIndex, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);
    jexclMaxdiff = WARP_SHUFFLE(WARP_FULL_MASK, jexclMaxdiff, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);
  }
}

template<bool doPairlist>
__device__ __forceinline__
void shuffleNext(float& xyzq_j_w, int& vdwtypej, int& jatomIndex) {
  xyzq_j_w = WARP_SHUFFLE(WARP_FULL_MASK, xyzq_j_w, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);
  vdwtypej = WARP_SHUFFLE(WARP_FULL_MASK, vdwtypej, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);
  if (doPairlist) {
    jatomIndex   = WARP_SHUFFLE(WARP_FULL_MASK, jatomIndex, (threadIdx.x+1) & (WARPSIZE-1), WARPSIZE);    
  }
}

template<bool doSlow, typename jForceType>
__device__ __forceinline__
void shuffleNext(jForceType& jforce, jForceType& jforceSlow) {
  jforce.x = WARP_SHUFFLE(WARP_FULL_MASK, jforce.x, (threadIdx.x+1)&(WARPSIZE-1), WARPSIZE);
  jforce.y = WARP_SHUFFLE(WARP_FULL_MASK, jforce.y, (threadIdx.x+1)&(WARPSIZE-1), WARPSIZE);
  jforce.z = WARP_SHUFFLE(WARP_FULL_MASK, jforce.z, (threadIdx.x+1)&(WARPSIZE-1), WARPSIZE);
  if (doSlow) {
    jforceSlow.x = WARP_SHUFFLE(WARP_FULL_MASK, jforceSlow.x, (threadIdx.x+1)&(WARPSIZE-1), WARPSIZE);
    jforceSlow.y = WARP_SHUFFLE(WARP_FULL_MASK, jforceSlow.y, (threadIdx.x+1)&(WARPSIZE-1), WARPSIZE);
    jforceSlow.z = WARP_SHUFFLE(WARP_FULL_MASK, jforceSlow.z, (threadIdx.x+1)&(WARPSIZE-1), WARPSIZE);
  }
}

//#define USE_NEW_EXCL_METHOD

//
// Returns the lower estimate for the distance between a bounding box and a set of atoms
//
__device__ __forceinline__ float distsq(const BoundingBox a, const float4 b) {
  float dx = max(0.0f, fabsf(a.x - b.x) - a.wx);
  float dy = max(0.0f, fabsf(a.y - b.y) - a.wy);
  float dz = max(0.0f, fabsf(a.z - b.z) - a.wz);
  float r2 = dx*dx + dy*dy + dz*dz;
  return r2;
}

#define LARGE_FLOAT (float)(1.0e10)

//
// Nonbonded force kernel
//
template <bool doEnergy, bool doVirial, bool doSlow, bool doPairlist, bool doAlch, bool doFEP, bool doTI, bool doStreaming, bool doTable, bool doAlchVdwForceSwitching>
__global__ void
__launch_bounds__(WARPSIZE*NONBONDKERNEL_NUM_WARP,
  doPairlist ? (10) : (doEnergy ? (10) : (12) )
  )
nonbondedForceKernel(
  const int start, const int numTileLists,
  const TileList* __restrict__ tileLists, TileExcl* __restrict__ tileExcls,
  const int* __restrict__ tileJatomStart,
  const int vdwCoefTableWidth, const float2* __restrict__ vdwCoefTable, const int* __restrict__ vdwTypes,
  const float3 lata, const float3 latb, const float3 latc,
  const float4* __restrict__ xyzq,
  const float cutoff2, const CudaNBConstants nbConstants,
  hipTextureObject_t vdwCoefTableTex,
  hipTextureObject_t forceTableTex, hipTextureObject_t energyTableTex,
  // ----------
  // doPairlist
  const int atomStorageSize, const float plcutoff2, const PatchPairRecord* __restrict__ patchPairs,
  const int* __restrict__ atomIndex,
  const int2* __restrict__ exclIndexMaxDiff, const unsigned int* __restrict__ overflowExclusions,
  unsigned int* __restrict__ tileListDepth, int* __restrict__ tileListOrder,
  int* __restrict__ jtiles, TileListStat* __restrict__ tileListStat,
  const BoundingBox* __restrict__ boundingBoxes,
#ifdef USE_NEW_EXCL_METHOD
  const int* __restrict__ minmaxExclAtom,
#endif
  // ----------
  float4* __restrict__ devForces, float4* __restrict__ devForcesSlow,
  float * __restrict__ devForce_x,
  float * __restrict__ devForce_y,
  float * __restrict__ devForce_z,
  float * __restrict__ devForce_w,
  float * __restrict__ devForceSlow_x,
  float * __restrict__ devForceSlow_y,
  float * __restrict__ devForceSlow_z,
  float * __restrict__ devForceSlow_w,                     
  // ---- USE_STREAMING_FORCES ----
  const int numPatches,
  unsigned int* __restrict__ patchNumCount,
  const CudaPatchRecord* __restrict__ cudaPatches,
  float4* __restrict__ mapForces, float4* __restrict__ mapForcesSlow,
  int* __restrict__ mapPatchReadyQueue,
  int* __restrict__ outputOrder,
  // ------------------------------
  TileListVirialEnergy* __restrict__ virialEnergy,
  // ---- doAlch ----
  char* __restrict__ p
  ) {
  using jForceType = typename std::conditional<doSlow, float3, float4>::type;
  // Single warp takes care of one list of tiles
  // for (int itileList = (threadIdx.x + blockDim.x*blockIdx.x)/WARPSIZE;itileList < numTileLists;itileList += blockDim.x*gridDim.x/WARPSIZE)
  const int itileListTemp = start + (threadIdx.x + blockDim.x*blockIdx.x) / WARPSIZE;
  const int itileList = WARP_SHUFFLE(WARP_FULL_MASK, itileListTemp, 0, WARPSIZE);
  if (itileList < numTileLists)
  {

    float3 iforce;
    float3 iforceSlow;
    float energyVdw, energyElec, energySlow;
    //FEP energies
    float energyVdw_s, energyElec_s, energySlow_s;
    //TI energies
    float energyVdw_ti_1, energyVdw_ti_2, energyElec_ti_1, energyElec_ti_2, energySlow_ti_1, energySlow_ti_2;
    int nexcluded;
    unsigned int itileListLen;
    int2 patchInd;
    int2 patchNumList;
    char part1, part2, p2;
    bool doShift = (alchflags.alchVdwShiftCoeff != 0.0f);
    __shared__ float4 s_xyzq[NONBONDKERNEL_NUM_WARP][WARPSIZE];
    __shared__ jForceType s_jforce[NONBONDKERNEL_NUM_WARP][WARPSIZE];
    __shared__ jForceType s_jforceSlow[NONBONDKERNEL_NUM_WARP][WARPSIZE];
    __shared__ int    s_vdwtypej[NONBONDKERNEL_NUM_WARP][WARPSIZE];
    __shared__ int    s_jatomIndex[NONBONDKERNEL_NUM_WARP][WARPSIZE];

    __shared__ int s_iatomStart[NONBONDKERNEL_NUM_WARP];
    __shared__ int s_jatomStart[NONBONDKERNEL_NUM_WARP];

    // Start computation
    {
      // Warp index (0...warpsize-1)
      const int wid = threadIdx.x & (WARPSIZE-1);
      const int iwarp = WARP_SHUFFLE(WARP_FULL_MASK, threadIdx.x / WARPSIZE, 0, WARPSIZE);

      TileList tmp = tileLists[itileList];
      int iatomStart = tmp.iatomStart;
      int jtileStart = tmp.jtileStart;
      int jtileEnd   = tmp.jtileEnd;
      patchInd     = tmp.patchInd;
      patchNumList = tmp.patchNumList;

      float shx = tmp.offsetXYZ.x*lata.x + tmp.offsetXYZ.y*latb.x + tmp.offsetXYZ.z*latc.x;
      float shy = tmp.offsetXYZ.x*lata.y + tmp.offsetXYZ.y*latb.y + tmp.offsetXYZ.z*latc.y;
      float shz = tmp.offsetXYZ.x*lata.z + tmp.offsetXYZ.y*latb.z + tmp.offsetXYZ.z*latc.z;

      // DH - set zeroShift flag if magnitude of shift vector is zero
      bool zeroShift = ! (shx*shx + shy*shy + shz*shz > 0);

      int iatomSize, iatomFreeSize, jatomSize, jatomFreeSize;
      if (doPairlist) {
        PatchPairRecord PPStmp = patchPairs[itileList];
        iatomSize     = PPStmp.iatomSize;
        iatomFreeSize = PPStmp.iatomFreeSize;
        jatomSize     = PPStmp.jatomSize;
        jatomFreeSize = PPStmp.jatomFreeSize;
      }

      // Write to global memory here to avoid register spilling
      if (doVirial) {
        if (wid == 0) {
          virialEnergy[itileList].shx = shx;
          virialEnergy[itileList].shy = shy;
          virialEnergy[itileList].shz = shz;
        }
      }

      // Load i-atom data (and shift coordinates)
      float4 xyzq_i = xyzq[iatomStart + wid];
      if (doAlch) part1 =  p[iatomStart + wid];
      xyzq_i.x += shx;
      xyzq_i.y += shy;
      xyzq_i.z += shz;
      int vdwtypei = vdwTypes[iatomStart + wid]*vdwCoefTableWidth;

      // Load i-atom data (and shift coordinates)
      BoundingBox boundingBoxI;
      if (doPairlist) {
        boundingBoxI = boundingBoxes[iatomStart/WARPSIZE];
        boundingBoxI.x += shx;
        boundingBoxI.y += shy;
        boundingBoxI.z += shz;
      }

      // Get i-atom global index
#ifdef USE_NEW_EXCL_METHOD
      int iatomIndex, minExclAtom, maxExclAtom;
#else
      int iatomIndex;
#endif
      if (doPairlist) {
#ifdef USE_NEW_EXCL_METHOD
        iatomIndex = atomIndex[iatomStart + wid];
        int2 tmp = minmaxExclAtom[iatomStart + wid];
        minExclAtom = tmp.x;
        maxExclAtom = tmp.y;
#else
        iatomIndex = atomIndex[iatomStart + wid];
#endif
      }

      // i-forces in registers
      // float3 iforce;
      iforce.x = 0.0f;
      iforce.y = 0.0f;
      iforce.z = 0.0f;

      // float3 iforceSlow;
      if (doSlow) {
        iforceSlow.x = 0.0f;
        iforceSlow.y = 0.0f;
        iforceSlow.z = 0.0f;
      }

      // float energyVdw, energyElec, energySlow;
      if (doEnergy) {
        energyVdw       = 0.0f;
        energyVdw_s     = 0.0f;
        energyVdw_ti_1  = 0.0f;
        energyVdw_ti_2  = 0.0f;
        energyElec      = 0.0f;
        energyElec_ti_1 = 0.0f;
        energyElec_ti_2 = 0.0f;
        energyElec_s    = 0.0f;
        if (doSlow){
          energySlow      = 0.0f;
          energySlow_s    = 0.0f;
          energySlow_ti_1 = 0.0f;
          energySlow_ti_2 = 0.0f;
        }
      }

      // Number of exclusions
      // NOTE: Lowest bit is used as indicator bit for tile pairs:
      //       bit 0 tile has no atoms within pairlist cutoff
      //       bit 1 tile has atoms within pairlist cutoff
      // int nexcluded;
      if (doPairlist) nexcluded = 0;

      // Number of i loops and free atoms
      int nfreei;
      if (doPairlist) {
        int nloopi = min(iatomSize - iatomStart, WARPSIZE);
        nfreei = max(iatomFreeSize - iatomStart, 0);
        if (wid >= nloopi) {
          xyzq_i.x = -LARGE_FLOAT;
          xyzq_i.y = -LARGE_FLOAT;
          xyzq_i.z = -LARGE_FLOAT;
        }
      }

      // tile list stuff
      // int itileListLen;
      // int minJatomStart;
      if (doPairlist) {
        // minJatomStart = tileJatomStart[jtileStart];
        itileListLen = 0;
      }

      // Exclusion index and maxdiff
      int iexclIndex, iexclMaxdiff;
      if (doPairlist) {
        int2 tmp = exclIndexMaxDiff[iatomStart + wid];
        iexclIndex   = tmp.x;
        iexclMaxdiff = tmp.y;
      }
      s_iatomStart[iwarp] = iatomStart;
      
      // If the tile is within a patch, then the first jtile is a self tile
      if (patchInd.x == patchInd.y & zeroShift) {
        int jtile = jtileStart;
         // Load j-atom starting index and exclusion mask
        int jatomStart = tileJatomStart[jtile];

        float4 xyzq_j = xyzq[jatomStart + wid];
        WARP_SYNC(WARP_FULL_MASK); 
        if (doAlch) p2 =  p[jatomStart + wid];

        // Check for early bail
        // No point of early bail for self

        unsigned int excl = (doPairlist) ? 0 : tileExcls[jtile].excl[wid];
        int vdwtypej = vdwTypes[jatomStart + wid];
        s_vdwtypej[iwarp][wid] = vdwtypej;

        // Get i-atom global index
        if (doPairlist) {
          s_jatomIndex[iwarp][wid] = atomIndex[jatomStart + wid];
        }

        // Number of j loops and free atoms
        int nfreej;
        if (doPairlist) {
          int nloopj = min(jatomSize - jatomStart, WARPSIZE);
          nfreej = max(jatomFreeSize - jatomStart, 0);
          //if (nfreei == 0 && nfreej == 0) continue;
          if (wid >= nloopj) {
            xyzq_j.x = LARGE_FLOAT;
            xyzq_j.y = LARGE_FLOAT;
            xyzq_j.z = LARGE_FLOAT;
          }
        }
        s_xyzq[iwarp][wid] = xyzq_j;

        // DH - self requires that zeroShift is also set
        // DC - In this case self is always true
        const int modval = 2*WARPSIZE-1;

        s_jforce[iwarp][wid] = make_zero<jForceType>();
        if (doSlow)
          s_jforceSlow[iwarp][wid] = make_zero<jForceType>();
        WARP_SYNC(WARP_FULL_MASK);

        if (doPairlist) {
          // Build pair list
          // NOTE: Pairlist update, we must also include the diagonal since this is used
          //       in GBIS phase 2.
          // Clear the lowest (indicator) bit
          nexcluded &= (~1);

          // For self tiles, do the diagonal term (t=0).
          // NOTE: No energies are computed here, since this self-diagonal term is only for GBIS phase 2
          int j = (0 + wid) & modval;
          xyzq_j = s_xyzq[iwarp][j];
          float dx = xyzq_j.x - xyzq_i.x;
          float dy = xyzq_j.y - xyzq_i.y;
          float dz = xyzq_j.z - xyzq_i.z;
          float r2 = dx*dx + dy*dy + dz*dz;

          if (j < WARPSIZE && r2 < plcutoff2) {
            // We have atom pair within the pairlist cutoff => Set indicator bit
            nexcluded |= 1;
          }
          WARP_SYNC(WARP_FULL_MASK);

          // TODO this can be done in fewer iterations if we take advantage of Newtons's 3rd
#pragma unroll 4
          for (int t = 1;t < WARPSIZE;t++) {
            int j = (t + wid) & modval;

            // NOTE: __shfl() operation can give non-sense here because j may be >= WARPSIZE.
            //       However, if (j < WARPSIZE ..) below makes sure that these non-sense
            //       results are not used
            if (doAlch) part2 = WARP_SHUFFLE(WARP_FULL_MASK, p2, j, WARPSIZE);

            excl >>= 1;
            if (j < WARPSIZE) {            
              xyzq_j = s_xyzq[iwarp][j];
              float dx = xyzq_j.x - xyzq_i.x;
              float dy = xyzq_j.y - xyzq_i.y;
              float dz = xyzq_j.z - xyzq_i.z;
              float r2 = dx*dx + dy*dy + dz*dz;
              if (r2 < plcutoff2) {
                // We have atom pair within the pairlist cutoff => Set indicator bit
                nexcluded |= 1;
                if (j < nfreej || wid < nfreei) {
                  bool excluded = false;
                  int indexdiff = s_jatomIndex[iwarp][j] - iatomIndex;
                  if ( abs(indexdiff) <= iexclMaxdiff) {
                    indexdiff += iexclIndex;
                    int indexword = ((unsigned int) indexdiff) >> 5;

                    if ( indexword < MAX_CONST_EXCLUSIONS ) {
                      indexword = constExclusions[indexword];
                    } else {
                      indexword = overflowExclusions[indexword];
                    }

                    excluded = ((indexword & (1<<(indexdiff&31))) != 0);
                  }
                  if (excluded) nexcluded += 2;
                  if (!excluded) excl |= 0x80000000;
                  if(doAlch){
                    if(!excluded && r2 < cutoff2){
                      if(doShift){
                        if(doFEP){
                          calcForceEnergyFEP<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_s,
                            energyElec, energySlow, energyElec_s, energySlow_s);
                        }else{
                          calcForceEnergyTI<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_ti_1, 
                            energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                            energySlow, energySlow_ti_1, energySlow_ti_2);
                        }//if doFEP
                      }else{
                        if(doFEP){
                          calcForceEnergyFEP<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_s,
                            energyElec, energySlow, energyElec_s, energySlow_s);
                        }else{
                          calcForceEnergyTI<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_ti_1, 
                            energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                            energySlow, energySlow_ti_1, energySlow_ti_2);
                        }
                      }//if doShift
                    }//if !excluded && r2 < cutoff2
                  }else{
                    if (!excluded && r2 < cutoff2) {
                      if (doTable) {
                        calcForceEnergy<doEnergy, doSlow>(
                          r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                          vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                          vdwCoefTableTex, forceTableTex, energyTableTex,
                          iforce, iforceSlow,
                          s_jforce[iwarp][j],
                          s_jforceSlow[iwarp][j],
                          energyVdw, energyElec, energySlow);
                      } else {
                        calcForceEnergyMath<doEnergy, doSlow, jForceType>(
                          r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                          vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                          vdwCoefTableTex, forceTableTex, energyTableTex,
                          iforce, iforceSlow,
                          s_jforce[iwarp][j],
                          s_jforceSlow[iwarp][j],
                          energyVdw, energyElec, energySlow,
                          nbConstants);
                      }
                    }
                  } 
                }
              }
            }
            WARP_SYNC(WARP_FULL_MASK);            
         } // t
       } else {
          // Just compute forces
          excl >>= 1;
#pragma unroll 4
          for (int t = 1;t < WARPSIZE;t++) {
            if (doAlch) {
              int j = (t + wid) & modval;
              part2 = WARP_SHUFFLE(WARP_FULL_MASK, p2, j, WARPSIZE);
            }
            if ((excl & 1)) {
              int j = ((t + wid) & (WARPSIZE-1));
              xyzq_j = s_xyzq[iwarp][j];
              float dx = xyzq_j.x - xyzq_i.x;
              float dy = xyzq_j.y - xyzq_i.y;
              float dz = xyzq_j.z - xyzq_i.z;              

              float r2 = dx*dx + dy*dy + dz*dz;
              if(doAlch){
                  if(r2 < cutoff2){ // (r2 < cutoff2)
                      if(doShift){
                            if (doFEP){
                                calcForceEnergyFEP<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                                  r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                                  vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                                  vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                                  iforce, iforceSlow,
                                  s_jforce[iwarp][j],
                                  s_jforceSlow[iwarp][j],
                                  energyVdw, energyVdw_s,
                                  energyElec, energySlow, energyElec_s, energySlow_s);
                            }else{
                                calcForceEnergyTI<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                                  r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                                  vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                                  vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                                  iforce, iforceSlow,
                                  s_jforce[iwarp][j],
                                  s_jforceSlow[iwarp][j],
                                  energyVdw, energyVdw_ti_1, 
                                  energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                                  energySlow, energySlow_ti_1, energySlow_ti_2);
                            }//if doFEP
                      }else{
                        if(doFEP){
                          calcForceEnergyFEP<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j],
                            s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_s, 
                            energyElec, energySlow, energyElec_s, energySlow_s);
                        }else{
                          calcForceEnergyTI<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j],
                            s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_ti_1, 
                            energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                            energySlow, energySlow_ti_1, energySlow_ti_2);
                        }//if doFEP
                      }//doShift 
                }//r2 < cutoff
              }else {
                if (r2 < cutoff2) {
                  if (doTable) {
                    calcForceEnergy<doEnergy, doSlow>(
                      r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                      vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                      vdwCoefTableTex, forceTableTex, energyTableTex,
                      iforce, iforceSlow,
                      s_jforce[iwarp][j],
                      s_jforceSlow[iwarp][j],
                      energyVdw, energyElec, energySlow);
                  } else {
                    calcForceEnergyMath<doEnergy, doSlow, jForceType>(
                      r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                      vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                      vdwCoefTableTex, forceTableTex, energyTableTex,
                      iforce, iforceSlow,
                      s_jforce[iwarp][j],
                      s_jforceSlow[iwarp][j],
                      energyVdw, energyElec, energySlow,
                      nbConstants);
                  }
                }// (r2 < cutoff2)
              }//doAlch 
            } // (excl & 1)
            excl >>= 1;
            WARP_SYNC(WARP_FULL_MASK);
          } // t
        }
        WARP_SYNC(WARP_FULL_MASK);

        // Write j-forces
        storeForces<doSlow, jForceType>(jatomStart + wid, s_jforce[iwarp][wid], s_jforceSlow[iwarp][wid],
                            devForce_x, devForce_y, devForce_z,
                            devForceSlow_x, devForceSlow_y, devForceSlow_z);
        // Write exclusions
        if (doPairlist) {
          const unsigned int warp_exclude = WARP_BALLOT(WARP_FULL_MASK, nexcluded & 1);
          const unsigned int warp_any_exclude = WARP_BALLOT(WARP_FULL_MASK, excl);
          if (warp_exclude) {
            int anyexcl = warp_any_exclude ? 1 : 0;
            anyexcl |= 65536;
            // Mark this jtile as non-empty:
            //  VdW:      1 if tile has atom pairs within pairlist cutoff and some these atoms interact
            //  GBIS: 65536 if tile has atom pairs within pairlist cutoff but not necessary interacting (i.e. these atoms are fixed or excluded)
            if (wid == 0 && anyexcl) jtiles[jtile] = anyexcl;
            // Store exclusions
            tileExcls[jtile].excl[wid] = excl;
            // itileListLen:
            // lower 16 bits number of tiles with atom pairs within pairlist cutoff that interact
            // upper 16 bits number of tiles with atom pairs within pairlist cutoff (but not necessary interacting)
            itileListLen += anyexcl;
            // NOTE, this minJatomStart is only stored once for the first tile list entry
            // minJatomStart = min(minJatomStart, jatomStart);
          }
        }
        jtileStart++;
      }

      WARP_SYNC(WARP_FULL_MASK);        

      for (int jtile=jtileStart; jtile <= jtileEnd; jtile++) {
        int jatomStart = 0;
        unsigned int excl = 0;
        int vdwtypej = 0;
        float4 xyzq_j;

         // Load j-atom starting index and exclusion mask
        jatomStart = tileJatomStart[jtile];

        xyzq_j = xyzq[jatomStart + wid];
        if (doAlch) p2 =  p[jatomStart + wid];

        // Check for early bail
        // DC - I found this was slower
        //if (doPairlist) {
        //  float r2bb = distsq(boundingBoxI, xyzq_j);
        //  if (WARP_ALL(WARP_FULL_MASK, r2bb > plcutoff2)) continue;
        //}

        excl = (doPairlist) ? 0 : tileExcls[jtile].excl[wid];
        vdwtypej = vdwTypes[jatomStart + wid];
        s_vdwtypej[iwarp][wid] = vdwtypej;

        // Get i-atom global index
        if (doPairlist) {
          s_jatomIndex[iwarp][wid] = atomIndex[jatomStart + wid];
        }

        // Number of j loops and free atoms
        int nfreej;
        if (doPairlist) {
          int nloopj = min(jatomSize - jatomStart, WARPSIZE);
          nfreej = max(jatomFreeSize - jatomStart, 0);
          //if (nfreei == 0 && nfreej == 0) continue;
          if (wid >= nloopj) {
            xyzq_j.x = LARGE_FLOAT;
            xyzq_j.y = LARGE_FLOAT;
            xyzq_j.z = LARGE_FLOAT;
          }
        }
        if (wid == 0) {
          s_jatomStart[iwarp] = jatomStart;
        }
        WARP_SYNC(WARP_FULL_MASK);
        s_xyzq[iwarp][wid] = xyzq_j;

        // DH - self requires that zeroShift is also set
        // DC - In this case self is always false
        const int modval = WARPSIZE-1;

        s_jforce[iwarp][wid] = make_zero<jForceType>();
        if (doSlow)
          s_jforceSlow[iwarp][wid] = make_zero<jForceType>(); 
        WARP_SYNC(WARP_FULL_MASK);

        if (doPairlist) {
          // Build pair list
          // NOTE: Pairlist update, we must also include the diagonal since this is used
          //       in GBIS phase 2.
          // Clear the lowest (indicator) bit
          nexcluded &= (~1);

#pragma unroll 4
          for (int t = 0;t < WARPSIZE;t++) {
            const int j = (t + wid) & modval;

            // NOTE: __shfl() operation can give non-sense here because j may be >= WARPSIZE.
            //       However, if (j < WARPSIZE ..) below makes sure that these non-sense
            //       results are not used
            if (doAlch) part2 = WARP_SHUFFLE(WARP_FULL_MASK, p2, j, WARPSIZE);

            excl >>= 1;
            xyzq_j = s_xyzq[iwarp][j];
            float dx = xyzq_j.x - xyzq_i.x;
            float dy = xyzq_j.y - xyzq_i.y;
            float dz = xyzq_j.z - xyzq_i.z;
            float r2 = dx*dx + dy*dy + dz*dz;
            if (r2 < plcutoff2) {
              // We have atom pair within the pairlist cutoff => Set indicator bit
              nexcluded |= 1;
              if (j < nfreej || wid < nfreei) {
                bool excluded = false;
                int indexdiff = s_jatomIndex[iwarp][j] - iatomIndex;
                if ( abs(indexdiff) <= iexclMaxdiff) {
                  indexdiff += iexclIndex;
                  int indexword = ((unsigned int) indexdiff) >> 5;

                  if ( indexword < MAX_CONST_EXCLUSIONS ) {
                    indexword = constExclusions[indexword];
                  } else {
                    indexword = overflowExclusions[indexword];
                  }

                  excluded = ((indexword & (1<<(indexdiff&31))) != 0);
                }
                if (excluded) nexcluded += 2;
                if (!excluded) excl |= 0x80000000;
                if(doAlch){
                  if(!excluded && r2 < cutoff2){
                    if(doShift){
                      if(doFEP){
                        calcForceEnergyFEP<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                          r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                          vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                          vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                          iforce, iforceSlow,
                          s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                          energyVdw, energyVdw_s,
                          energyElec, energySlow, energyElec_s, energySlow_s);
                      }else{
                        calcForceEnergyTI<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                          r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                          vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                          vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                          iforce, iforceSlow,
                          s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                          energyVdw, energyVdw_ti_1, 
                          energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                          energySlow, energySlow_ti_1, energySlow_ti_2);
                      }//if doFEP
                    }else{
                      if(doFEP){
                        calcForceEnergyFEP<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                          r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                          vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                          vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                          iforce, iforceSlow,
                          s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                          energyVdw, energyVdw_s,
                          energyElec, energySlow, energyElec_s, energySlow_s);
                      }else{
                        calcForceEnergyTI<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                          r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                          vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                          vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                          iforce, iforceSlow,
                          s_jforce[iwarp][j], s_jforceSlow[iwarp][j],
                          energyVdw, energyVdw_ti_1, 
                          energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                          energySlow, energySlow_ti_1, energySlow_ti_2);
                      }
                    }//if doShift
                  }//if !excluded && r2 < cutoff2
                }else{
                  if (!excluded && r2 < cutoff2) {
                    if (doTable) {
                      calcForceEnergy<doEnergy, doSlow>(
                        r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                        vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                        vdwCoefTableTex, forceTableTex, energyTableTex,
                        iforce, iforceSlow,
                        s_jforce[iwarp][j],
                        s_jforceSlow[iwarp][j],
                        energyVdw, energyElec, energySlow);
                    } else {
                      calcForceEnergyMath<doEnergy, doSlow, jForceType>(
                        r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                        vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                        vdwCoefTableTex, forceTableTex, energyTableTex,
                        iforce, iforceSlow,
                        s_jforce[iwarp][j],
                        s_jforceSlow[iwarp][j],
                        energyVdw, energyElec, energySlow,
                        nbConstants);
                    }
                  } 
                }
              }
            }
          WARP_SYNC(WARP_FULL_MASK);            
         } // t
       } else {
          // Just compute forces
#pragma unroll 4
          for (int t = 0; t < WARPSIZE; t++) {
            const int j = ((t + wid) & (WARPSIZE-1));
            if (doAlch) {
              part2 = WARP_SHUFFLE(WARP_FULL_MASK, p2, j, WARPSIZE);
            }
            if ((excl & 1)) {
              xyzq_j = s_xyzq[iwarp][j];
              float dx = xyzq_j.x - xyzq_i.x;
              float dy = xyzq_j.y - xyzq_i.y;
              float dz = xyzq_j.z - xyzq_i.z;              

              float r2 = dx*dx + dy*dy + dz*dz;
              if(doAlch){
                  if(r2 < cutoff2){ // (r2 < cutoff2)
                      if(doShift){
                            if (doFEP){
                                calcForceEnergyFEP<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                                  r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                                  vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                                  vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                                  iforce, iforceSlow,
                                  s_jforce[iwarp][j],
                                  s_jforceSlow[iwarp][j],
                                  energyVdw, energyVdw_s,
                                  energyElec, energySlow, energyElec_s, energySlow_s);
                            }else{
                                calcForceEnergyTI<doEnergy, doSlow, true, doAlchVdwForceSwitching, jForceType>(
                                  r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                                  vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                                  vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                                  iforce, iforceSlow,
                                  s_jforce[iwarp][j],
                                  s_jforceSlow[iwarp][j],
                                  energyVdw, energyVdw_ti_1, 
                                  energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                                  energySlow, energySlow_ti_1, energySlow_ti_2);
                            }//if doFEP
                      }else{
                        if(doFEP){
                          calcForceEnergyFEP<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j],
                            s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_s, 
                            energyElec, energySlow, energyElec_s, energySlow_s);
                        }else{
                          calcForceEnergyTI<doEnergy, doSlow, false, doAlchVdwForceSwitching, jForceType>(
                            r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                            vdwtypei, s_vdwtypej[iwarp][j], part1, part2,
                            vdwCoefTable, vdwCoefTableTex, forceTableTex, energyTableTex,
                            iforce, iforceSlow,
                            s_jforce[iwarp][j],
                            s_jforceSlow[iwarp][j],
                            energyVdw, energyVdw_ti_1, 
                            energyVdw_ti_2, energyElec, energyElec_ti_1, energyElec_ti_2, 
                            energySlow, energySlow_ti_1, energySlow_ti_2);
                        }//if doFEP
                      }//doShift 
                }//r2 < cutoff
              } else {
                if (r2 < cutoff2) {
                  if (doTable) {
                    calcForceEnergy<doEnergy, doSlow>(
                      r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                      vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                      vdwCoefTableTex, forceTableTex, energyTableTex,
                      iforce, iforceSlow,
                      s_jforce[iwarp][j],
                      s_jforceSlow[iwarp][j],
                      energyVdw, energyElec, energySlow);
                  } else {
                    calcForceEnergyMath<doEnergy, doSlow, jForceType>(
                      r2, xyzq_i.w, xyzq_j.w, dx, dy, dz,
                      vdwtypei, s_vdwtypej[iwarp][j], vdwCoefTable,
                      vdwCoefTableTex, forceTableTex, energyTableTex,
                      iforce, iforceSlow,
                      s_jforce[iwarp][j],
                      s_jforceSlow[iwarp][j],
                      energyVdw, energyElec, energySlow,
                      nbConstants);
                  }
                }// (r2 < cutoff2)
              }//doAlch 
            } // (excl & 1)
            excl >>= 1;
            WARP_SYNC(WARP_FULL_MASK);
          } // t
        }

        // Write j-forces
        storeForces<doSlow, jForceType>(s_jatomStart[iwarp] + wid, s_jforce[iwarp][wid], s_jforceSlow[iwarp][wid],
                            devForce_x, devForce_y, devForce_z,
                            devForceSlow_x, devForceSlow_y, devForceSlow_z);
        // Write exclusions
        if (doPairlist) {
          const unsigned int warp_exclude = WARP_BALLOT(WARP_FULL_MASK, nexcluded & 1);
          const unsigned int warp_any_exclude = WARP_BALLOT(WARP_FULL_MASK, excl);
          if (warp_exclude) {
            int anyexcl = warp_any_exclude ? 1 : 0;
            anyexcl |= 65536;
            // Mark this jtile as non-empty:
            //  VdW:      1 if tile has atom pairs within pairlist cutoff and some these atoms interact
            //  GBIS: 65536 if tile has atom pairs within pairlist cutoff but not necessary interacting (i.e. these atoms are fixed or excluded)
            if (wid == 0 && anyexcl) jtiles[jtile] = anyexcl;
            // Store exclusions
            tileExcls[jtile].excl[wid] = excl;
            // itileListLen:
            // lower 16 bits number of tiles with atom pairs within pairlist cutoff that interact
            // upper 16 bits number of tiles with atom pairs within pairlist cutoff (but not necessary interacting)
            itileListLen += anyexcl;
            // NOTE, this minJatomStart is only stored once for the first tile list entry
            // minJatomStart = min(minJatomStart, jatomStart);
          }
          WARP_SYNC(WARP_FULL_MASK);        
        }
      } // jtile

      // Write i-forces
      storeForces<doSlow, float3>(s_iatomStart[iwarp] + wid, iforce, iforceSlow,
                          devForce_x, devForce_y, devForce_z,
                          devForceSlow_x, devForceSlow_y, devForceSlow_z);
    }
    // Done with computation

    // Save pairlist stuff
    if (doPairlist) {

      // Warp index (0...warpsize-1)
      const int wid = threadIdx.x % WARPSIZE;

      if (wid == 0) {
        // minJatomStart is in range [0 ... atomStorageSize-1]
        //int atom0 = (minJatomStart)/WARPSIZE;
        // int atom0 = 0;
        // int storageOffset = atomStorageSize/WARPSIZE;
        // int itileListLen = 0;
        // for (int jtile=jtileStart;jtile <= jtileEnd;jtile++) itileListLen += jtiles[jtile];
        // Store 0 if itileListLen == 0
        // tileListDepth[itileList] = (itileListLen > 0)*(itileListLen*storageOffset + atom0);
        tileListDepth[itileList] = itileListLen;
        tileListOrder[itileList] = itileList;
        // Number of active tilelists with tile with atom pairs within pairlist cutoff that interact
        if ((itileListLen & 65535) > 0) atomicAdd(&tileListStat->numTileLists, 1);
        // Number of active tilelists with tiles with atom pairs within pairlist cutoff (but not necessary interacting)
        if (itileListLen > 0) atomicAdd(&tileListStat->numTileListsGBIS, 1);
        // NOTE: always numTileListsGBIS >= numTileLists
      }

      typedef hipcub::WarpReduce<int> WarpReduceInt;
      __shared__ typename WarpReduceInt::TempStorage tempStorage[NONBONDKERNEL_NUM_WARP];
      const int warpId = threadIdx.x / WARPSIZE;
      // Remove indicator bit
      nexcluded >>= 1;
      volatile int nexcludedWarp = WarpReduceInt(tempStorage[warpId]).Sum(nexcluded);
      if (wid == 0) atomicAdd(&tileListStat->numExcluded, nexcludedWarp);

    }

    if (doVirial) {
      // Warp index (0...warpsize-1)
      const int wid = threadIdx.x % WARPSIZE;

      typedef hipcub::WarpReduce<float> WarpReduce;
      __shared__ typename WarpReduce::TempStorage tempStorage[NONBONDKERNEL_NUM_WARP];
      const int warpId = threadIdx.x / WARPSIZE;
      volatile float iforcexSum = WarpReduce(tempStorage[warpId]).Sum(iforce.x);
      WARP_SYNC(WARP_FULL_MASK);
      volatile float iforceySum = WarpReduce(tempStorage[warpId]).Sum(iforce.y);
      WARP_SYNC(WARP_FULL_MASK);
      volatile float iforcezSum = WarpReduce(tempStorage[warpId]).Sum(iforce.z);
      WARP_SYNC(WARP_FULL_MASK);
      if (wid == 0) {
        virialEnergy[itileList].forcex = iforcexSum;
        virialEnergy[itileList].forcey = iforceySum;
        virialEnergy[itileList].forcez = iforcezSum;
      }

      if (doSlow) {
        iforcexSum = WarpReduce(tempStorage[warpId]).Sum(iforceSlow.x);
        WARP_SYNC(WARP_FULL_MASK);
        iforceySum = WarpReduce(tempStorage[warpId]).Sum(iforceSlow.y);
        WARP_SYNC(WARP_FULL_MASK);
        iforcezSum = WarpReduce(tempStorage[warpId]).Sum(iforceSlow.z);
        WARP_SYNC(WARP_FULL_MASK);
        if (wid == 0) {
          virialEnergy[itileList].forceSlowx = iforcexSum;
          virialEnergy[itileList].forceSlowy = iforceySum;
          virialEnergy[itileList].forceSlowz = iforcezSum;
        }
      }
    }

    // Reduce energy
    if (doEnergy) {
      // NOTE: We must hand write these warp-wide reductions to avoid excess register spillage
      //       (Why does CUB suck here?)
#pragma unroll
      for (int i=16;i >= 1;i/=2) {
        energyVdw += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyVdw, i, 32);
        energyElec += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyElec, i, 32);
        if(doFEP) energyVdw_s += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyVdw_s, i, 32);
        if(doFEP) energyElec_s += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyElec_s, i, 32);
        if(doTI){
           energyVdw_ti_1  += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyVdw_ti_1, i, 32);
           energyVdw_ti_2  += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyVdw_ti_2, i, 32);
           energyElec_ti_1 += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyElec_ti_1, i, 32);
           energyElec_ti_2 += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energyElec_ti_2, i, 32);
        }
        if (doSlow){ 
          energySlow += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energySlow, i, 32);
          if(doFEP) energySlow_s += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energySlow_s, i, 32);
          if(doTI){
            energySlow_ti_1 += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energySlow_ti_1, i, 32);
            energySlow_ti_2 += WARP_SHUFFLE_XOR(WARP_FULL_MASK, energySlow_ti_2, i, 32);
          }
        }
      }

      if (threadIdx.x % WARPSIZE == 0) {
        virialEnergy[itileList].energyVdw  = energyVdw;
        virialEnergy[itileList].energyElec = energyElec;
        if (doFEP) virialEnergy[itileList].energyVdw_s  = energyVdw_s;
        if (doFEP) virialEnergy[itileList].energyElec_s = energyElec_s;
        if(doTI){
           virialEnergy[itileList].energyVdw_ti_1   = energyVdw_ti_1;
           virialEnergy[itileList].energyVdw_ti_2   = energyVdw_ti_2;
           virialEnergy[itileList].energyElec_ti_1  = energyElec_ti_1;
           virialEnergy[itileList].energyElec_ti_2  = energyElec_ti_2;
        }
        if (doSlow) {
          virialEnergy[itileList].energySlow = energySlow;
          if(doFEP) virialEnergy[itileList].energySlow_s = energySlow_s;
          if (doTI){
            virialEnergy[itileList].energySlow_ti_1 = energySlow_ti_1;
            virialEnergy[itileList].energySlow_ti_2 = energySlow_ti_2;
          }
        }
      }
    }
    // XXX TODO: Disable streaming and see what happens
    // Let's try to set
    if (doStreaming) {
      // Make sure devForces and devForcesSlow have been written into device memory
      WARP_SYNC(WARP_FULL_MASK);
      __threadfence();

      int patchDone[2] = {false, false};
      const int wid = threadIdx.x % WARPSIZE;
      if (wid == 0) {
        int patchCountOld0 = atomicInc(&patchNumCount[patchInd.x], (unsigned int)(patchNumList.x-1));
        patchDone[0] = (patchCountOld0 + 1 == patchNumList.x);
        if (patchInd.x != patchInd.y) {
          int patchCountOld1 = atomicInc(&patchNumCount[patchInd.y], (unsigned int)(patchNumList.y-1));
          patchDone[1] = (patchCountOld1 + 1 == patchNumList.y);
        }
      }

      patchDone[0] = WARP_ANY(WARP_FULL_MASK, patchDone[0]);
      patchDone[1] = WARP_ANY(WARP_FULL_MASK, patchDone[1]);

      if (patchDone[0]) {
        // Patch 1 is done, write onto host-mapped memory
        CudaPatchRecord patch = cudaPatches[patchInd.x];
        int start = patch.atomStart;
        int end   = start + patch.numAtoms;
        for (int i=start+wid;i < end;i+=WARPSIZE) {
          mapForces[i] = make_float4(devForce_x[i],
              devForce_y[i], devForce_z[i], devForce_w[i]);
          if (doSlow) {
            mapForcesSlow[i] = make_float4(devForceSlow_x[i],
                devForceSlow_y[i], devForceSlow_z[i], devForceSlow_w[i]);
          }
        }
      }
      if (patchDone[1]) {
        // Patch 2 is done
        CudaPatchRecord patch = cudaPatches[patchInd.y];
        int start = patch.atomStart;
        int end   = start + patch.numAtoms;
        for (int i=start+wid;i < end;i+=WARPSIZE) {
          mapForces[i] = make_float4(devForce_x[i],
              devForce_y[i], devForce_z[i], devForce_w[i]);
          if (doSlow) {
            mapForcesSlow[i] = make_float4(devForceSlow_x[i],
                devForceSlow_y[i], devForceSlow_z[i], devForceSlow_w[i]);
          }
        }
      }

      if (patchDone[0] || patchDone[1]) {
        // Make sure mapForces and mapForcesSlow are up-to-date
        WARP_SYNC(WARP_FULL_MASK);
        __threadfence_system();
        // Add patch into "patchReadyQueue"
        if (wid == 0) {
          if (patchDone[0]) {
            int ind = atomicAdd(&tileListStat->patchReadyQueueCount, 1);
            // int ind = atomicInc((unsigned int *)&mapPatchReadyQueue[numPatches], numPatches-1);
            mapPatchReadyQueue[ind] = patchInd.x;
          }
          if (patchDone[1]) {
            int ind = atomicAdd(&tileListStat->patchReadyQueueCount, 1);
            // int ind = atomicInc((unsigned int *)&mapPatchReadyQueue[numPatches], numPatches-1);
            mapPatchReadyQueue[ind] = patchInd.y;
          }
        }
      }
    }

    if (doStreaming && outputOrder != NULL && threadIdx.x % WARPSIZE == 0) {
      int index = atomicAdd(&tileListStat->outputOrderIndex, 1);
      outputOrder[index] = itileList;
    }
  } // if (itileList < numTileLists)
}

//
// Finish up - reduce virials from nonbonded kernel
//
#define REDUCENONBONDEDVIRIALKERNEL_NUM_WARP 32
__global__ void reduceNonbondedVirialKernel(const bool doSlow,
  const int atomStorageSize,
  const float4* __restrict__ xyzq,
  const float4* __restrict__ devForces, const float4* __restrict__ devForcesSlow,
  VirialEnergy* __restrict__ virialEnergy) {

  for (int ibase = blockIdx.x*blockDim.x;ibase < atomStorageSize;ibase += blockDim.x*gridDim.x)
  {
    int i = ibase + threadIdx.x;

    // Set to zero to avoid nan*0
    float4 pos;
    pos.x = 0.0f;
    pos.y = 0.0f;
    pos.z = 0.0f;
    float4 force, forceSlow;
    force.x = 0.0f;
    force.y = 0.0f;
    force.z = 0.0f;
    forceSlow.x = 0.0f;
    forceSlow.y = 0.0f;
    forceSlow.z = 0.0f;
    if (i < atomStorageSize) {
      pos = xyzq[i];
      force = devForces[i];
      if (doSlow) forceSlow = devForcesSlow[i];
    }
    // Reduce across the entire thread block
    float vxxt = force.x*pos.x;
    float vxyt = force.x*pos.y;
    float vxzt = force.x*pos.z;
    float vyxt = force.y*pos.x;
    float vyyt = force.y*pos.y;
    float vyzt = force.y*pos.z;
    float vzxt = force.z*pos.x;
    float vzyt = force.z*pos.y;
    float vzzt = force.z*pos.z;
    // atomicAdd(&virialEnergy->virial[0], (double)vxx);
    // atomicAdd(&virialEnergy->virial[1], (double)vxy);
    // atomicAdd(&virialEnergy->virial[2], (double)vxz);
    // atomicAdd(&virialEnergy->virial[3], (double)vyx);
    // atomicAdd(&virialEnergy->virial[4], (double)vyy);
    // atomicAdd(&virialEnergy->virial[5], (double)vyz);
    // atomicAdd(&virialEnergy->virial[6], (double)vzx);
    // atomicAdd(&virialEnergy->virial[7], (double)vzy);
    // atomicAdd(&virialEnergy->virial[8], (double)vzz);

    typedef hipcub::BlockReduce<float, REDUCENONBONDEDVIRIALKERNEL_NUM_WARP*WARPSIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    volatile float vxx = BlockReduce(tempStorage).Sum(vxxt); BLOCK_SYNC;
    volatile float vxy = BlockReduce(tempStorage).Sum(vxyt); BLOCK_SYNC;
    volatile float vxz = BlockReduce(tempStorage).Sum(vxzt); BLOCK_SYNC;
    volatile float vyx = BlockReduce(tempStorage).Sum(vyxt); BLOCK_SYNC;
    volatile float vyy = BlockReduce(tempStorage).Sum(vyyt); BLOCK_SYNC;
    volatile float vyz = BlockReduce(tempStorage).Sum(vyzt); BLOCK_SYNC;
    volatile float vzx = BlockReduce(tempStorage).Sum(vzxt); BLOCK_SYNC;
    volatile float vzy = BlockReduce(tempStorage).Sum(vzyt); BLOCK_SYNC;
    volatile float vzz = BlockReduce(tempStorage).Sum(vzzt); BLOCK_SYNC;
    if (threadIdx.x == 0) {
      atomicAdd(&virialEnergy->virial[0], (double)vxx);
      atomicAdd(&virialEnergy->virial[1], (double)vxy);
      atomicAdd(&virialEnergy->virial[2], (double)vxz);
      atomicAdd(&virialEnergy->virial[3], (double)vyx);
      atomicAdd(&virialEnergy->virial[4], (double)vyy);
      atomicAdd(&virialEnergy->virial[5], (double)vyz);
      atomicAdd(&virialEnergy->virial[6], (double)vzx);
      atomicAdd(&virialEnergy->virial[7], (double)vzy);
      atomicAdd(&virialEnergy->virial[8], (double)vzz);
    }

    if (doSlow) {
      // if (isnan(forceSlow.x) || isnan(forceSlow.y) || isnan(forceSlow.z))
      float vxxSlowt = forceSlow.x*pos.x;
      float vxySlowt = forceSlow.x*pos.y;
      float vxzSlowt = forceSlow.x*pos.z;
      float vyxSlowt = forceSlow.y*pos.x;
      float vyySlowt = forceSlow.y*pos.y;
      float vyzSlowt = forceSlow.y*pos.z;
      float vzxSlowt = forceSlow.z*pos.x;
      float vzySlowt = forceSlow.z*pos.y;
      float vzzSlowt = forceSlow.z*pos.z;
      // atomicAdd(&virialEnergy->virialSlow[0], (double)vxxSlow);
      // atomicAdd(&virialEnergy->virialSlow[1], (double)vxySlow);
      // atomicAdd(&virialEnergy->virialSlow[2], (double)vxzSlow);
      // atomicAdd(&virialEnergy->virialSlow[3], (double)vyxSlow);
      // atomicAdd(&virialEnergy->virialSlow[4], (double)vyySlow);
      // atomicAdd(&virialEnergy->virialSlow[5], (double)vyzSlow);
      // atomicAdd(&virialEnergy->virialSlow[6], (double)vzxSlow);
      // atomicAdd(&virialEnergy->virialSlow[7], (double)vzySlow);
      // atomicAdd(&virialEnergy->virialSlow[8], (double)vzzSlow);
      volatile float vxxSlow = BlockReduce(tempStorage).Sum(vxxSlowt); BLOCK_SYNC;
      volatile float vxySlow = BlockReduce(tempStorage).Sum(vxySlowt); BLOCK_SYNC;
      volatile float vxzSlow = BlockReduce(tempStorage).Sum(vxzSlowt); BLOCK_SYNC;
      volatile float vyxSlow = BlockReduce(tempStorage).Sum(vyxSlowt); BLOCK_SYNC;
      volatile float vyySlow = BlockReduce(tempStorage).Sum(vyySlowt); BLOCK_SYNC;
      volatile float vyzSlow = BlockReduce(tempStorage).Sum(vyzSlowt); BLOCK_SYNC;
      volatile float vzxSlow = BlockReduce(tempStorage).Sum(vzxSlowt); BLOCK_SYNC;
      volatile float vzySlow = BlockReduce(tempStorage).Sum(vzySlowt); BLOCK_SYNC;
      volatile float vzzSlow = BlockReduce(tempStorage).Sum(vzzSlowt); BLOCK_SYNC;
      if (threadIdx.x == 0) {
        atomicAdd(&virialEnergy->virialSlow[0], (double)vxxSlow);
        atomicAdd(&virialEnergy->virialSlow[1], (double)vxySlow);
        atomicAdd(&virialEnergy->virialSlow[2], (double)vxzSlow);
        atomicAdd(&virialEnergy->virialSlow[3], (double)vyxSlow);
        atomicAdd(&virialEnergy->virialSlow[4], (double)vyySlow);
        atomicAdd(&virialEnergy->virialSlow[5], (double)vyzSlow);
        atomicAdd(&virialEnergy->virialSlow[6], (double)vzxSlow);
        atomicAdd(&virialEnergy->virialSlow[7], (double)vzySlow);
        atomicAdd(&virialEnergy->virialSlow[8], (double)vzzSlow);
      }
    }
  
  }
}

#define REDUCEVIRIALENERGYKERNEL_NUM_WARP 32
__global__ void reduceVirialEnergyKernel(
  const bool doEnergy, const bool doVirial, const bool doSlow,
  const int numTileLists,
  const TileListVirialEnergy* __restrict__ tileListVirialEnergy,
  VirialEnergy* __restrict__ virialEnergy) {

  for (int ibase = blockIdx.x*blockDim.x;ibase < numTileLists;ibase += blockDim.x*gridDim.x)
  {
    int itileList = ibase + threadIdx.x;
    TileListVirialEnergy ve;
    if (itileList < numTileLists) {
      ve = tileListVirialEnergy[itileList];
    } else {
      // Set to zero to avoid nan*0
      if (doVirial) {
        ve.shx = 0.0f;
        ve.shy = 0.0f;
        ve.shz = 0.0f;
        ve.forcex = 0.0f;
        ve.forcey = 0.0f;
        ve.forcez = 0.0f;
        ve.forceSlowx = 0.0f;
        ve.forceSlowy = 0.0f;
        ve.forceSlowz = 0.0f;
      }
      if (doEnergy) {
         ve.energyVdw    = 0.0;
         ve.energyVdw_s  = 0.0;
         ve.energyElec   = 0.0;
         ve.energySlow   = 0.0;
         ve.energyElec_s = 0.0;
         ve.energySlow_s = 0.0;
         
         /* TI stuff */
         ve.energyVdw_ti_1 = 0.0;
         ve.energyVdw_ti_2 = 0.0;
         ve.energyElec_ti_1 = 0.0;
         ve.energyElec_ti_2 = 0.0;
         ve.energySlow_ti_1 = 0.0;
         ve.energySlow_ti_2 = 0.0;
        // ve.energyGBIS = 0.0;
      }
    }

    if (doVirial) {
      typedef hipcub::BlockReduce<float, REDUCEVIRIALENERGYKERNEL_NUM_WARP*WARPSIZE> BlockReduce;
      __shared__ typename BlockReduce::TempStorage tempStorage;
      float vxxt = ve.forcex*ve.shx;
      float vxyt = ve.forcex*ve.shy;
      float vxzt = ve.forcex*ve.shz;
      float vyxt = ve.forcey*ve.shx;
      float vyyt = ve.forcey*ve.shy;
      float vyzt = ve.forcey*ve.shz;
      float vzxt = ve.forcez*ve.shx;
      float vzyt = ve.forcez*ve.shy;
      float vzzt = ve.forcez*ve.shz;
      volatile float vxx = BlockReduce(tempStorage).Sum(vxxt); BLOCK_SYNC;
      volatile float vxy = BlockReduce(tempStorage).Sum(vxyt); BLOCK_SYNC;
      volatile float vxz = BlockReduce(tempStorage).Sum(vxzt); BLOCK_SYNC;
      volatile float vyx = BlockReduce(tempStorage).Sum(vyxt); BLOCK_SYNC;
      volatile float vyy = BlockReduce(tempStorage).Sum(vyyt); BLOCK_SYNC;
      volatile float vyz = BlockReduce(tempStorage).Sum(vyzt); BLOCK_SYNC;
      volatile float vzx = BlockReduce(tempStorage).Sum(vzxt); BLOCK_SYNC;
      volatile float vzy = BlockReduce(tempStorage).Sum(vzyt); BLOCK_SYNC;
      volatile float vzz = BlockReduce(tempStorage).Sum(vzzt); BLOCK_SYNC;
      if (threadIdx.x == 0) {
        atomicAdd(&virialEnergy->virial[0], (double)vxx);
        atomicAdd(&virialEnergy->virial[1], (double)vxy);
        atomicAdd(&virialEnergy->virial[2], (double)vxz);
        atomicAdd(&virialEnergy->virial[3], (double)vyx);
        atomicAdd(&virialEnergy->virial[4], (double)vyy);
        atomicAdd(&virialEnergy->virial[5], (double)vyz);
        atomicAdd(&virialEnergy->virial[6], (double)vzx);
        atomicAdd(&virialEnergy->virial[7], (double)vzy);
        atomicAdd(&virialEnergy->virial[8], (double)vzz);
      }

      if (doSlow) {
        typedef hipcub::BlockReduce<float, REDUCEVIRIALENERGYKERNEL_NUM_WARP*WARPSIZE> BlockReduce;
        __shared__ typename BlockReduce::TempStorage tempStorage;
        float vxxt = ve.forceSlowx*ve.shx;
        float vxyt = ve.forceSlowx*ve.shy;
        float vxzt = ve.forceSlowx*ve.shz;
        float vyxt = ve.forceSlowy*ve.shx;
        float vyyt = ve.forceSlowy*ve.shy;
        float vyzt = ve.forceSlowy*ve.shz;
        float vzxt = ve.forceSlowz*ve.shx;
        float vzyt = ve.forceSlowz*ve.shy;
        float vzzt = ve.forceSlowz*ve.shz;
        volatile float vxx = BlockReduce(tempStorage).Sum(vxxt); BLOCK_SYNC;
        volatile float vxy = BlockReduce(tempStorage).Sum(vxyt); BLOCK_SYNC;
        volatile float vxz = BlockReduce(tempStorage).Sum(vxzt); BLOCK_SYNC;
        volatile float vyx = BlockReduce(tempStorage).Sum(vyxt); BLOCK_SYNC;
        volatile float vyy = BlockReduce(tempStorage).Sum(vyyt); BLOCK_SYNC;
        volatile float vyz = BlockReduce(tempStorage).Sum(vyzt); BLOCK_SYNC;
        volatile float vzx = BlockReduce(tempStorage).Sum(vzxt); BLOCK_SYNC;
        volatile float vzy = BlockReduce(tempStorage).Sum(vzyt); BLOCK_SYNC;
        volatile float vzz = BlockReduce(tempStorage).Sum(vzzt); BLOCK_SYNC;
        if (threadIdx.x == 0) {
          atomicAdd(&virialEnergy->virialSlow[0], (double)vxx);
          atomicAdd(&virialEnergy->virialSlow[1], (double)vxy);
          atomicAdd(&virialEnergy->virialSlow[2], (double)vxz);
          atomicAdd(&virialEnergy->virialSlow[3], (double)vyx);
          atomicAdd(&virialEnergy->virialSlow[4], (double)vyy);
          atomicAdd(&virialEnergy->virialSlow[5], (double)vyz);
          atomicAdd(&virialEnergy->virialSlow[6], (double)vzx);
          atomicAdd(&virialEnergy->virialSlow[7], (double)vzy);
          atomicAdd(&virialEnergy->virialSlow[8], (double)vzz);
        }
      }
    }

    if (doEnergy) {
      typedef hipcub::BlockReduce<double, REDUCEVIRIALENERGYKERNEL_NUM_WARP*WARPSIZE> BlockReduce;
      /* Maybe we should guard the TI and FEP energies, since those are not to be calculated on regular MDs */
      __shared__ typename BlockReduce::TempStorage tempStorage;
      volatile double energyVdw      = BlockReduce(tempStorage).Sum(ve.energyVdw); BLOCK_SYNC;
      volatile double energyVdw_s    = BlockReduce(tempStorage).Sum(ve.energyVdw_s); BLOCK_SYNC;
      volatile double energyElec     = BlockReduce(tempStorage).Sum(ve.energyElec); BLOCK_SYNC;
      volatile double energyElec_s   = BlockReduce(tempStorage).Sum(ve.energyElec_s); BLOCK_SYNC;
      volatile double energyVdw_ti_1 = BlockReduce(tempStorage).Sum(ve.energyVdw_ti_1); BLOCK_SYNC;
      volatile double energyVdw_ti_2 = BlockReduce(tempStorage).Sum(ve.energyVdw_ti_2); BLOCK_SYNC;
      volatile double energyElec_ti_1= BlockReduce(tempStorage).Sum(ve.energyElec_ti_1); BLOCK_SYNC;
      volatile double energyElec_ti_2= BlockReduce(tempStorage).Sum(ve.energyElec_ti_2); BLOCK_SYNC;
      if (threadIdx.x == 0){
          atomicAdd(&virialEnergy->energyVdw,    (double)energyVdw);
          atomicAdd(&virialEnergy->energyVdw_s,  (double)energyVdw_s);
          atomicAdd(&virialEnergy->energyElec,   (double)energyElec);
          atomicAdd(&virialEnergy->energyElec_s, (double)energyElec_s);
          atomicAdd(&virialEnergy->energyVdw_ti_1,  (double)energyVdw_ti_1);
          atomicAdd(&virialEnergy->energyVdw_ti_2,  (double)energyVdw_ti_2);
          atomicAdd(&virialEnergy->energyElec_ti_1, (double)energyElec_ti_1);
          atomicAdd(&virialEnergy->energyElec_ti_2, (double)energyElec_ti_2);
      }
      if (doSlow) {
        volatile double energySlow     = BlockReduce(tempStorage).Sum(ve.energySlow); BLOCK_SYNC;
        volatile double energySlow_s   = BlockReduce(tempStorage).Sum(ve.energySlow_s); BLOCK_SYNC;
        volatile double energySlow_ti_1 = BlockReduce(tempStorage).Sum(ve.energySlow_ti_1); BLOCK_SYNC;
        volatile double energySlow_ti_2 = BlockReduce(tempStorage).Sum(ve.energySlow_ti_2); BLOCK_SYNC;
        if (threadIdx.x == 0) {
          atomicAdd(&virialEnergy->energySlow,     (double)energySlow);
          atomicAdd(&virialEnergy->energySlow_s,   (double)energySlow_s);
          atomicAdd(&virialEnergy->energySlow_ti_1,(double)energySlow_ti_1);
          atomicAdd(&virialEnergy->energySlow_ti_2,(double)energySlow_ti_2);
        }
      }
      // if (doGBIS) {
      //   double energyGBIS = BlockReduce(tempStorage).Sum(ve.energyGBIS); BLOCK_SYNC;
      //   if (threadIdx.x == 0) atomicAdd(&virialEnergy->energyGBIS, (double)energyGBIS);
      // }
    }

  }

}

#define REDUCEGBISENERGYKERNEL_NUM_WARP 32
__global__ void reduceGBISEnergyKernel(const int numTileLists,
  const TileListVirialEnergy* __restrict__ tileListVirialEnergy,
  VirialEnergy* __restrict__ virialEnergy) {

  for (int ibase = blockIdx.x*blockDim.x;ibase < numTileLists;ibase += blockDim.x*gridDim.x)
  {
    int itileList = ibase + threadIdx.x;
    double energyGBISt = 0.0;
    if (itileList < numTileLists) {
      energyGBISt = tileListVirialEnergy[itileList].energyGBIS;
    }

    typedef hipcub::BlockReduce<double, REDUCEVIRIALENERGYKERNEL_NUM_WARP*WARPSIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    volatile double energyGBIS = BlockReduce(tempStorage).Sum(energyGBISt); BLOCK_SYNC;
    if (threadIdx.x == 0) atomicAdd(&virialEnergy->energyGBIS, (double)energyGBIS);
  }

}

// ##############################################################################################
// ##############################################################################################
// ##############################################################################################

CudaComputeNonbondedKernel::CudaComputeNonbondedKernel(int deviceID, CudaNonbondedTables& cudaNonbondedTables,
  bool doStreaming) : deviceID(deviceID), cudaNonbondedTables(cudaNonbondedTables), doStreaming(doStreaming) {
  
  cudaCheck(hipSetDevice(deviceID));

  d_exclusionsByAtom = NULL;

  overflowExclusions = NULL;
  overflowExclusionsSize = 0;

  exclIndexMaxDiff = NULL;
  exclIndexMaxDiffSize = 0;

  atomIndex = NULL;
  atomIndexSize = 0;

  vdwTypes = NULL;
  vdwTypesSize = 0;

  patchNumCount = NULL;
  patchNumCountSize = 0;

  patchReadyQueue = NULL;
  patchReadyQueueSize = 0;

  force_x = force_y = force_z = force_w = NULL;
  forceSize = 0;
  forceSlow_x = forceSlow_y = forceSlow_z = forceSlow_w = NULL;
  forceSlowSize = 0;
}

void CudaComputeNonbondedKernel::reallocate_forceSOA(int atomStorageSize)
{
#if 0
  size_t forceSizeCurrent;

  // reallocate_device will update forceSizeCurrent, so we need to reset it to the current
  // value for each array
  forceSizeCurrent = forceSize;
  reallocate_device<float>(&force_x, &forceSizeCurrent, atomStorageSize, 1.4f);
  forceSizeCurrent = forceSize;
  reallocate_device<float>(&force_y, &forceSizeCurrent, atomStorageSize, 1.4f);
  forceSizeCurrent = forceSize;
  reallocate_device<float>(&force_z, &forceSizeCurrent, atomStorageSize, 1.4f);
  forceSizeCurrent = forceSize;
  reallocate_device<float>(&force_w, &forceSizeCurrent, atomStorageSize, 1.4f);


  size_t forceSlowSizeCurrent;
  forceSlowSizeCurrent = forceSlowSize;
  reallocate_device<float>(&forceSlow_x, &forceSlowSizeCurrent, atomStorageSize, 1.4f);
  forceSlowSizeCurrent = forceSlowSize;
  reallocate_device<float>(&forceSlow_y, &forceSlowSizeCurrent, atomStorageSize, 1.4f);
  forceSlowSizeCurrent = forceSlowSize;
  reallocate_device<float>(&forceSlow_z, &forceSlowSizeCurrent, atomStorageSize, 1.4f);
  forceSlowSizeCurrent = forceSlowSize;
  reallocate_device<float>(&forceSlow_w, &forceSlowSizeCurrent, atomStorageSize, 1.4f);  
#else
  reallocate_device<float>(&force_x, &forceSize, atomStorageSize*8, 1.4f);
  force_y = force_x + atomStorageSize;
  force_z = force_y + atomStorageSize;
  force_w = force_z + atomStorageSize;
  forceSlow_x = force_w + atomStorageSize;
  forceSlow_y = forceSlow_x + atomStorageSize;
  forceSlow_z = forceSlow_y + atomStorageSize;
  forceSlow_w = forceSlow_z + atomStorageSize;
#endif
}

CudaComputeNonbondedKernel::~CudaComputeNonbondedKernel() {
  cudaCheck(hipSetDevice(deviceID));
  if (overflowExclusions != NULL) deallocate_device<unsigned int>(&overflowExclusions);
  if (exclIndexMaxDiff != NULL) deallocate_device<int2>(&exclIndexMaxDiff);
  if (atomIndex != NULL) deallocate_device<int>(&atomIndex);
  if (vdwTypes != NULL) deallocate_device<int>(&vdwTypes);
  if (patchNumCount != NULL) deallocate_device<unsigned int>(&patchNumCount);
  if (patchReadyQueue != NULL) deallocate_host<int>(&patchReadyQueue);
#if 0
  if (force_x != NULL) deallocate_device<float>(&force_x);
  if (force_y != NULL) deallocate_device<float>(&force_y);
  if (force_z != NULL) deallocate_device<float>(&force_z);
  if (force_w != NULL) deallocate_device<float>(&force_w);
  if (forceSlow_x != NULL) deallocate_device<float>(&forceSlow_x);
  if (forceSlow_y != NULL) deallocate_device<float>(&forceSlow_y);
  if (forceSlow_z != NULL) deallocate_device<float>(&forceSlow_z);
  if (forceSlow_w != NULL) deallocate_device<float>(&forceSlow_w);  
#else
  if (force_x != NULL) deallocate_device<float>(&force_x);
#endif
}

void CudaComputeNonbondedKernel::updateVdwTypesExcl(const int atomStorageSize, const int* h_vdwTypes,
  const int2* h_exclIndexMaxDiff, const int* h_atomIndex, hipStream_t stream) {

  reallocate_device<int>(&vdwTypes, &vdwTypesSize, atomStorageSize, OVERALLOC);
  reallocate_device<int2>(&exclIndexMaxDiff, &exclIndexMaxDiffSize, atomStorageSize, OVERALLOC);
  reallocate_device<int>(&atomIndex, &atomIndexSize, atomStorageSize, OVERALLOC);

  copy_HtoD<int>(h_vdwTypes, vdwTypes, atomStorageSize, stream);
  copy_HtoD<int2>(h_exclIndexMaxDiff, exclIndexMaxDiff, atomStorageSize, stream);
  copy_HtoD<int>(h_atomIndex, atomIndex, atomStorageSize, stream);
}

int* CudaComputeNonbondedKernel::getPatchReadyQueue() {
  if (!doStreaming) {
    NAMD_die("CudaComputeNonbondedKernel::getPatchReadyQueue() called on non-streaming kernel");
  }
  return patchReadyQueue;
}

template <int doSlow>
__global__ void transposeForcesKernel(float4 *f, float4 *fSlow,
                                      float *fx, float *fy, float *fz, float *fw,
                                      float *fSlowx, float *fSlowy, float *fSlowz, float *fSloww,
                                      int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (tid < n) {
    f[tid] = make_float4(fx[tid], fy[tid], fz[tid], fw[tid]);
    fx[tid] = 0.f; fy[tid] = 0.f; fz[tid] = 0.f; fw[tid] = 0.f;
    if (doSlow) {
      fSlow[tid] = make_float4(fSlowx[tid], fSlowy[tid], fSlowz[tid], fSloww[tid]);
      fSlowx[tid] = 0.f; fSlowy[tid] = 0.f; fSlowz[tid] = 0.f; fSloww[tid] = 0.f;
    }
  }
}



void CudaComputeNonbondedKernel::nonbondedForce(CudaTileListKernel& tlKernel,
  const int atomStorageSize, const bool atomsChanged, const bool doMinimize,
  const bool doPairlist, const bool doEnergy, const bool doVirial, 
  const bool doSlow, const bool doAlch, const bool doAlchVdwForceSwitching,
  const bool doFEP, const bool doTI, const bool doTable,
  const float3 lata, const float3 latb, const float3 latc,
  const float4* h_xyzq, const float cutoff2,
  const CudaNBConstants nbConstants,
  float4* d_forces, float4* d_forcesSlow,
  float4* h_forces, float4* h_forcesSlow, AlchData *srcFlags, 
  bool lambdaWindowUpdated, char *part,
  bool CUDASOAintegrator, bool useDeviceMigration,
  hipStream_t stream) {  

#ifdef NODEGROUP_FORCE_REGISTER
  if (!atomsChanged && !CUDASOAintegrator) copy_HtoD<float4>(h_xyzq, tlKernel.get_xyzq(), atomStorageSize, stream);
#else
  if (!doPairlist) copy_HtoD<float4>(h_xyzq, tlKernel.get_xyzq(), atomStorageSize, stream);
#endif

  if (doAlch){
    // Copy partition to device. This is not necessary if both CUDASOAintegrator and useDeviceMigration
    // are true.
    if (doPairlist && (!CUDASOAintegrator || !useDeviceMigration)) { 
      copy_HtoD< char>(part, tlKernel.get_part(), atomStorageSize, stream);
    }
    //Copies flags to constant memory
    if(lambdaWindowUpdated) cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(alchflags), srcFlags, sizeof(AlchData)));
  }
  
  // XXX TODO: Get rid of the clears
  if(1){
     // clear_device_array<float4>(d_forces, atomStorageSize, stream);
     // if (doSlow) clear_device_array<float4>(d_forcesSlow, atomStorageSize, stream);
     // two clears
     // This needs to go. 
     if (doStreaming)  tlKernel.clearTileListStat(stream);
     if(atomsChanged || doMinimize){
      clear_device_array<float>(force_x, atomStorageSize*4, stream);
      if(doSlow) clear_device_array<float>(forceSlow_x, atomStorageSize*4, stream);
     }
  }

  // --- streaming ----
  float4* m_forces = NULL;
  float4* m_forcesSlow = NULL;
  int* m_patchReadyQueue = NULL;
  int numPatches = 0;
  unsigned int* patchNumCountPtr = NULL;
  if (doStreaming) {
    numPatches = tlKernel.getNumPatches();
    if (reallocate_device<unsigned int>(&patchNumCount, &patchNumCountSize, numPatches)) {
      // If re-allocated, clear array
      clear_device_array<unsigned int>(patchNumCount, numPatches, stream);
    }
    patchNumCountPtr = patchNumCount;
    bool re = reallocate_host<int>(&patchReadyQueue, &patchReadyQueueSize, numPatches, hipHostMallocMapped);
    if (re) {
      // If re-allocated, re-set to "-1"
      for (int i=0;i < numPatches;i++) patchReadyQueue[i] = -1;
    }
    cudaCheck(hipHostGetDevicePointer(&m_patchReadyQueue, patchReadyQueue, 0));
    cudaCheck(hipHostGetDevicePointer(&m_forces, h_forces, 0));
    cudaCheck(hipHostGetDevicePointer(&m_forcesSlow, h_forcesSlow, 0));
  }
  // -----------------

  if (doVirial || doEnergy) {
    tlKernel.setTileListVirialEnergyLength(tlKernel.getNumTileLists());
  }

  int shMemSize = 0;

  int* outputOrderPtr = tlKernel.getOutputOrder();

  int nwarp = NONBONDKERNEL_NUM_WARP;
  int nthread = WARPSIZE*nwarp;
  int start = 0;

#define APVERSION
#undef APVERSION

#ifdef APVERSION
#else
  int options = doEnergy + (doVirial << 1) + (doSlow << 2) +
    (doPairlist << 3) + (doAlch << 4) + (doFEP << 5) + (doTI << 6) + (doStreaming << 7) + (doTable << 8) + (doAlchVdwForceSwitching << 9);
#endif

  while (start < tlKernel.getNumTileLists()) {

    int nleft = tlKernel.getNumTileLists() - start;
    int nblock = min(deviceCUDA->getMaxNumBlocks(), (nleft-1)/nwarp+1);
#ifdef APVERSION
#define CALL(DOENERGY, DOVIRIAL, DOSLOW, DOPAIRLIST, DOALCH, DOFEP, DOTI, DOSTREAMING, DOALCHWDWFORCESWITCHING) \
    nonbondedForceKernel<DOENERGY, DOVIRIAL, DOSLOW, DOPAIRLIST, DOALCH, DOFEP, DOTI, DOSTREAMING, DOALCHWDWFORCESWITCHING> \
  <<< nblock, nthread, shMemSize, stream >>>  \
  (start, tlKernel.getNumTileLists(), tlKernel.getTileLists(), tlKernel.getTileExcls(), tlKernel.getTileJatomStart(), \
    cudaNonbondedTables.getVdwCoefTableWidth(), cudaNonbondedTables.getVdwCoefTable(), \
   vdwTypes, lata, latb, latc, tlKernel.get_xyzq(), cutoff2, nbConstants, \
    cudaNonbondedTables.getVdwCoefTableTex(), cudaNonbondedTables.getForceTableTex(), cudaNonbondedTables.getEnergyTableTex(), \
    atomStorageSize, tlKernel.get_plcutoff2(), tlKernel.getPatchPairs(), atomIndex, exclIndexMaxDiff, overflowExclusions, \
    tlKernel.getTileListDepth(), tlKernel.getTileListOrder(), tlKernel.getJtiles(), tlKernel.getTileListStatDevPtr(), \
    tlKernel.getBoundingBoxes(), d_forces, d_forcesSlow, \
    force_x, force_y, force_z, force_w, \
    forceSlow_x, forceSlow_y, forceSlow_z, forceSlow_w, \
    numPatches, patchNumCountPtr, tlKernel.getCudaPatches(), m_forces, m_forcesSlow, m_patchReadyQueue, \
    outputOrderPtr, tlKernel.getTileListVirialEnergy(), tlKernel.get_part()); called=true

    bool called = false;
    if (doStreaming) {
      if(!doAlch){
        if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 0, 0, 0, 1, 0);
        if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 0, 0, 0, 1, 0);
        if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 0, 0, 0, 1, 0);
        if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 0, 0, 0, 1, 0);
        if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 0, 0, 0, 1, 0);
        if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 0, 0, 0, 1, 0);
        if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 0, 0, 0, 1, 0);
        if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 0, 0, 0, 1, 0);
      

        if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 0, 0, 0, 1, 0);
        if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 0, 0, 0, 1, 0);
        if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 0, 0, 0, 1, 0);
        if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 0, 0, 0, 1, 0);
        if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 0, 0, 0, 1, 0);
        if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 0, 0, 0, 1, 0);
        if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 0, 0, 0, 1, 0);
        if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 0, 0, 0, 1, 0);
      }else{
        if(doFEP){
          if (doAlchVdwForceSwitching) {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 1, 0, 1, 1);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 1, 0, 1, 1);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 1, 0, 1, 1);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 1, 0, 1, 1);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 1, 0, 1, 1);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 1, 0, 1, 1);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 1, 0, 1, 1);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 1, 0, 1, 1);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 1, 0, 1, 1);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 1, 0, 1, 1);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 1, 0, 1, 1);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 1, 0, 1, 1);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 1, 0, 1, 1);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 1, 0, 1, 1);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 1, 0, 1, 1);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 1, 0, 1, 1);
          } else {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 1, 0, 1, 0);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 1, 0, 1, 0);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 1, 0, 1, 0);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 1, 0, 1, 0);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 1, 0, 1, 0);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 1, 0, 1, 0);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 1, 0, 1, 0);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 1, 0, 1, 0);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 1, 0, 1, 0);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 1, 0, 1, 0);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 1, 0, 1, 0);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 1, 0, 1, 0);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 1, 0, 1, 0);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 1, 0, 1, 0);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 1, 0, 1, 0);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 1, 0, 1, 0);
          } // doAlchVdwForceSwitching
         }else{
          // TI
          if (doAlchVdwForceSwitching) {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 0, 1, 1, 1);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 0, 1, 1, 1);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 0, 1, 1, 1);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 0, 1, 1, 1);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 0, 1, 1, 1);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 0, 1, 1, 1);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 0, 1, 1, 1);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 0, 1, 1, 1);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 0, 1, 1, 1);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 0, 1, 1, 1);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 0, 1, 1, 1);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 0, 1, 1, 1);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 0, 1, 1, 1);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 0, 1, 1, 1);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 0, 1, 1, 1);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 0, 1, 1, 1);
          } else {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 0, 1, 1, 0);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 0, 1, 1, 0);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 0, 1, 1, 0);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 0, 1, 1, 0);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 0, 1, 1, 0);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 0, 1, 1, 0);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 0, 1, 1, 0);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 0, 1, 1, 0);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 0, 1, 1, 0);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 0, 1, 1, 0);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 0, 1, 1, 0);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 0, 1, 1, 0);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 0, 1, 1, 0);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 0, 1, 1, 0);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 0, 1, 1, 0);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 0, 1, 1, 0);
          } // doAlchVdwForceSwitching
        } // doFEP
      } // doAlch
    } 
    else {
      // no streaming
      if(!doAlch){
        if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 0, 0, 0, 0, 0);
        if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 0, 0, 0, 0, 0);
        if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 0, 0, 0, 0, 0);
        if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 0, 0, 0, 0, 0);
        if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 0, 0, 0, 0, 0);
        if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 0, 0, 0, 0, 0);
        if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 0, 0, 0, 0, 0);
        if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 0, 0, 0, 0, 0);
      

        if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 0, 0, 0, 0, 0);
        if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 0, 0, 0, 0, 0);
        if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 0, 0, 0, 0, 0);
        if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 0, 0, 0, 0, 0);
        if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 0, 0, 0, 0, 0);
        if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 0, 0, 0, 0, 0);
        if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 0, 0, 0, 0, 0);
        if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 0, 0, 0, 0, 0);
      }else{
        if(doFEP){
          if (doAlchVdwForceSwitching) {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 1, 0, 0, 1);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 1, 0, 0, 1);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 1, 0, 0, 1);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 1, 0, 0, 1);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 1, 0, 0, 1);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 1, 0, 0, 1);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 1, 0, 0, 1);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 1, 0, 0, 1);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 1, 0, 0, 1);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 1, 0, 0, 1);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 1, 0, 0, 1);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 1, 0, 0, 1);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 1, 0, 0, 1);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 1, 0, 0, 1);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 1, 0, 0, 1);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 1, 0, 0, 1);
          } else {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 1, 0, 0, 0);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 1, 0, 0, 0);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 1, 0, 0, 0);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 1, 0, 0, 0);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 1, 0, 0, 0);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 1, 0, 0, 0);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 1, 0, 0, 0);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 1, 0, 0, 0);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 1, 0, 0, 0);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 1, 0, 0, 0);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 1, 0, 0, 0);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 1, 0, 0, 0);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 1, 0, 0, 0);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 1, 0, 0, 0);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 1, 0, 0, 0);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 1, 0, 0, 0);
          }
        }else{
          // TI
          if (doAlchVdwForceSwitching) {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 0, 1, 0, 1);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 0, 1, 0, 1);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 0, 1, 0, 1);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 0, 1, 0, 1);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 0, 1, 0, 1);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 0, 1, 0, 1);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 0, 1, 0, 1);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 0, 1, 0, 1);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 0, 1, 0, 1);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 0, 1, 0, 1);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 0, 1, 0, 1);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 0, 1, 0, 1);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 0, 1, 0, 1);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 0, 1, 0, 1);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 0, 1, 0, 1);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 0, 1, 0, 1);
          } else {
            if (!doEnergy && !doVirial && !doSlow && !doPairlist) CALL(0, 0, 0, 0, 1, 0, 1, 0, 0);
            if (!doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(0, 0, 1, 0, 1, 0, 1, 0, 0);
            if (!doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(0, 1, 0, 0, 1, 0, 1, 0, 0);
            if (!doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(0, 1, 1, 0, 1, 0, 1, 0, 0);
            if ( doEnergy && !doVirial && !doSlow && !doPairlist) CALL(1, 0, 0, 0, 1, 0, 1, 0, 0);
            if ( doEnergy && !doVirial &&  doSlow && !doPairlist) CALL(1, 0, 1, 0, 1, 0, 1, 0, 0);
            if ( doEnergy &&  doVirial && !doSlow && !doPairlist) CALL(1, 1, 0, 0, 1, 0, 1, 0, 0);
            if ( doEnergy &&  doVirial &&  doSlow && !doPairlist) CALL(1, 1, 1, 0, 1, 0, 1, 0, 0);

            if (!doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(0, 0, 0, 1, 1, 0, 1, 0, 0);
            if (!doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(0, 0, 1, 1, 1, 0, 1, 0, 0);
            if (!doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(0, 1, 0, 1, 1, 0, 1, 0, 0);
            if (!doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(0, 1, 1, 1, 1, 0, 1, 0, 0);
            if ( doEnergy && !doVirial && !doSlow &&  doPairlist) CALL(1, 0, 0, 1, 1, 0, 1, 0, 0);
            if ( doEnergy && !doVirial &&  doSlow &&  doPairlist) CALL(1, 0, 1, 1, 1, 0, 1, 0, 0);
            if ( doEnergy &&  doVirial && !doSlow &&  doPairlist) CALL(1, 1, 0, 1, 1, 0, 1, 0, 0);
            if ( doEnergy &&  doVirial &&  doSlow &&  doPairlist) CALL(1, 1, 1, 1, 1, 0, 1, 0, 0);
          }
        }//if doFEP
      }//if doAlch
    }//if doStreaming

    if (!called) {
      NAMD_die("CudaComputeNonbondedKernel::nonbondedForce, none of the kernels called");
    }

#else


#define CALL(DOENERGY, DOVIRIAL, DOSLOW, DOPAIRLIST, DOALCH, DOFEP, DOTI, DOSTREAMING, DOTABLE, DOALCHWDWFORCESWITCHING) \
    nonbondedForceKernel<DOENERGY, DOVIRIAL, DOSLOW, DOPAIRLIST, DOALCH, DOFEP, DOTI, DOSTREAMING, DOTABLE, DOALCHWDWFORCESWITCHING> \
  <<< nblock, nthread, shMemSize, stream >>>  \
  (start, tlKernel.getNumTileLists(), tlKernel.getTileLists(), tlKernel.getTileExcls(), tlKernel.getTileJatomStart(), \
    cudaNonbondedTables.getVdwCoefTableWidth(), cudaNonbondedTables.getVdwCoefTable(), \
    vdwTypes, lata, latb, latc, tlKernel.get_xyzq(), cutoff2, nbConstants, \
    cudaNonbondedTables.getVdwCoefTableTex(), cudaNonbondedTables.getForceTableTex(), cudaNonbondedTables.getEnergyTableTex(), \
    atomStorageSize, tlKernel.get_plcutoff2(), tlKernel.getPatchPairs(), atomIndex, exclIndexMaxDiff, overflowExclusions, \
    tlKernel.getTileListDepth(), tlKernel.getTileListOrder(), tlKernel.getJtiles(), tlKernel.getTileListStatDevPtr(), \
    tlKernel.getBoundingBoxes(), d_forces, d_forcesSlow, \
    force_x, force_y, force_z, force_w, \
    forceSlow_x, forceSlow_y, forceSlow_z, forceSlow_w, \
    numPatches, patchNumCountPtr, tlKernel.getCudaPatches(), m_forces, m_forcesSlow, m_patchReadyQueue, \
    outputOrderPtr, tlKernel.getTileListVirialEnergy(), tlKernel.get_part())

#ifdef DEBUG
    char errmsg[256];
#endif


    switch (options) {
      case   0: CALL(0, 0, 0, 0, 0, 0, 0, 0, 0, 0); break;
      case   1: CALL(1, 0, 0, 0, 0, 0, 0, 0, 0, 0); break;
      case   2: CALL(0, 1, 0, 0, 0, 0, 0, 0, 0, 0); break;
      case   3: CALL(1, 1, 0, 0, 0, 0, 0, 0, 0, 0); break;
      case   4: CALL(0, 0, 1, 0, 0, 0, 0, 0, 0, 0); break;
      case   5: CALL(1, 0, 1, 0, 0, 0, 0, 0, 0, 0); break;
      case   6: CALL(0, 1, 1, 0, 0, 0, 0, 0, 0, 0); break;
      case   7: CALL(1, 1, 1, 0, 0, 0, 0, 0, 0, 0); break;
      case   8: CALL(0, 0, 0, 1, 0, 0, 0, 0, 0, 0); break;
      case   9: CALL(1, 0, 0, 1, 0, 0, 0, 0, 0, 0); break;
      case  10: CALL(0, 1, 0, 1, 0, 0, 0, 0, 0, 0); break;
      case  11: CALL(1, 1, 0, 1, 0, 0, 0, 0, 0, 0); break;
      case  12: CALL(0, 0, 1, 1, 0, 0, 0, 0, 0, 0); break;
      case  13: CALL(1, 0, 1, 1, 0, 0, 0, 0, 0, 0); break;
      case  14: CALL(0, 1, 1, 1, 0, 0, 0, 0, 0, 0); break;
      case  15: CALL(1, 1, 1, 1, 0, 0, 0, 0, 0, 0); break;

#if 0
      case  16: CALL(0, 0, 0, 0, 1, 0, 0, 0, 0, 0); break;
      case  17: CALL(1, 0, 0, 0, 1, 0, 0, 0, 0, 0); break;
      case  18: CALL(0, 1, 0, 0, 1, 0, 0, 0, 0, 0); break;
      case  19: CALL(1, 1, 0, 0, 1, 0, 0, 0, 0, 0); break;
      case  20: CALL(0, 0, 1, 0, 1, 0, 0, 0, 0, 0); break;
      case  21: CALL(1, 0, 1, 0, 1, 0, 0, 0, 0, 0); break;
      case  22: CALL(0, 1, 1, 0, 1, 0, 0, 0, 0, 0); break;
      case  23: CALL(1, 1, 1, 0, 1, 0, 0, 0, 0, 0); break;
      case  24: CALL(0, 0, 0, 1, 1, 0, 0, 0, 0, 0); break;
      case  25: CALL(1, 0, 0, 1, 1, 0, 0, 0, 0, 0); break;
      case  26: CALL(0, 1, 0, 1, 1, 0, 0, 0, 0, 0); break;
      case  27: CALL(1, 1, 0, 1, 1, 0, 0, 0, 0, 0); break;
      case  28: CALL(0, 0, 1, 1, 1, 0, 0, 0, 0, 0); break;
      case  29: CALL(1, 0, 1, 1, 1, 0, 0, 0, 0, 0); break;
      case  30: CALL(0, 1, 1, 1, 1, 0, 0, 0, 0, 0); break;
      case  31: CALL(1, 1, 1, 1, 1, 0, 0, 0, 0, 0); break;

      case  32: CALL(0, 0, 0, 0, 0, 1, 0, 0, 0, 0); break;
      case  33: CALL(1, 0, 0, 0, 0, 1, 0, 0, 0, 0); break;
      case  34: CALL(0, 1, 0, 0, 0, 1, 0, 0, 0, 0); break;
      case  35: CALL(1, 1, 0, 0, 0, 1, 0, 0, 0, 0); break;
      case  36: CALL(0, 0, 1, 0, 0, 1, 0, 0, 0, 0); break;
      case  37: CALL(1, 0, 1, 0, 0, 1, 0, 0, 0, 0); break;
      case  38: CALL(0, 1, 1, 0, 0, 1, 0, 0, 0, 0); break;
      case  39: CALL(1, 1, 1, 0, 0, 1, 0, 0, 0, 0); break;
      case  40: CALL(0, 0, 0, 1, 0, 1, 0, 0, 0, 0); break;
      case  41: CALL(1, 0, 0, 1, 0, 1, 0, 0, 0, 0); break;
      case  42: CALL(0, 1, 0, 1, 0, 1, 0, 0, 0, 0); break;
      case  43: CALL(1, 1, 0, 1, 0, 1, 0, 0, 0, 0); break;
      case  44: CALL(0, 0, 1, 1, 0, 1, 0, 0, 0, 0); break;
      case  45: CALL(1, 0, 1, 1, 0, 1, 0, 0, 0, 0); break;
      case  46: CALL(0, 1, 1, 1, 0, 1, 0, 0, 0, 0); break;
      case  47: CALL(1, 1, 1, 1, 0, 1, 0, 0, 0, 0); break;

      case  48: CALL(0, 0, 0, 0, 1, 1, 0, 0, 0, 0); break;
      case  49: CALL(1, 0, 0, 0, 1, 1, 0, 0, 0, 0); break;
      case  50: CALL(0, 1, 0, 0, 1, 1, 0, 0, 0, 0); break;
      case  51: CALL(1, 1, 0, 0, 1, 1, 0, 0, 0, 0); break;
      case  52: CALL(0, 0, 1, 0, 1, 1, 0, 0, 0, 0); break;
      case  53: CALL(1, 0, 1, 0, 1, 1, 0, 0, 0, 0); break;
      case  54: CALL(0, 1, 1, 0, 1, 1, 0, 0, 0, 0); break;
      case  55: CALL(1, 1, 1, 0, 1, 1, 0, 0, 0, 0); break;
      case  56: CALL(0, 0, 0, 1, 1, 1, 0, 0, 0, 0); break;
      case  57: CALL(1, 0, 0, 1, 1, 1, 0, 0, 0, 0); break;
      case  58: CALL(0, 1, 0, 1, 1, 1, 0, 0, 0, 0); break;
      case  59: CALL(1, 1, 0, 1, 1, 1, 0, 0, 0, 0); break;
      case  60: CALL(0, 0, 1, 1, 1, 1, 0, 0, 0, 0); break;
      case  61: CALL(1, 0, 1, 1, 1, 1, 0, 0, 0, 0); break;
      case  62: CALL(0, 1, 1, 1, 1, 1, 0, 0, 0, 0); break;
      case  63: CALL(1, 1, 1, 1, 1, 1, 0, 0, 0, 0); break;

      case  64: CALL(0, 0, 0, 0, 0, 0, 1, 0, 0, 0); break;
      case  65: CALL(1, 0, 0, 0, 0, 0, 1, 0, 0, 0); break;
      case  66: CALL(0, 1, 0, 0, 0, 0, 1, 0, 0, 0); break;
      case  67: CALL(1, 1, 0, 0, 0, 0, 1, 0, 0, 0); break;
      case  68: CALL(0, 0, 1, 0, 0, 0, 1, 0, 0, 0); break;
      case  69: CALL(1, 0, 1, 0, 0, 0, 1, 0, 0, 0); break;
      case  70: CALL(0, 1, 1, 0, 0, 0, 1, 0, 0, 0); break;
      case  71: CALL(1, 1, 1, 0, 0, 0, 1, 0, 0, 0); break;
      case  72: CALL(0, 0, 0, 1, 0, 0, 1, 0, 0, 0); break;
      case  73: CALL(1, 0, 0, 1, 0, 0, 1, 0, 0, 0); break;
      case  74: CALL(0, 1, 0, 1, 0, 0, 1, 0, 0, 0); break;
      case  75: CALL(1, 1, 0, 1, 0, 0, 1, 0, 0, 0); break;
      case  76: CALL(0, 0, 1, 1, 0, 0, 1, 0, 0, 0); break;
      case  77: CALL(1, 0, 1, 1, 0, 0, 1, 0, 0, 0); break;
      case  78: CALL(0, 1, 1, 1, 0, 0, 1, 0, 0, 0); break;
      case  79: CALL(1, 1, 1, 1, 0, 0, 1, 0, 0, 0); break;

      case  80: CALL(0, 0, 0, 0, 1, 0, 1, 0, 0, 0); break;
      case  81: CALL(1, 0, 0, 0, 1, 0, 1, 0, 0, 0); break;
      case  82: CALL(0, 1, 0, 0, 1, 0, 1, 0, 0, 0); break;
      case  83: CALL(1, 1, 0, 0, 1, 0, 1, 0, 0, 0); break;
      case  84: CALL(0, 0, 1, 0, 1, 0, 1, 0, 0, 0); break;
      case  85: CALL(1, 0, 1, 0, 1, 0, 1, 0, 0, 0); break;
      case  86: CALL(0, 1, 1, 0, 1, 0, 1, 0, 0, 0); break;
      case  87: CALL(1, 1, 1, 0, 1, 0, 1, 0, 0, 0); break;
      case  88: CALL(0, 0, 0, 1, 1, 0, 1, 0, 0, 0); break;
      case  89: CALL(1, 0, 0, 1, 1, 0, 1, 0, 0, 0); break;
      case  90: CALL(0, 1, 0, 1, 1, 0, 1, 0, 0, 0); break;
      case  91: CALL(1, 1, 0, 1, 1, 0, 1, 0, 0, 0); break;
      case  92: CALL(0, 0, 1, 1, 1, 0, 1, 0, 0, 0); break;
      case  93: CALL(1, 0, 1, 1, 1, 0, 1, 0, 0, 0); break;
      case  94: CALL(0, 1, 1, 1, 1, 0, 1, 0, 0, 0); break;
      case  95: CALL(1, 1, 1, 1, 1, 0, 1, 0, 0, 0); break;

      case   96: CALL(0, 0, 0, 0, 0, 1, 1, 0, 0, 0); break;
      case   97: CALL(1, 0, 0, 0, 0, 1, 1, 0, 0, 0); break;
      case   98: CALL(0, 1, 0, 0, 0, 1, 1, 0, 0, 0); break;
      case   99: CALL(1, 1, 0, 0, 0, 1, 1, 0, 0, 0); break;
      case  100: CALL(0, 0, 1, 0, 0, 1, 1, 0, 0, 0); break;
      case  101: CALL(1, 0, 1, 0, 0, 1, 1, 0, 0, 0); break;
      case  102: CALL(0, 1, 1, 0, 0, 1, 1, 0, 0, 0); break;
      case  103: CALL(1, 1, 1, 0, 0, 1, 1, 0, 0, 0); break;
      case  104: CALL(0, 0, 0, 1, 0, 1, 1, 0, 0, 0); break;
      case  105: CALL(1, 0, 0, 1, 0, 1, 1, 0, 0, 0); break;
      case  106: CALL(0, 1, 0, 1, 0, 1, 1, 0, 0, 0); break;
      case  107: CALL(1, 1, 0, 1, 0, 1, 1, 0, 0, 0); break;
      case  108: CALL(0, 0, 1, 1, 0, 1, 1, 0, 0, 0); break;
      case  109: CALL(1, 0, 1, 1, 0, 1, 1, 0, 0, 0); break;
      case  110: CALL(0, 1, 1, 1, 0, 1, 1, 0, 0, 0); break;
      case  111: CALL(1, 1, 1, 1, 0, 1, 1, 0, 0, 0); break;

      case  112: CALL(0, 0, 0, 0, 1, 1, 1, 0, 0, 0); break;
      case  113: CALL(1, 0, 0, 0, 1, 1, 1, 0, 0, 0); break;
      case  114: CALL(0, 1, 0, 0, 1, 1, 1, 0, 0, 0); break;
      case  115: CALL(1, 1, 0, 0, 1, 1, 1, 0, 0, 0); break;
      case  116: CALL(0, 0, 1, 0, 1, 1, 1, 0, 0, 0); break;
      case  117: CALL(1, 0, 1, 0, 1, 1, 1, 0, 0, 0); break;
      case  118: CALL(0, 1, 1, 0, 1, 1, 1, 0, 0, 0); break;
      case  119: CALL(1, 1, 1, 0, 1, 1, 1, 0, 0, 0); break;
      case  120: CALL(0, 0, 0, 1, 1, 1, 1, 0, 0, 0); break;
      case  121: CALL(1, 0, 0, 1, 1, 1, 1, 0, 0, 0); break;
      case  122: CALL(0, 1, 0, 1, 1, 1, 1, 0, 0, 0); break;
      case  123: CALL(1, 1, 0, 1, 1, 1, 1, 0, 0, 0); break;
      case  124: CALL(0, 0, 1, 1, 1, 1, 1, 0, 0, 0); break;
      case  125: CALL(1, 0, 1, 1, 1, 1, 1, 0, 0, 0); break;
      case  126: CALL(0, 1, 1, 1, 1, 1, 1, 0, 0, 0); break;
      case  127: CALL(1, 1, 1, 1, 1, 1, 1, 0, 0, 0); break;

#endif
      case  128: CALL(0, 0, 0, 0, 0, 0, 0, 1, 0, 0); break;
      case  129: CALL(1, 0, 0, 0, 0, 0, 0, 1, 0, 0); break;
      case  130: CALL(0, 1, 0, 0, 0, 0, 0, 1, 0, 0); break;
      case  131: CALL(1, 1, 0, 0, 0, 0, 0, 1, 0, 0); break;
      case  132: CALL(0, 0, 1, 0, 0, 0, 0, 1, 0, 0); break;
      case  133: CALL(1, 0, 1, 0, 0, 0, 0, 1, 0, 0); break;
      case  134: CALL(0, 1, 1, 0, 0, 0, 0, 1, 0, 0); break;
      case  135: CALL(1, 1, 1, 0, 0, 0, 0, 1, 0, 0); break;
      case  136: CALL(0, 0, 0, 1, 0, 0, 0, 1, 0, 0); break;
      case  137: CALL(1, 0, 0, 1, 0, 0, 0, 1, 0, 0); break;
      case  138: CALL(0, 1, 0, 1, 0, 0, 0, 1, 0, 0); break;
      case  139: CALL(1, 1, 0, 1, 0, 0, 0, 1, 0, 0); break;
      case  140: CALL(0, 0, 1, 1, 0, 0, 0, 1, 0, 0); break;
      case  141: CALL(1, 0, 1, 1, 0, 0, 0, 1, 0, 0); break;
      case  142: CALL(0, 1, 1, 1, 0, 0, 0, 1, 0, 0); break;
      case  143: CALL(1, 1, 1, 1, 0, 0, 0, 1, 0, 0); break;

#if 0
      case  144: CALL(0, 0, 0, 0, 1, 0, 0, 1, 0, 0); break;
      case  145: CALL(1, 0, 0, 0, 1, 0, 0, 1, 0, 0); break;
      case  146: CALL(0, 1, 0, 0, 1, 0, 0, 1, 0, 0); break;
      case  147: CALL(1, 1, 0, 0, 1, 0, 0, 1, 0, 0); break;
      case  148: CALL(0, 0, 1, 0, 1, 0, 0, 1, 0, 0); break;
      case  149: CALL(1, 0, 1, 0, 1, 0, 0, 1, 0, 0); break;
      case  150: CALL(0, 1, 1, 0, 1, 0, 0, 1, 0, 0); break;
      case  151: CALL(1, 1, 1, 0, 1, 0, 0, 1, 0, 0); break;
      case  152: CALL(0, 0, 0, 1, 1, 0, 0, 1, 0, 0); break;
      case  153: CALL(1, 0, 0, 1, 1, 0, 0, 1, 0, 0); break;
      case  154: CALL(0, 1, 0, 1, 1, 0, 0, 1, 0, 0); break;
      case  155: CALL(1, 1, 0, 1, 1, 0, 0, 1, 0, 0); break;
      case  156: CALL(0, 0, 1, 1, 1, 0, 0, 1, 0, 0); break;
      case  157: CALL(1, 0, 1, 1, 1, 0, 0, 1, 0, 0); break;
      case  158: CALL(0, 1, 1, 1, 1, 0, 0, 1, 0, 0); break;
      case  159: CALL(1, 1, 1, 1, 1, 0, 0, 1, 0, 0); break;

      case  160: CALL(0, 0, 0, 0, 0, 1, 0, 1, 0, 0); break;
      case  161: CALL(1, 0, 0, 0, 0, 1, 0, 1, 0, 0); break;
      case  162: CALL(0, 1, 0, 0, 0, 1, 0, 1, 0, 0); break;
      case  163: CALL(1, 1, 0, 0, 0, 1, 0, 1, 0, 0); break;
      case  164: CALL(0, 0, 1, 0, 0, 1, 0, 1, 0, 0); break;
      case  165: CALL(1, 0, 1, 0, 0, 1, 0, 1, 0, 0); break;
      case  166: CALL(0, 1, 1, 0, 0, 1, 0, 1, 0, 0); break;
      case  167: CALL(1, 1, 1, 0, 0, 1, 0, 1, 0, 0); break;
      case  168: CALL(0, 0, 0, 1, 0, 1, 0, 1, 0, 0); break;
      case  169: CALL(1, 0, 0, 1, 0, 1, 0, 1, 0, 0); break;
      case  170: CALL(0, 1, 0, 1, 0, 1, 0, 1, 0, 0); break;
      case  171: CALL(1, 1, 0, 1, 0, 1, 0, 1, 0, 0); break;
      case  172: CALL(0, 0, 1, 1, 0, 1, 0, 1, 0, 0); break;
      case  173: CALL(1, 0, 1, 1, 0, 1, 0, 1, 0, 0); break;
      case  174: CALL(0, 1, 1, 1, 0, 1, 0, 1, 0, 0); break;
      case  175: CALL(1, 1, 1, 1, 0, 1, 0, 1, 0, 0); break;

      case  176: CALL(0, 0, 0, 0, 1, 1, 0, 1, 0, 0); break;
      case  177: CALL(1, 0, 0, 0, 1, 1, 0, 1, 0, 0); break;
      case  178: CALL(0, 1, 0, 0, 1, 1, 0, 1, 0, 0); break;
      case  179: CALL(1, 1, 0, 0, 1, 1, 0, 1, 0, 0); break;
      case  180: CALL(0, 0, 1, 0, 1, 1, 0, 1, 0, 0); break;
      case  181: CALL(1, 0, 1, 0, 1, 1, 0, 1, 0, 0); break;
      case  182: CALL(0, 1, 1, 0, 1, 1, 0, 1, 0, 0); break;
      case  183: CALL(1, 1, 1, 0, 1, 1, 0, 1, 0, 0); break;
      case  184: CALL(0, 0, 0, 1, 1, 1, 0, 1, 0, 0); break;
      case  185: CALL(1, 0, 0, 1, 1, 1, 0, 1, 0, 0); break;
      case  186: CALL(0, 1, 0, 1, 1, 1, 0, 1, 0, 0); break;
      case  187: CALL(1, 1, 0, 1, 1, 1, 0, 1, 0, 0); break;
      case  188: CALL(0, 0, 1, 1, 1, 1, 0, 1, 0, 0); break;
      case  189: CALL(1, 0, 1, 1, 1, 1, 0, 1, 0, 0); break;
      case  190: CALL(0, 1, 1, 1, 1, 1, 0, 1, 0, 0); break;
      case  191: CALL(1, 1, 1, 1, 1, 1, 0, 1, 0, 0); break;

      case  192: CALL(0, 0, 0, 0, 0, 0, 1, 1, 0, 0); break;
      case  193: CALL(1, 0, 0, 0, 0, 0, 1, 1, 0, 0); break;
      case  194: CALL(0, 1, 0, 0, 0, 0, 1, 1, 0, 0); break;
      case  195: CALL(1, 1, 0, 0, 0, 0, 1, 1, 0, 0); break;
      case  196: CALL(0, 0, 1, 0, 0, 0, 1, 1, 0, 0); break;
      case  197: CALL(1, 0, 1, 0, 0, 0, 1, 1, 0, 0); break;
      case  198: CALL(0, 1, 1, 0, 0, 0, 1, 1, 0, 0); break;
      case  199: CALL(1, 1, 1, 0, 0, 0, 1, 1, 0, 0); break;
      case  200: CALL(0, 0, 0, 1, 0, 0, 1, 1, 0, 0); break;
      case  201: CALL(1, 0, 0, 1, 0, 0, 1, 1, 0, 0); break;
      case  202: CALL(0, 1, 0, 1, 0, 0, 1, 1, 0, 0); break;
      case  203: CALL(1, 1, 0, 1, 0, 0, 1, 1, 0, 0); break;
      case  204: CALL(0, 0, 1, 1, 0, 0, 1, 1, 0, 0); break;
      case  205: CALL(1, 0, 1, 1, 0, 0, 1, 1, 0, 0); break;
      case  206: CALL(0, 1, 1, 1, 0, 0, 1, 1, 0, 0); break;
      case  207: CALL(1, 1, 1, 1, 0, 0, 1, 1, 0, 0); break;

      case  208: CALL(0, 0, 0, 0, 1, 0, 1, 1, 0, 0); break;
      case  209: CALL(1, 0, 0, 0, 1, 0, 1, 1, 0, 0); break;
      case  210: CALL(0, 1, 0, 0, 1, 0, 1, 1, 0, 0); break;
      case  211: CALL(1, 1, 0, 0, 1, 0, 1, 1, 0, 0); break;
      case  212: CALL(0, 0, 1, 0, 1, 0, 1, 1, 0, 0); break;
      case  213: CALL(1, 0, 1, 0, 1, 0, 1, 1, 0, 0); break;
      case  214: CALL(0, 1, 1, 0, 1, 0, 1, 1, 0, 0); break;
      case  215: CALL(1, 1, 1, 0, 1, 0, 1, 1, 0, 0); break;
      case  216: CALL(0, 0, 0, 1, 1, 0, 1, 1, 0, 0); break;
      case  217: CALL(1, 0, 0, 1, 1, 0, 1, 1, 0, 0); break;
      case  218: CALL(0, 1, 0, 1, 1, 0, 1, 1, 0, 0); break;
      case  219: CALL(1, 1, 0, 1, 1, 0, 1, 1, 0, 0); break;
      case  220: CALL(0, 0, 1, 1, 1, 0, 1, 1, 0, 0); break;
      case  221: CALL(1, 0, 1, 1, 1, 0, 1, 1, 0, 0); break;
      case  222: CALL(0, 1, 1, 1, 1, 0, 1, 1, 0, 0); break;
      case  223: CALL(1, 1, 1, 1, 1, 0, 1, 1, 0, 0); break;

      case  224: CALL(0, 0, 0, 0, 0, 1, 1, 1, 0, 0); break;
      case  225: CALL(1, 0, 0, 0, 0, 1, 1, 1, 0, 0); break;
      case  226: CALL(0, 1, 0, 0, 0, 1, 1, 1, 0, 0); break;
      case  227: CALL(1, 1, 0, 0, 0, 1, 1, 1, 0, 0); break;
      case  228: CALL(0, 0, 1, 0, 0, 1, 1, 1, 0, 0); break;
      case  229: CALL(1, 0, 1, 0, 0, 1, 1, 1, 0, 0); break;
      case  230: CALL(0, 1, 1, 0, 0, 1, 1, 1, 0, 0); break;
      case  231: CALL(1, 1, 1, 0, 0, 1, 1, 1, 0, 0); break;
      case  232: CALL(0, 0, 0, 1, 0, 1, 1, 1, 0, 0); break;
      case  233: CALL(1, 0, 0, 1, 0, 1, 1, 1, 0, 0); break;
      case  234: CALL(0, 1, 0, 1, 0, 1, 1, 1, 0, 0); break;
      case  235: CALL(1, 1, 0, 1, 0, 1, 1, 1, 0, 0); break;
      case  236: CALL(0, 0, 1, 1, 0, 1, 1, 1, 0, 0); break;
      case  237: CALL(1, 0, 1, 1, 0, 1, 1, 1, 0, 0); break;
      case  238: CALL(0, 1, 1, 1, 0, 1, 1, 1, 0, 0); break;
      case  239: CALL(1, 1, 1, 1, 0, 1, 1, 1, 0, 0); break;

      case  240: CALL(0, 0, 0, 0, 1, 1, 1, 1, 0, 0); break;
      case  241: CALL(1, 0, 0, 0, 1, 1, 1, 1, 0, 0); break;
      case  242: CALL(0, 1, 0, 0, 1, 1, 1, 1, 0, 0); break;
      case  243: CALL(1, 1, 0, 0, 1, 1, 1, 1, 0, 0); break;
      case  244: CALL(0, 0, 1, 0, 1, 1, 1, 1, 0, 0); break;
      case  245: CALL(1, 0, 1, 0, 1, 1, 1, 1, 0, 0); break;
      case  246: CALL(0, 1, 1, 0, 1, 1, 1, 1, 0, 0); break;
      case  247: CALL(1, 1, 1, 0, 1, 1, 1, 1, 0, 0); break;
      case  248: CALL(0, 0, 0, 1, 1, 1, 1, 1, 0, 0); break;
      case  249: CALL(1, 0, 0, 1, 1, 1, 1, 1, 0, 0); break;
      case  250: CALL(0, 1, 0, 1, 1, 1, 1, 1, 0, 0); break;
      case  251: CALL(1, 1, 0, 1, 1, 1, 1, 1, 0, 0); break;
      case  252: CALL(0, 0, 1, 1, 1, 1, 1, 1, 0, 0); break;
      case  253: CALL(1, 0, 1, 1, 1, 1, 1, 1, 0, 0); break;
      case  254: CALL(0, 1, 1, 1, 1, 1, 1, 1, 0, 0); break;
      case  255: CALL(1, 1, 1, 1, 1, 1, 1, 1, 0, 0); break;

#endif
      case  256: CALL(0, 0, 0, 0, 0, 0, 0, 0, 1, 0); break;
      case  257: CALL(1, 0, 0, 0, 0, 0, 0, 0, 1, 0); break;
      case  258: CALL(0, 1, 0, 0, 0, 0, 0, 0, 1, 0); break;
      case  259: CALL(1, 1, 0, 0, 0, 0, 0, 0, 1, 0); break;
      case  260: CALL(0, 0, 1, 0, 0, 0, 0, 0, 1, 0); break;
      case  261: CALL(1, 0, 1, 0, 0, 0, 0, 0, 1, 0); break;
      case  262: CALL(0, 1, 1, 0, 0, 0, 0, 0, 1, 0); break;
      case  263: CALL(1, 1, 1, 0, 0, 0, 0, 0, 1, 0); break;
      case  264: CALL(0, 0, 0, 1, 0, 0, 0, 0, 1, 0); break;
      case  265: CALL(1, 0, 0, 1, 0, 0, 0, 0, 1, 0); break;
      case  266: CALL(0, 1, 0, 1, 0, 0, 0, 0, 1, 0); break;
      case  267: CALL(1, 1, 0, 1, 0, 0, 0, 0, 1, 0); break;
      case  268: CALL(0, 0, 1, 1, 0, 0, 0, 0, 1, 0); break;
      case  269: CALL(1, 0, 1, 1, 0, 0, 0, 0, 1, 0); break;
      case  270: CALL(0, 1, 1, 1, 0, 0, 0, 0, 1, 0); break;
      case  271: CALL(1, 1, 1, 1, 0, 0, 0, 0, 1, 0); break;

#if 0
      case  272: CALL(0, 0, 0, 0, 1, 0, 0, 0, 1, 0); break;
      case  273: CALL(1, 0, 0, 0, 1, 0, 0, 0, 1, 0); break;
      case  274: CALL(0, 1, 0, 0, 1, 0, 0, 0, 1, 0); break;
      case  275: CALL(1, 1, 0, 0, 1, 0, 0, 0, 1, 0); break;
      case  276: CALL(0, 0, 1, 0, 1, 0, 0, 0, 1, 0); break;
      case  277: CALL(1, 0, 1, 0, 1, 0, 0, 0, 1, 0); break;
      case  278: CALL(0, 1, 1, 0, 1, 0, 0, 0, 1, 0); break;
      case  279: CALL(1, 1, 1, 0, 1, 0, 0, 0, 1, 0); break;
      case  280: CALL(0, 0, 0, 1, 1, 0, 0, 0, 1, 0); break;
      case  281: CALL(1, 0, 0, 1, 1, 0, 0, 0, 1, 0); break;
      case  282: CALL(0, 1, 0, 1, 1, 0, 0, 0, 1, 0); break;
      case  283: CALL(1, 1, 0, 1, 1, 0, 0, 0, 1, 0); break;
      case  284: CALL(0, 0, 1, 1, 1, 0, 0, 0, 1, 0); break;
      case  285: CALL(1, 0, 1, 1, 1, 0, 0, 0, 1, 0); break;
      case  286: CALL(0, 1, 1, 1, 1, 0, 0, 0, 1, 0); break;
      case  287: CALL(1, 1, 1, 1, 1, 0, 0, 0, 1, 0); break;

      case  288: CALL(0, 0, 0, 0, 0, 1, 0, 0, 1, 0); break;
      case  289: CALL(1, 0, 0, 0, 0, 1, 0, 0, 1, 0); break;
      case  290: CALL(0, 1, 0, 0, 0, 1, 0, 0, 1, 0); break;
      case  291: CALL(1, 1, 0, 0, 0, 1, 0, 0, 1, 0); break;
      case  292: CALL(0, 0, 1, 0, 0, 1, 0, 0, 1, 0); break;
      case  293: CALL(1, 0, 1, 0, 0, 1, 0, 0, 1, 0); break;
      case  294: CALL(0, 1, 1, 0, 0, 1, 0, 0, 1, 0); break;
      case  295: CALL(1, 1, 1, 0, 0, 1, 0, 0, 1, 0); break;
      case  296: CALL(0, 0, 0, 1, 0, 1, 0, 0, 1, 0); break;
      case  297: CALL(1, 0, 0, 1, 0, 1, 0, 0, 1, 0); break;
      case  298: CALL(0, 1, 0, 1, 0, 1, 0, 0, 1, 0); break;
      case  299: CALL(1, 1, 0, 1, 0, 1, 0, 0, 1, 0); break;
      case  300: CALL(0, 0, 1, 1, 0, 1, 0, 0, 1, 0); break;
      case  301: CALL(1, 0, 1, 1, 0, 1, 0, 0, 1, 0); break;
      case  302: CALL(0, 1, 1, 1, 0, 1, 0, 0, 1, 0); break;
      case  303: CALL(1, 1, 1, 1, 0, 1, 0, 0, 1, 0); break;

#endif
      case  304: CALL(0, 0, 0, 0, 1, 1, 0, 0, 1, 0); break;
      case  305: CALL(1, 0, 0, 0, 1, 1, 0, 0, 1, 0); break;
      case  306: CALL(0, 1, 0, 0, 1, 1, 0, 0, 1, 0); break;
      case  307: CALL(1, 1, 0, 0, 1, 1, 0, 0, 1, 0); break;
      case  308: CALL(0, 0, 1, 0, 1, 1, 0, 0, 1, 0); break;
      case  309: CALL(1, 0, 1, 0, 1, 1, 0, 0, 1, 0); break;
      case  310: CALL(0, 1, 1, 0, 1, 1, 0, 0, 1, 0); break;
      case  311: CALL(1, 1, 1, 0, 1, 1, 0, 0, 1, 0); break;
      case  312: CALL(0, 0, 0, 1, 1, 1, 0, 0, 1, 0); break;
      case  313: CALL(1, 0, 0, 1, 1, 1, 0, 0, 1, 0); break;
      case  314: CALL(0, 1, 0, 1, 1, 1, 0, 0, 1, 0); break;
      case  315: CALL(1, 1, 0, 1, 1, 1, 0, 0, 1, 0); break;
      case  316: CALL(0, 0, 1, 1, 1, 1, 0, 0, 1, 0); break;
      case  317: CALL(1, 0, 1, 1, 1, 1, 0, 0, 1, 0); break;
      case  318: CALL(0, 1, 1, 1, 1, 1, 0, 0, 1, 0); break;
      case  319: CALL(1, 1, 1, 1, 1, 1, 0, 0, 1, 0); break;

#if 0
      case  320: CALL(0, 0, 0, 0, 0, 0, 1, 0, 1, 0); break;
      case  321: CALL(1, 0, 0, 0, 0, 0, 1, 0, 1, 0); break;
      case  322: CALL(0, 1, 0, 0, 0, 0, 1, 0, 1, 0); break;
      case  323: CALL(1, 1, 0, 0, 0, 0, 1, 0, 1, 0); break;
      case  324: CALL(0, 0, 1, 0, 0, 0, 1, 0, 1, 0); break;
      case  325: CALL(1, 0, 1, 0, 0, 0, 1, 0, 1, 0); break;
      case  326: CALL(0, 1, 1, 0, 0, 0, 1, 0, 1, 0); break;
      case  327: CALL(1, 1, 1, 0, 0, 0, 1, 0, 1, 0); break;
      case  328: CALL(0, 0, 0, 1, 0, 0, 1, 0, 1, 0); break;
      case  329: CALL(1, 0, 0, 1, 0, 0, 1, 0, 1, 0); break;
      case  330: CALL(0, 1, 0, 1, 0, 0, 1, 0, 1, 0); break;
      case  331: CALL(1, 1, 0, 1, 0, 0, 1, 0, 1, 0); break;
      case  332: CALL(0, 0, 1, 1, 0, 0, 1, 0, 1, 0); break;
      case  333: CALL(1, 0, 1, 1, 0, 0, 1, 0, 1, 0); break;
      case  334: CALL(0, 1, 1, 1, 0, 0, 1, 0, 1, 0); break;
      case  335: CALL(1, 1, 1, 1, 0, 0, 1, 0, 1, 0); break;

#endif
      case  336: CALL(0, 0, 0, 0, 1, 0, 1, 0, 1, 0); break;
      case  337: CALL(1, 0, 0, 0, 1, 0, 1, 0, 1, 0); break;
      case  338: CALL(0, 1, 0, 0, 1, 0, 1, 0, 1, 0); break;
      case  339: CALL(1, 1, 0, 0, 1, 0, 1, 0, 1, 0); break;
      case  340: CALL(0, 0, 1, 0, 1, 0, 1, 0, 1, 0); break;
      case  341: CALL(1, 0, 1, 0, 1, 0, 1, 0, 1, 0); break;
      case  342: CALL(0, 1, 1, 0, 1, 0, 1, 0, 1, 0); break;
      case  343: CALL(1, 1, 1, 0, 1, 0, 1, 0, 1, 0); break;
      case  344: CALL(0, 0, 0, 1, 1, 0, 1, 0, 1, 0); break;
      case  345: CALL(1, 0, 0, 1, 1, 0, 1, 0, 1, 0); break;
      case  346: CALL(0, 1, 0, 1, 1, 0, 1, 0, 1, 0); break;
      case  347: CALL(1, 1, 0, 1, 1, 0, 1, 0, 1, 0); break;
      case  348: CALL(0, 0, 1, 1, 1, 0, 1, 0, 1, 0); break;
      case  349: CALL(1, 0, 1, 1, 1, 0, 1, 0, 1, 0); break;
      case  350: CALL(0, 1, 1, 1, 1, 0, 1, 0, 1, 0); break;
      case  351: CALL(1, 1, 1, 1, 1, 0, 1, 0, 1, 0); break;

#if 0
      case  352: CALL(0, 0, 0, 0, 0, 1, 1, 0, 1, 0); break;
      case  353: CALL(1, 0, 0, 0, 0, 1, 1, 0, 1, 0); break;
      case  354: CALL(0, 1, 0, 0, 0, 1, 1, 0, 1, 0); break;
      case  355: CALL(1, 1, 0, 0, 0, 1, 1, 0, 1, 0); break;
      case  356: CALL(0, 0, 1, 0, 0, 1, 1, 0, 1, 0); break;
      case  357: CALL(1, 0, 1, 0, 0, 1, 1, 0, 1, 0); break;
      case  358: CALL(0, 1, 1, 0, 0, 1, 1, 0, 1, 0); break;
      case  359: CALL(1, 1, 1, 0, 0, 1, 1, 0, 1, 0); break;
      case  360: CALL(0, 0, 0, 1, 0, 1, 1, 0, 1, 0); break;
      case  361: CALL(1, 0, 0, 1, 0, 1, 1, 0, 1, 0); break;
      case  362: CALL(0, 1, 0, 1, 0, 1, 1, 0, 1, 0); break;
      case  363: CALL(1, 1, 0, 1, 0, 1, 1, 0, 1, 0); break;
      case  364: CALL(0, 0, 1, 1, 0, 1, 1, 0, 1, 0); break;
      case  365: CALL(1, 0, 1, 1, 0, 1, 1, 0, 1, 0); break;
      case  366: CALL(0, 1, 1, 1, 0, 1, 1, 0, 1, 0); break;
      case  367: CALL(1, 1, 1, 1, 0, 1, 1, 0, 1, 0); break;

      case  368: CALL(0, 0, 0, 0, 1, 1, 1, 0, 1, 0); break;
      case  369: CALL(1, 0, 0, 0, 1, 1, 1, 0, 1, 0); break;
      case  370: CALL(0, 1, 0, 0, 1, 1, 1, 0, 1, 0); break;
      case  371: CALL(1, 1, 0, 0, 1, 1, 1, 0, 1, 0); break;
      case  372: CALL(0, 0, 1, 0, 1, 1, 1, 0, 1, 0); break;
      case  373: CALL(1, 0, 1, 0, 1, 1, 1, 0, 1, 0); break;
      case  374: CALL(0, 1, 1, 0, 1, 1, 1, 0, 1, 0); break;
      case  375: CALL(1, 1, 1, 0, 1, 1, 1, 0, 1, 0); break;
      case  376: CALL(0, 0, 0, 1, 1, 1, 1, 0, 1, 0); break;
      case  377: CALL(1, 0, 0, 1, 1, 1, 1, 0, 1, 0); break;
      case  378: CALL(0, 1, 0, 1, 1, 1, 1, 0, 1, 0); break;
      case  379: CALL(1, 1, 0, 1, 1, 1, 1, 0, 1, 0); break;
      case  380: CALL(0, 0, 1, 1, 1, 1, 1, 0, 1, 0); break;
      case  381: CALL(1, 0, 1, 1, 1, 1, 1, 0, 1, 0); break;
      case  382: CALL(0, 1, 1, 1, 1, 1, 1, 0, 1, 0); break;
      case  383: CALL(1, 1, 1, 1, 1, 1, 1, 0, 1, 0); break;

#endif
      case  384: CALL(0, 0, 0, 0, 0, 0, 0, 1, 1, 0); break;
      case  385: CALL(1, 0, 0, 0, 0, 0, 0, 1, 1, 0); break;
      case  386: CALL(0, 1, 0, 0, 0, 0, 0, 1, 1, 0); break;
      case  387: CALL(1, 1, 0, 0, 0, 0, 0, 1, 1, 0); break;
      case  388: CALL(0, 0, 1, 0, 0, 0, 0, 1, 1, 0); break;
      case  389: CALL(1, 0, 1, 0, 0, 0, 0, 1, 1, 0); break;
      case  390: CALL(0, 1, 1, 0, 0, 0, 0, 1, 1, 0); break;
      case  391: CALL(1, 1, 1, 0, 0, 0, 0, 1, 1, 0); break;
      case  392: CALL(0, 0, 0, 1, 0, 0, 0, 1, 1, 0); break;
      case  393: CALL(1, 0, 0, 1, 0, 0, 0, 1, 1, 0); break;
      case  394: CALL(0, 1, 0, 1, 0, 0, 0, 1, 1, 0); break;
      case  395: CALL(1, 1, 0, 1, 0, 0, 0, 1, 1, 0); break;
      case  396: CALL(0, 0, 1, 1, 0, 0, 0, 1, 1, 0); break;
      case  397: CALL(1, 0, 1, 1, 0, 0, 0, 1, 1, 0); break;
      case  398: CALL(0, 1, 1, 1, 0, 0, 0, 1, 1, 0); break;
      case  399: CALL(1, 1, 1, 1, 0, 0, 0, 1, 1, 0); break;

#if 0
      case  400: CALL(0, 0, 0, 0, 1, 0, 0, 1, 1, 0); break;
      case  401: CALL(1, 0, 0, 0, 1, 0, 0, 1, 1, 0); break;
      case  402: CALL(0, 1, 0, 0, 1, 0, 0, 1, 1, 0); break;
      case  403: CALL(1, 1, 0, 0, 1, 0, 0, 1, 1, 0); break;
      case  404: CALL(0, 0, 1, 0, 1, 0, 0, 1, 1, 0); break;
      case  405: CALL(1, 0, 1, 0, 1, 0, 0, 1, 1, 0); break;
      case  406: CALL(0, 1, 1, 0, 1, 0, 0, 1, 1, 0); break;
      case  407: CALL(1, 1, 1, 0, 1, 0, 0, 1, 1, 0); break;
      case  408: CALL(0, 0, 0, 1, 1, 0, 0, 1, 1, 0); break;
      case  409: CALL(1, 0, 0, 1, 1, 0, 0, 1, 1, 0); break;
      case  410: CALL(0, 1, 0, 1, 1, 0, 0, 1, 1, 0); break;
      case  411: CALL(1, 1, 0, 1, 1, 0, 0, 1, 1, 0); break;
      case  412: CALL(0, 0, 1, 1, 1, 0, 0, 1, 1, 0); break;
      case  413: CALL(1, 0, 1, 1, 1, 0, 0, 1, 1, 0); break;
      case  414: CALL(0, 1, 1, 1, 1, 0, 0, 1, 1, 0); break;
      case  415: CALL(1, 1, 1, 1, 1, 0, 0, 1, 1, 0); break;

      case  416: CALL(0, 0, 0, 0, 0, 1, 0, 1, 1, 0); break;
      case  417: CALL(1, 0, 0, 0, 0, 1, 0, 1, 1, 0); break;
      case  418: CALL(0, 1, 0, 0, 0, 1, 0, 1, 1, 0); break;
      case  419: CALL(1, 1, 0, 0, 0, 1, 0, 1, 1, 0); break;
      case  420: CALL(0, 0, 1, 0, 0, 1, 0, 1, 1, 0); break;
      case  421: CALL(1, 0, 1, 0, 0, 1, 0, 1, 1, 0); break;
      case  422: CALL(0, 1, 1, 0, 0, 1, 0, 1, 1, 0); break;
      case  423: CALL(1, 1, 1, 0, 0, 1, 0, 1, 1, 0); break;
      case  424: CALL(0, 0, 0, 1, 0, 1, 0, 1, 1, 0); break;
      case  425: CALL(1, 0, 0, 1, 0, 1, 0, 1, 1, 0); break;
      case  426: CALL(0, 1, 0, 1, 0, 1, 0, 1, 1, 0); break;
      case  427: CALL(1, 1, 0, 1, 0, 1, 0, 1, 1, 0); break;
      case  428: CALL(0, 0, 1, 1, 0, 1, 0, 1, 1, 0); break;
      case  429: CALL(1, 0, 1, 1, 0, 1, 0, 1, 1, 0); break;
      case  430: CALL(0, 1, 1, 1, 0, 1, 0, 1, 1, 0); break;
      case  431: CALL(1, 1, 1, 1, 0, 1, 0, 1, 1, 0); break;

#endif
      case  432: CALL(0, 0, 0, 0, 1, 1, 0, 1, 1, 0); break;
      case  433: CALL(1, 0, 0, 0, 1, 1, 0, 1, 1, 0); break;
      case  434: CALL(0, 1, 0, 0, 1, 1, 0, 1, 1, 0); break;
      case  435: CALL(1, 1, 0, 0, 1, 1, 0, 1, 1, 0); break;
      case  436: CALL(0, 0, 1, 0, 1, 1, 0, 1, 1, 0); break;
      case  437: CALL(1, 0, 1, 0, 1, 1, 0, 1, 1, 0); break;
      case  438: CALL(0, 1, 1, 0, 1, 1, 0, 1, 1, 0); break;
      case  439: CALL(1, 1, 1, 0, 1, 1, 0, 1, 1, 0); break;
      case  440: CALL(0, 0, 0, 1, 1, 1, 0, 1, 1, 0); break;
      case  441: CALL(1, 0, 0, 1, 1, 1, 0, 1, 1, 0); break;
      case  442: CALL(0, 1, 0, 1, 1, 1, 0, 1, 1, 0); break;
      case  443: CALL(1, 1, 0, 1, 1, 1, 0, 1, 1, 0); break;
      case  444: CALL(0, 0, 1, 1, 1, 1, 0, 1, 1, 0); break;
      case  445: CALL(1, 0, 1, 1, 1, 1, 0, 1, 1, 0); break;
      case  446: CALL(0, 1, 1, 1, 1, 1, 0, 1, 1, 0); break;
      case  447: CALL(1, 1, 1, 1, 1, 1, 0, 1, 1, 0); break;

#if 0
      case  448: CALL(0, 0, 0, 0, 0, 0, 1, 1, 1, 0); break;
      case  449: CALL(1, 0, 0, 0, 0, 0, 1, 1, 1, 0); break;
      case  450: CALL(0, 1, 0, 0, 0, 0, 1, 1, 1, 0); break;
      case  451: CALL(1, 1, 0, 0, 0, 0, 1, 1, 1, 0); break;
      case  452: CALL(0, 0, 1, 0, 0, 0, 1, 1, 1, 0); break;
      case  453: CALL(1, 0, 1, 0, 0, 0, 1, 1, 1, 0); break;
      case  454: CALL(0, 1, 1, 0, 0, 0, 1, 1, 1, 0); break;
      case  455: CALL(1, 1, 1, 0, 0, 0, 1, 1, 1, 0); break;
      case  456: CALL(0, 0, 0, 1, 0, 0, 1, 1, 1, 0); break;
      case  457: CALL(1, 0, 0, 1, 0, 0, 1, 1, 1, 0); break;
      case  458: CALL(0, 1, 0, 1, 0, 0, 1, 1, 1, 0); break;
      case  459: CALL(1, 1, 0, 1, 0, 0, 1, 1, 1, 0); break;
      case  460: CALL(0, 0, 1, 1, 0, 0, 1, 1, 1, 0); break;
      case  461: CALL(1, 0, 1, 1, 0, 0, 1, 1, 1, 0); break;
      case  462: CALL(0, 1, 1, 1, 0, 0, 1, 1, 1, 0); break;
      case  463: CALL(1, 1, 1, 1, 0, 0, 1, 1, 1, 0); break;

#endif
      case  464: CALL(0, 0, 0, 0, 1, 0, 1, 1, 1, 0); break;
      case  465: CALL(1, 0, 0, 0, 1, 0, 1, 1, 1, 0); break;
      case  466: CALL(0, 1, 0, 0, 1, 0, 1, 1, 1, 0); break;
      case  467: CALL(1, 1, 0, 0, 1, 0, 1, 1, 1, 0); break;
      case  468: CALL(0, 0, 1, 0, 1, 0, 1, 1, 1, 0); break;
      case  469: CALL(1, 0, 1, 0, 1, 0, 1, 1, 1, 0); break;
      case  470: CALL(0, 1, 1, 0, 1, 0, 1, 1, 1, 0); break;
      case  471: CALL(1, 1, 1, 0, 1, 0, 1, 1, 1, 0); break;
      case  472: CALL(0, 0, 0, 1, 1, 0, 1, 1, 1, 0); break;
      case  473: CALL(1, 0, 0, 1, 1, 0, 1, 1, 1, 0); break;
      case  474: CALL(0, 1, 0, 1, 1, 0, 1, 1, 1, 0); break;
      case  475: CALL(1, 1, 0, 1, 1, 0, 1, 1, 1, 0); break;
      case  476: CALL(0, 0, 1, 1, 1, 0, 1, 1, 1, 0); break;
      case  477: CALL(1, 0, 1, 1, 1, 0, 1, 1, 1, 0); break;
      case  478: CALL(0, 1, 1, 1, 1, 0, 1, 1, 1, 0); break;
      case  479: CALL(1, 1, 1, 1, 1, 0, 1, 1, 1, 0); break;

#if 0
      case  480: CALL(0, 0, 0, 0, 0, 1, 1, 1, 1, 0); break;
      case  481: CALL(1, 0, 0, 0, 0, 1, 1, 1, 1, 0); break;
      case  482: CALL(0, 1, 0, 0, 0, 1, 1, 1, 1, 0); break;
      case  483: CALL(1, 1, 0, 0, 0, 1, 1, 1, 1, 0); break;
      case  484: CALL(0, 0, 1, 0, 0, 1, 1, 1, 1, 0); break;
      case  485: CALL(1, 0, 1, 0, 0, 1, 1, 1, 1, 0); break;
      case  486: CALL(0, 1, 1, 0, 0, 1, 1, 1, 1, 0); break;
      case  487: CALL(1, 1, 1, 0, 0, 1, 1, 1, 1, 0); break;
      case  488: CALL(0, 0, 0, 1, 0, 1, 1, 1, 1, 0); break;
      case  489: CALL(1, 0, 0, 1, 0, 1, 1, 1, 1, 0); break;
      case  490: CALL(0, 1, 0, 1, 0, 1, 1, 1, 1, 0); break;
      case  491: CALL(1, 1, 0, 1, 0, 1, 1, 1, 1, 0); break;
      case  492: CALL(0, 0, 1, 1, 0, 1, 1, 1, 1, 0); break;
      case  493: CALL(1, 0, 1, 1, 0, 1, 1, 1, 1, 0); break;
      case  494: CALL(0, 1, 1, 1, 0, 1, 1, 1, 1, 0); break;
      case  495: CALL(1, 1, 1, 1, 0, 1, 1, 1, 1, 0); break;

      case  496: CALL(0, 0, 0, 0, 1, 1, 1, 1, 1, 0); break;
      case  497: CALL(1, 0, 0, 0, 1, 1, 1, 1, 1, 0); break;
      case  498: CALL(0, 1, 0, 0, 1, 1, 1, 1, 1, 0); break;
      case  499: CALL(1, 1, 0, 0, 1, 1, 1, 1, 1, 0); break;
      case  500: CALL(0, 0, 1, 0, 1, 1, 1, 1, 1, 0); break;
      case  501: CALL(1, 0, 1, 0, 1, 1, 1, 1, 1, 0); break;
      case  502: CALL(0, 1, 1, 0, 1, 1, 1, 1, 1, 0); break;
      case  503: CALL(1, 1, 1, 0, 1, 1, 1, 1, 1, 0); break;
      case  504: CALL(0, 0, 0, 1, 1, 1, 1, 1, 1, 0); break;
      case  505: CALL(1, 0, 0, 1, 1, 1, 1, 1, 1, 0); break;
      case  506: CALL(0, 1, 0, 1, 1, 1, 1, 1, 1, 0); break;
      case  507: CALL(1, 1, 0, 1, 1, 1, 1, 1, 1, 0); break;
      case  508: CALL(0, 0, 1, 1, 1, 1, 1, 1, 1, 0); break;
      case  509: CALL(1, 0, 1, 1, 1, 1, 1, 1, 1, 0); break;
      case  510: CALL(0, 1, 1, 1, 1, 1, 1, 1, 1, 0); break;
      case  511: CALL(1, 1, 1, 1, 1, 1, 1, 1, 1, 0); break;
#endif
      /*
       * Haochuan: the calls starting from 512 to 1023 were generated by the following python script
       * #!/usr/bin/env python3
       * def gen_call(option: int):
       *     doEnergy = option & 1
       *     doVirial = (option >> 1) & 1
       *     doSlow = (option >> 2) & 1
       *     doPairlist = (option >> 3) & 1
       *     doAlch = (option >> 4) & 1
       *     doFEP = (option >> 5) & 1
       *     doTI = (option >> 6) & 1
       *     doStreaming = (option >> 7) & 1
       *     doTable = (option >> 8) & 1
       *     doAlchVdwForceSwitching = (option >> 9) & 1
       *     incompatible = False
       *     incompatible = incompatible | (doFEP and doTI)
       *     incompatible = incompatible | (doAlch and ((not doFEP) and (not doTI)))
       *     incompatible = incompatible | ((not doAlch) and (doFEP or doTI or doAlchVdwForceSwitching))
       *     incompatible = incompatible | ((not doTable) and (doAlch or doTI or doFEP or doAlchVdwForceSwitching))
       *     if incompatible:
       *         pass
       *         print(f'      // case {option}: CALL({doEnergy}, {doVirial}, {doSlow}, {doPairlist}, {doAlch}, {doFEP}, {doTI}, {doStreaming}, {doTable}, {doAlchVdwForceSwitching}); break;')
       *     else:
       *         print(f'      case {option}: CALL({doEnergy}, {doVirial}, {doSlow}, {doPairlist}, {doAlch}, {doFEP}, {doTI}, {doStreaming}, {doTable}, {doAlchVdwForceSwitching}); break;')
       *     return
       *
       * for i in range(512, 1024):
       *     gen_call(i)
       *
       */
      // case 512: CALL(0, 0, 0, 0, 0, 0, 0, 0, 0, 1); break;
      // case 513: CALL(1, 0, 0, 0, 0, 0, 0, 0, 0, 1); break;
      // case 514: CALL(0, 1, 0, 0, 0, 0, 0, 0, 0, 1); break;
      // case 515: CALL(1, 1, 0, 0, 0, 0, 0, 0, 0, 1); break;
      // case 516: CALL(0, 0, 1, 0, 0, 0, 0, 0, 0, 1); break;
      // case 517: CALL(1, 0, 1, 0, 0, 0, 0, 0, 0, 1); break;
      // case 518: CALL(0, 1, 1, 0, 0, 0, 0, 0, 0, 1); break;
      // case 519: CALL(1, 1, 1, 0, 0, 0, 0, 0, 0, 1); break;
      // case 520: CALL(0, 0, 0, 1, 0, 0, 0, 0, 0, 1); break;
      // case 521: CALL(1, 0, 0, 1, 0, 0, 0, 0, 0, 1); break;
      // case 522: CALL(0, 1, 0, 1, 0, 0, 0, 0, 0, 1); break;
      // case 523: CALL(1, 1, 0, 1, 0, 0, 0, 0, 0, 1); break;
      // case 524: CALL(0, 0, 1, 1, 0, 0, 0, 0, 0, 1); break;
      // case 525: CALL(1, 0, 1, 1, 0, 0, 0, 0, 0, 1); break;
      // case 526: CALL(0, 1, 1, 1, 0, 0, 0, 0, 0, 1); break;
      // case 527: CALL(1, 1, 1, 1, 0, 0, 0, 0, 0, 1); break;
      // case 528: CALL(0, 0, 0, 0, 1, 0, 0, 0, 0, 1); break;
      // case 529: CALL(1, 0, 0, 0, 1, 0, 0, 0, 0, 1); break;
      // case 530: CALL(0, 1, 0, 0, 1, 0, 0, 0, 0, 1); break;
      // case 531: CALL(1, 1, 0, 0, 1, 0, 0, 0, 0, 1); break;
      // case 532: CALL(0, 0, 1, 0, 1, 0, 0, 0, 0, 1); break;
      // case 533: CALL(1, 0, 1, 0, 1, 0, 0, 0, 0, 1); break;
      // case 534: CALL(0, 1, 1, 0, 1, 0, 0, 0, 0, 1); break;
      // case 535: CALL(1, 1, 1, 0, 1, 0, 0, 0, 0, 1); break;
      // case 536: CALL(0, 0, 0, 1, 1, 0, 0, 0, 0, 1); break;
      // case 537: CALL(1, 0, 0, 1, 1, 0, 0, 0, 0, 1); break;
      // case 538: CALL(0, 1, 0, 1, 1, 0, 0, 0, 0, 1); break;
      // case 539: CALL(1, 1, 0, 1, 1, 0, 0, 0, 0, 1); break;
      // case 540: CALL(0, 0, 1, 1, 1, 0, 0, 0, 0, 1); break;
      // case 541: CALL(1, 0, 1, 1, 1, 0, 0, 0, 0, 1); break;
      // case 542: CALL(0, 1, 1, 1, 1, 0, 0, 0, 0, 1); break;
      // case 543: CALL(1, 1, 1, 1, 1, 0, 0, 0, 0, 1); break;
      // case 544: CALL(0, 0, 0, 0, 0, 1, 0, 0, 0, 1); break;
      // case 545: CALL(1, 0, 0, 0, 0, 1, 0, 0, 0, 1); break;
      // case 546: CALL(0, 1, 0, 0, 0, 1, 0, 0, 0, 1); break;
      // case 547: CALL(1, 1, 0, 0, 0, 1, 0, 0, 0, 1); break;
      // case 548: CALL(0, 0, 1, 0, 0, 1, 0, 0, 0, 1); break;
      // case 549: CALL(1, 0, 1, 0, 0, 1, 0, 0, 0, 1); break;
      // case 550: CALL(0, 1, 1, 0, 0, 1, 0, 0, 0, 1); break;
      // case 551: CALL(1, 1, 1, 0, 0, 1, 0, 0, 0, 1); break;
      // case 552: CALL(0, 0, 0, 1, 0, 1, 0, 0, 0, 1); break;
      // case 553: CALL(1, 0, 0, 1, 0, 1, 0, 0, 0, 1); break;
      // case 554: CALL(0, 1, 0, 1, 0, 1, 0, 0, 0, 1); break;
      // case 555: CALL(1, 1, 0, 1, 0, 1, 0, 0, 0, 1); break;
      // case 556: CALL(0, 0, 1, 1, 0, 1, 0, 0, 0, 1); break;
      // case 557: CALL(1, 0, 1, 1, 0, 1, 0, 0, 0, 1); break;
      // case 558: CALL(0, 1, 1, 1, 0, 1, 0, 0, 0, 1); break;
      // case 559: CALL(1, 1, 1, 1, 0, 1, 0, 0, 0, 1); break;
      // case 560: CALL(0, 0, 0, 0, 1, 1, 0, 0, 0, 1); break;
      // case 561: CALL(1, 0, 0, 0, 1, 1, 0, 0, 0, 1); break;
      // case 562: CALL(0, 1, 0, 0, 1, 1, 0, 0, 0, 1); break;
      // case 563: CALL(1, 1, 0, 0, 1, 1, 0, 0, 0, 1); break;
      // case 564: CALL(0, 0, 1, 0, 1, 1, 0, 0, 0, 1); break;
      // case 565: CALL(1, 0, 1, 0, 1, 1, 0, 0, 0, 1); break;
      // case 566: CALL(0, 1, 1, 0, 1, 1, 0, 0, 0, 1); break;
      // case 567: CALL(1, 1, 1, 0, 1, 1, 0, 0, 0, 1); break;
      // case 568: CALL(0, 0, 0, 1, 1, 1, 0, 0, 0, 1); break;
      // case 569: CALL(1, 0, 0, 1, 1, 1, 0, 0, 0, 1); break;
      // case 570: CALL(0, 1, 0, 1, 1, 1, 0, 0, 0, 1); break;
      // case 571: CALL(1, 1, 0, 1, 1, 1, 0, 0, 0, 1); break;
      // case 572: CALL(0, 0, 1, 1, 1, 1, 0, 0, 0, 1); break;
      // case 573: CALL(1, 0, 1, 1, 1, 1, 0, 0, 0, 1); break;
      // case 574: CALL(0, 1, 1, 1, 1, 1, 0, 0, 0, 1); break;
      // case 575: CALL(1, 1, 1, 1, 1, 1, 0, 0, 0, 1); break;
      // case 576: CALL(0, 0, 0, 0, 0, 0, 1, 0, 0, 1); break;
      // case 577: CALL(1, 0, 0, 0, 0, 0, 1, 0, 0, 1); break;
      // case 578: CALL(0, 1, 0, 0, 0, 0, 1, 0, 0, 1); break;
      // case 579: CALL(1, 1, 0, 0, 0, 0, 1, 0, 0, 1); break;
      // case 580: CALL(0, 0, 1, 0, 0, 0, 1, 0, 0, 1); break;
      // case 581: CALL(1, 0, 1, 0, 0, 0, 1, 0, 0, 1); break;
      // case 582: CALL(0, 1, 1, 0, 0, 0, 1, 0, 0, 1); break;
      // case 583: CALL(1, 1, 1, 0, 0, 0, 1, 0, 0, 1); break;
      // case 584: CALL(0, 0, 0, 1, 0, 0, 1, 0, 0, 1); break;
      // case 585: CALL(1, 0, 0, 1, 0, 0, 1, 0, 0, 1); break;
      // case 586: CALL(0, 1, 0, 1, 0, 0, 1, 0, 0, 1); break;
      // case 587: CALL(1, 1, 0, 1, 0, 0, 1, 0, 0, 1); break;
      // case 588: CALL(0, 0, 1, 1, 0, 0, 1, 0, 0, 1); break;
      // case 589: CALL(1, 0, 1, 1, 0, 0, 1, 0, 0, 1); break;
      // case 590: CALL(0, 1, 1, 1, 0, 0, 1, 0, 0, 1); break;
      // case 591: CALL(1, 1, 1, 1, 0, 0, 1, 0, 0, 1); break;
      // case 592: CALL(0, 0, 0, 0, 1, 0, 1, 0, 0, 1); break;
      // case 593: CALL(1, 0, 0, 0, 1, 0, 1, 0, 0, 1); break;
      // case 594: CALL(0, 1, 0, 0, 1, 0, 1, 0, 0, 1); break;
      // case 595: CALL(1, 1, 0, 0, 1, 0, 1, 0, 0, 1); break;
      // case 596: CALL(0, 0, 1, 0, 1, 0, 1, 0, 0, 1); break;
      // case 597: CALL(1, 0, 1, 0, 1, 0, 1, 0, 0, 1); break;
      // case 598: CALL(0, 1, 1, 0, 1, 0, 1, 0, 0, 1); break;
      // case 599: CALL(1, 1, 1, 0, 1, 0, 1, 0, 0, 1); break;
      // case 600: CALL(0, 0, 0, 1, 1, 0, 1, 0, 0, 1); break;
      // case 601: CALL(1, 0, 0, 1, 1, 0, 1, 0, 0, 1); break;
      // case 602: CALL(0, 1, 0, 1, 1, 0, 1, 0, 0, 1); break;
      // case 603: CALL(1, 1, 0, 1, 1, 0, 1, 0, 0, 1); break;
      // case 604: CALL(0, 0, 1, 1, 1, 0, 1, 0, 0, 1); break;
      // case 605: CALL(1, 0, 1, 1, 1, 0, 1, 0, 0, 1); break;
      // case 606: CALL(0, 1, 1, 1, 1, 0, 1, 0, 0, 1); break;
      // case 607: CALL(1, 1, 1, 1, 1, 0, 1, 0, 0, 1); break;
      // case 608: CALL(0, 0, 0, 0, 0, 1, 1, 0, 0, 1); break;
      // case 609: CALL(1, 0, 0, 0, 0, 1, 1, 0, 0, 1); break;
      // case 610: CALL(0, 1, 0, 0, 0, 1, 1, 0, 0, 1); break;
      // case 611: CALL(1, 1, 0, 0, 0, 1, 1, 0, 0, 1); break;
      // case 612: CALL(0, 0, 1, 0, 0, 1, 1, 0, 0, 1); break;
      // case 613: CALL(1, 0, 1, 0, 0, 1, 1, 0, 0, 1); break;
      // case 614: CALL(0, 1, 1, 0, 0, 1, 1, 0, 0, 1); break;
      // case 615: CALL(1, 1, 1, 0, 0, 1, 1, 0, 0, 1); break;
      // case 616: CALL(0, 0, 0, 1, 0, 1, 1, 0, 0, 1); break;
      // case 617: CALL(1, 0, 0, 1, 0, 1, 1, 0, 0, 1); break;
      // case 618: CALL(0, 1, 0, 1, 0, 1, 1, 0, 0, 1); break;
      // case 619: CALL(1, 1, 0, 1, 0, 1, 1, 0, 0, 1); break;
      // case 620: CALL(0, 0, 1, 1, 0, 1, 1, 0, 0, 1); break;
      // case 621: CALL(1, 0, 1, 1, 0, 1, 1, 0, 0, 1); break;
      // case 622: CALL(0, 1, 1, 1, 0, 1, 1, 0, 0, 1); break;
      // case 623: CALL(1, 1, 1, 1, 0, 1, 1, 0, 0, 1); break;
      // case 624: CALL(0, 0, 0, 0, 1, 1, 1, 0, 0, 1); break;
      // case 625: CALL(1, 0, 0, 0, 1, 1, 1, 0, 0, 1); break;
      // case 626: CALL(0, 1, 0, 0, 1, 1, 1, 0, 0, 1); break;
      // case 627: CALL(1, 1, 0, 0, 1, 1, 1, 0, 0, 1); break;
      // case 628: CALL(0, 0, 1, 0, 1, 1, 1, 0, 0, 1); break;
      // case 629: CALL(1, 0, 1, 0, 1, 1, 1, 0, 0, 1); break;
      // case 630: CALL(0, 1, 1, 0, 1, 1, 1, 0, 0, 1); break;
      // case 631: CALL(1, 1, 1, 0, 1, 1, 1, 0, 0, 1); break;
      // case 632: CALL(0, 0, 0, 1, 1, 1, 1, 0, 0, 1); break;
      // case 633: CALL(1, 0, 0, 1, 1, 1, 1, 0, 0, 1); break;
      // case 634: CALL(0, 1, 0, 1, 1, 1, 1, 0, 0, 1); break;
      // case 635: CALL(1, 1, 0, 1, 1, 1, 1, 0, 0, 1); break;
      // case 636: CALL(0, 0, 1, 1, 1, 1, 1, 0, 0, 1); break;
      // case 637: CALL(1, 0, 1, 1, 1, 1, 1, 0, 0, 1); break;
      // case 638: CALL(0, 1, 1, 1, 1, 1, 1, 0, 0, 1); break;
      // case 639: CALL(1, 1, 1, 1, 1, 1, 1, 0, 0, 1); break;
      // case 640: CALL(0, 0, 0, 0, 0, 0, 0, 1, 0, 1); break;
      // case 641: CALL(1, 0, 0, 0, 0, 0, 0, 1, 0, 1); break;
      // case 642: CALL(0, 1, 0, 0, 0, 0, 0, 1, 0, 1); break;
      // case 643: CALL(1, 1, 0, 0, 0, 0, 0, 1, 0, 1); break;
      // case 644: CALL(0, 0, 1, 0, 0, 0, 0, 1, 0, 1); break;
      // case 645: CALL(1, 0, 1, 0, 0, 0, 0, 1, 0, 1); break;
      // case 646: CALL(0, 1, 1, 0, 0, 0, 0, 1, 0, 1); break;
      // case 647: CALL(1, 1, 1, 0, 0, 0, 0, 1, 0, 1); break;
      // case 648: CALL(0, 0, 0, 1, 0, 0, 0, 1, 0, 1); break;
      // case 649: CALL(1, 0, 0, 1, 0, 0, 0, 1, 0, 1); break;
      // case 650: CALL(0, 1, 0, 1, 0, 0, 0, 1, 0, 1); break;
      // case 651: CALL(1, 1, 0, 1, 0, 0, 0, 1, 0, 1); break;
      // case 652: CALL(0, 0, 1, 1, 0, 0, 0, 1, 0, 1); break;
      // case 653: CALL(1, 0, 1, 1, 0, 0, 0, 1, 0, 1); break;
      // case 654: CALL(0, 1, 1, 1, 0, 0, 0, 1, 0, 1); break;
      // case 655: CALL(1, 1, 1, 1, 0, 0, 0, 1, 0, 1); break;
      // case 656: CALL(0, 0, 0, 0, 1, 0, 0, 1, 0, 1); break;
      // case 657: CALL(1, 0, 0, 0, 1, 0, 0, 1, 0, 1); break;
      // case 658: CALL(0, 1, 0, 0, 1, 0, 0, 1, 0, 1); break;
      // case 659: CALL(1, 1, 0, 0, 1, 0, 0, 1, 0, 1); break;
      // case 660: CALL(0, 0, 1, 0, 1, 0, 0, 1, 0, 1); break;
      // case 661: CALL(1, 0, 1, 0, 1, 0, 0, 1, 0, 1); break;
      // case 662: CALL(0, 1, 1, 0, 1, 0, 0, 1, 0, 1); break;
      // case 663: CALL(1, 1, 1, 0, 1, 0, 0, 1, 0, 1); break;
      // case 664: CALL(0, 0, 0, 1, 1, 0, 0, 1, 0, 1); break;
      // case 665: CALL(1, 0, 0, 1, 1, 0, 0, 1, 0, 1); break;
      // case 666: CALL(0, 1, 0, 1, 1, 0, 0, 1, 0, 1); break;
      // case 667: CALL(1, 1, 0, 1, 1, 0, 0, 1, 0, 1); break;
      // case 668: CALL(0, 0, 1, 1, 1, 0, 0, 1, 0, 1); break;
      // case 669: CALL(1, 0, 1, 1, 1, 0, 0, 1, 0, 1); break;
      // case 670: CALL(0, 1, 1, 1, 1, 0, 0, 1, 0, 1); break;
      // case 671: CALL(1, 1, 1, 1, 1, 0, 0, 1, 0, 1); break;
      // case 672: CALL(0, 0, 0, 0, 0, 1, 0, 1, 0, 1); break;
      // case 673: CALL(1, 0, 0, 0, 0, 1, 0, 1, 0, 1); break;
      // case 674: CALL(0, 1, 0, 0, 0, 1, 0, 1, 0, 1); break;
      // case 675: CALL(1, 1, 0, 0, 0, 1, 0, 1, 0, 1); break;
      // case 676: CALL(0, 0, 1, 0, 0, 1, 0, 1, 0, 1); break;
      // case 677: CALL(1, 0, 1, 0, 0, 1, 0, 1, 0, 1); break;
      // case 678: CALL(0, 1, 1, 0, 0, 1, 0, 1, 0, 1); break;
      // case 679: CALL(1, 1, 1, 0, 0, 1, 0, 1, 0, 1); break;
      // case 680: CALL(0, 0, 0, 1, 0, 1, 0, 1, 0, 1); break;
      // case 681: CALL(1, 0, 0, 1, 0, 1, 0, 1, 0, 1); break;
      // case 682: CALL(0, 1, 0, 1, 0, 1, 0, 1, 0, 1); break;
      // case 683: CALL(1, 1, 0, 1, 0, 1, 0, 1, 0, 1); break;
      // case 684: CALL(0, 0, 1, 1, 0, 1, 0, 1, 0, 1); break;
      // case 685: CALL(1, 0, 1, 1, 0, 1, 0, 1, 0, 1); break;
      // case 686: CALL(0, 1, 1, 1, 0, 1, 0, 1, 0, 1); break;
      // case 687: CALL(1, 1, 1, 1, 0, 1, 0, 1, 0, 1); break;
      // case 688: CALL(0, 0, 0, 0, 1, 1, 0, 1, 0, 1); break;
      // case 689: CALL(1, 0, 0, 0, 1, 1, 0, 1, 0, 1); break;
      // case 690: CALL(0, 1, 0, 0, 1, 1, 0, 1, 0, 1); break;
      // case 691: CALL(1, 1, 0, 0, 1, 1, 0, 1, 0, 1); break;
      // case 692: CALL(0, 0, 1, 0, 1, 1, 0, 1, 0, 1); break;
      // case 693: CALL(1, 0, 1, 0, 1, 1, 0, 1, 0, 1); break;
      // case 694: CALL(0, 1, 1, 0, 1, 1, 0, 1, 0, 1); break;
      // case 695: CALL(1, 1, 1, 0, 1, 1, 0, 1, 0, 1); break;
      // case 696: CALL(0, 0, 0, 1, 1, 1, 0, 1, 0, 1); break;
      // case 697: CALL(1, 0, 0, 1, 1, 1, 0, 1, 0, 1); break;
      // case 698: CALL(0, 1, 0, 1, 1, 1, 0, 1, 0, 1); break;
      // case 699: CALL(1, 1, 0, 1, 1, 1, 0, 1, 0, 1); break;
      // case 700: CALL(0, 0, 1, 1, 1, 1, 0, 1, 0, 1); break;
      // case 701: CALL(1, 0, 1, 1, 1, 1, 0, 1, 0, 1); break;
      // case 702: CALL(0, 1, 1, 1, 1, 1, 0, 1, 0, 1); break;
      // case 703: CALL(1, 1, 1, 1, 1, 1, 0, 1, 0, 1); break;
      // case 704: CALL(0, 0, 0, 0, 0, 0, 1, 1, 0, 1); break;
      // case 705: CALL(1, 0, 0, 0, 0, 0, 1, 1, 0, 1); break;
      // case 706: CALL(0, 1, 0, 0, 0, 0, 1, 1, 0, 1); break;
      // case 707: CALL(1, 1, 0, 0, 0, 0, 1, 1, 0, 1); break;
      // case 708: CALL(0, 0, 1, 0, 0, 0, 1, 1, 0, 1); break;
      // case 709: CALL(1, 0, 1, 0, 0, 0, 1, 1, 0, 1); break;
      // case 710: CALL(0, 1, 1, 0, 0, 0, 1, 1, 0, 1); break;
      // case 711: CALL(1, 1, 1, 0, 0, 0, 1, 1, 0, 1); break;
      // case 712: CALL(0, 0, 0, 1, 0, 0, 1, 1, 0, 1); break;
      // case 713: CALL(1, 0, 0, 1, 0, 0, 1, 1, 0, 1); break;
      // case 714: CALL(0, 1, 0, 1, 0, 0, 1, 1, 0, 1); break;
      // case 715: CALL(1, 1, 0, 1, 0, 0, 1, 1, 0, 1); break;
      // case 716: CALL(0, 0, 1, 1, 0, 0, 1, 1, 0, 1); break;
      // case 717: CALL(1, 0, 1, 1, 0, 0, 1, 1, 0, 1); break;
      // case 718: CALL(0, 1, 1, 1, 0, 0, 1, 1, 0, 1); break;
      // case 719: CALL(1, 1, 1, 1, 0, 0, 1, 1, 0, 1); break;
      // case 720: CALL(0, 0, 0, 0, 1, 0, 1, 1, 0, 1); break;
      // case 721: CALL(1, 0, 0, 0, 1, 0, 1, 1, 0, 1); break;
      // case 722: CALL(0, 1, 0, 0, 1, 0, 1, 1, 0, 1); break;
      // case 723: CALL(1, 1, 0, 0, 1, 0, 1, 1, 0, 1); break;
      // case 724: CALL(0, 0, 1, 0, 1, 0, 1, 1, 0, 1); break;
      // case 725: CALL(1, 0, 1, 0, 1, 0, 1, 1, 0, 1); break;
      // case 726: CALL(0, 1, 1, 0, 1, 0, 1, 1, 0, 1); break;
      // case 727: CALL(1, 1, 1, 0, 1, 0, 1, 1, 0, 1); break;
      // case 728: CALL(0, 0, 0, 1, 1, 0, 1, 1, 0, 1); break;
      // case 729: CALL(1, 0, 0, 1, 1, 0, 1, 1, 0, 1); break;
      // case 730: CALL(0, 1, 0, 1, 1, 0, 1, 1, 0, 1); break;
      // case 731: CALL(1, 1, 0, 1, 1, 0, 1, 1, 0, 1); break;
      // case 732: CALL(0, 0, 1, 1, 1, 0, 1, 1, 0, 1); break;
      // case 733: CALL(1, 0, 1, 1, 1, 0, 1, 1, 0, 1); break;
      // case 734: CALL(0, 1, 1, 1, 1, 0, 1, 1, 0, 1); break;
      // case 735: CALL(1, 1, 1, 1, 1, 0, 1, 1, 0, 1); break;
      // case 736: CALL(0, 0, 0, 0, 0, 1, 1, 1, 0, 1); break;
      // case 737: CALL(1, 0, 0, 0, 0, 1, 1, 1, 0, 1); break;
      // case 738: CALL(0, 1, 0, 0, 0, 1, 1, 1, 0, 1); break;
      // case 739: CALL(1, 1, 0, 0, 0, 1, 1, 1, 0, 1); break;
      // case 740: CALL(0, 0, 1, 0, 0, 1, 1, 1, 0, 1); break;
      // case 741: CALL(1, 0, 1, 0, 0, 1, 1, 1, 0, 1); break;
      // case 742: CALL(0, 1, 1, 0, 0, 1, 1, 1, 0, 1); break;
      // case 743: CALL(1, 1, 1, 0, 0, 1, 1, 1, 0, 1); break;
      // case 744: CALL(0, 0, 0, 1, 0, 1, 1, 1, 0, 1); break;
      // case 745: CALL(1, 0, 0, 1, 0, 1, 1, 1, 0, 1); break;
      // case 746: CALL(0, 1, 0, 1, 0, 1, 1, 1, 0, 1); break;
      // case 747: CALL(1, 1, 0, 1, 0, 1, 1, 1, 0, 1); break;
      // case 748: CALL(0, 0, 1, 1, 0, 1, 1, 1, 0, 1); break;
      // case 749: CALL(1, 0, 1, 1, 0, 1, 1, 1, 0, 1); break;
      // case 750: CALL(0, 1, 1, 1, 0, 1, 1, 1, 0, 1); break;
      // case 751: CALL(1, 1, 1, 1, 0, 1, 1, 1, 0, 1); break;
      // case 752: CALL(0, 0, 0, 0, 1, 1, 1, 1, 0, 1); break;
      // case 753: CALL(1, 0, 0, 0, 1, 1, 1, 1, 0, 1); break;
      // case 754: CALL(0, 1, 0, 0, 1, 1, 1, 1, 0, 1); break;
      // case 755: CALL(1, 1, 0, 0, 1, 1, 1, 1, 0, 1); break;
      // case 756: CALL(0, 0, 1, 0, 1, 1, 1, 1, 0, 1); break;
      // case 757: CALL(1, 0, 1, 0, 1, 1, 1, 1, 0, 1); break;
      // case 758: CALL(0, 1, 1, 0, 1, 1, 1, 1, 0, 1); break;
      // case 759: CALL(1, 1, 1, 0, 1, 1, 1, 1, 0, 1); break;
      // case 760: CALL(0, 0, 0, 1, 1, 1, 1, 1, 0, 1); break;
      // case 761: CALL(1, 0, 0, 1, 1, 1, 1, 1, 0, 1); break;
      // case 762: CALL(0, 1, 0, 1, 1, 1, 1, 1, 0, 1); break;
      // case 763: CALL(1, 1, 0, 1, 1, 1, 1, 1, 0, 1); break;
      // case 764: CALL(0, 0, 1, 1, 1, 1, 1, 1, 0, 1); break;
      // case 765: CALL(1, 0, 1, 1, 1, 1, 1, 1, 0, 1); break;
      // case 766: CALL(0, 1, 1, 1, 1, 1, 1, 1, 0, 1); break;
      // case 767: CALL(1, 1, 1, 1, 1, 1, 1, 1, 0, 1); break;
      // case 768: CALL(0, 0, 0, 0, 0, 0, 0, 0, 1, 1); break;
      // case 769: CALL(1, 0, 0, 0, 0, 0, 0, 0, 1, 1); break;
      // case 770: CALL(0, 1, 0, 0, 0, 0, 0, 0, 1, 1); break;
      // case 771: CALL(1, 1, 0, 0, 0, 0, 0, 0, 1, 1); break;
      // case 772: CALL(0, 0, 1, 0, 0, 0, 0, 0, 1, 1); break;
      // case 773: CALL(1, 0, 1, 0, 0, 0, 0, 0, 1, 1); break;
      // case 774: CALL(0, 1, 1, 0, 0, 0, 0, 0, 1, 1); break;
      // case 775: CALL(1, 1, 1, 0, 0, 0, 0, 0, 1, 1); break;
      // case 776: CALL(0, 0, 0, 1, 0, 0, 0, 0, 1, 1); break;
      // case 777: CALL(1, 0, 0, 1, 0, 0, 0, 0, 1, 1); break;
      // case 778: CALL(0, 1, 0, 1, 0, 0, 0, 0, 1, 1); break;
      // case 779: CALL(1, 1, 0, 1, 0, 0, 0, 0, 1, 1); break;
      // case 780: CALL(0, 0, 1, 1, 0, 0, 0, 0, 1, 1); break;
      // case 781: CALL(1, 0, 1, 1, 0, 0, 0, 0, 1, 1); break;
      // case 782: CALL(0, 1, 1, 1, 0, 0, 0, 0, 1, 1); break;
      // case 783: CALL(1, 1, 1, 1, 0, 0, 0, 0, 1, 1); break;
      // case 784: CALL(0, 0, 0, 0, 1, 0, 0, 0, 1, 1); break;
      // case 785: CALL(1, 0, 0, 0, 1, 0, 0, 0, 1, 1); break;
      // case 786: CALL(0, 1, 0, 0, 1, 0, 0, 0, 1, 1); break;
      // case 787: CALL(1, 1, 0, 0, 1, 0, 0, 0, 1, 1); break;
      // case 788: CALL(0, 0, 1, 0, 1, 0, 0, 0, 1, 1); break;
      // case 789: CALL(1, 0, 1, 0, 1, 0, 0, 0, 1, 1); break;
      // case 790: CALL(0, 1, 1, 0, 1, 0, 0, 0, 1, 1); break;
      // case 791: CALL(1, 1, 1, 0, 1, 0, 0, 0, 1, 1); break;
      // case 792: CALL(0, 0, 0, 1, 1, 0, 0, 0, 1, 1); break;
      // case 793: CALL(1, 0, 0, 1, 1, 0, 0, 0, 1, 1); break;
      // case 794: CALL(0, 1, 0, 1, 1, 0, 0, 0, 1, 1); break;
      // case 795: CALL(1, 1, 0, 1, 1, 0, 0, 0, 1, 1); break;
      // case 796: CALL(0, 0, 1, 1, 1, 0, 0, 0, 1, 1); break;
      // case 797: CALL(1, 0, 1, 1, 1, 0, 0, 0, 1, 1); break;
      // case 798: CALL(0, 1, 1, 1, 1, 0, 0, 0, 1, 1); break;
      // case 799: CALL(1, 1, 1, 1, 1, 0, 0, 0, 1, 1); break;
      // case 800: CALL(0, 0, 0, 0, 0, 1, 0, 0, 1, 1); break;
      // case 801: CALL(1, 0, 0, 0, 0, 1, 0, 0, 1, 1); break;
      // case 802: CALL(0, 1, 0, 0, 0, 1, 0, 0, 1, 1); break;
      // case 803: CALL(1, 1, 0, 0, 0, 1, 0, 0, 1, 1); break;
      // case 804: CALL(0, 0, 1, 0, 0, 1, 0, 0, 1, 1); break;
      // case 805: CALL(1, 0, 1, 0, 0, 1, 0, 0, 1, 1); break;
      // case 806: CALL(0, 1, 1, 0, 0, 1, 0, 0, 1, 1); break;
      // case 807: CALL(1, 1, 1, 0, 0, 1, 0, 0, 1, 1); break;
      // case 808: CALL(0, 0, 0, 1, 0, 1, 0, 0, 1, 1); break;
      // case 809: CALL(1, 0, 0, 1, 0, 1, 0, 0, 1, 1); break;
      // case 810: CALL(0, 1, 0, 1, 0, 1, 0, 0, 1, 1); break;
      // case 811: CALL(1, 1, 0, 1, 0, 1, 0, 0, 1, 1); break;
      // case 812: CALL(0, 0, 1, 1, 0, 1, 0, 0, 1, 1); break;
      // case 813: CALL(1, 0, 1, 1, 0, 1, 0, 0, 1, 1); break;
      // case 814: CALL(0, 1, 1, 1, 0, 1, 0, 0, 1, 1); break;
      // case 815: CALL(1, 1, 1, 1, 0, 1, 0, 0, 1, 1); break;
      case 816: CALL(0, 0, 0, 0, 1, 1, 0, 0, 1, 1); break;
      case 817: CALL(1, 0, 0, 0, 1, 1, 0, 0, 1, 1); break;
      case 818: CALL(0, 1, 0, 0, 1, 1, 0, 0, 1, 1); break;
      case 819: CALL(1, 1, 0, 0, 1, 1, 0, 0, 1, 1); break;
      case 820: CALL(0, 0, 1, 0, 1, 1, 0, 0, 1, 1); break;
      case 821: CALL(1, 0, 1, 0, 1, 1, 0, 0, 1, 1); break;
      case 822: CALL(0, 1, 1, 0, 1, 1, 0, 0, 1, 1); break;
      case 823: CALL(1, 1, 1, 0, 1, 1, 0, 0, 1, 1); break;
      case 824: CALL(0, 0, 0, 1, 1, 1, 0, 0, 1, 1); break;
      case 825: CALL(1, 0, 0, 1, 1, 1, 0, 0, 1, 1); break;
      case 826: CALL(0, 1, 0, 1, 1, 1, 0, 0, 1, 1); break;
      case 827: CALL(1, 1, 0, 1, 1, 1, 0, 0, 1, 1); break;
      case 828: CALL(0, 0, 1, 1, 1, 1, 0, 0, 1, 1); break;
      case 829: CALL(1, 0, 1, 1, 1, 1, 0, 0, 1, 1); break;
      case 830: CALL(0, 1, 1, 1, 1, 1, 0, 0, 1, 1); break;
      case 831: CALL(1, 1, 1, 1, 1, 1, 0, 0, 1, 1); break;
      // case 832: CALL(0, 0, 0, 0, 0, 0, 1, 0, 1, 1); break;
      // case 833: CALL(1, 0, 0, 0, 0, 0, 1, 0, 1, 1); break;
      // case 834: CALL(0, 1, 0, 0, 0, 0, 1, 0, 1, 1); break;
      // case 835: CALL(1, 1, 0, 0, 0, 0, 1, 0, 1, 1); break;
      // case 836: CALL(0, 0, 1, 0, 0, 0, 1, 0, 1, 1); break;
      // case 837: CALL(1, 0, 1, 0, 0, 0, 1, 0, 1, 1); break;
      // case 838: CALL(0, 1, 1, 0, 0, 0, 1, 0, 1, 1); break;
      // case 839: CALL(1, 1, 1, 0, 0, 0, 1, 0, 1, 1); break;
      // case 840: CALL(0, 0, 0, 1, 0, 0, 1, 0, 1, 1); break;
      // case 841: CALL(1, 0, 0, 1, 0, 0, 1, 0, 1, 1); break;
      // case 842: CALL(0, 1, 0, 1, 0, 0, 1, 0, 1, 1); break;
      // case 843: CALL(1, 1, 0, 1, 0, 0, 1, 0, 1, 1); break;
      // case 844: CALL(0, 0, 1, 1, 0, 0, 1, 0, 1, 1); break;
      // case 845: CALL(1, 0, 1, 1, 0, 0, 1, 0, 1, 1); break;
      // case 846: CALL(0, 1, 1, 1, 0, 0, 1, 0, 1, 1); break;
      // case 847: CALL(1, 1, 1, 1, 0, 0, 1, 0, 1, 1); break;
      case 848: CALL(0, 0, 0, 0, 1, 0, 1, 0, 1, 1); break;
      case 849: CALL(1, 0, 0, 0, 1, 0, 1, 0, 1, 1); break;
      case 850: CALL(0, 1, 0, 0, 1, 0, 1, 0, 1, 1); break;
      case 851: CALL(1, 1, 0, 0, 1, 0, 1, 0, 1, 1); break;
      case 852: CALL(0, 0, 1, 0, 1, 0, 1, 0, 1, 1); break;
      case 853: CALL(1, 0, 1, 0, 1, 0, 1, 0, 1, 1); break;
      case 854: CALL(0, 1, 1, 0, 1, 0, 1, 0, 1, 1); break;
      case 855: CALL(1, 1, 1, 0, 1, 0, 1, 0, 1, 1); break;
      case 856: CALL(0, 0, 0, 1, 1, 0, 1, 0, 1, 1); break;
      case 857: CALL(1, 0, 0, 1, 1, 0, 1, 0, 1, 1); break;
      case 858: CALL(0, 1, 0, 1, 1, 0, 1, 0, 1, 1); break;
      case 859: CALL(1, 1, 0, 1, 1, 0, 1, 0, 1, 1); break;
      case 860: CALL(0, 0, 1, 1, 1, 0, 1, 0, 1, 1); break;
      case 861: CALL(1, 0, 1, 1, 1, 0, 1, 0, 1, 1); break;
      case 862: CALL(0, 1, 1, 1, 1, 0, 1, 0, 1, 1); break;
      case 863: CALL(1, 1, 1, 1, 1, 0, 1, 0, 1, 1); break;
      // case 864: CALL(0, 0, 0, 0, 0, 1, 1, 0, 1, 1); break;
      // case 865: CALL(1, 0, 0, 0, 0, 1, 1, 0, 1, 1); break;
      // case 866: CALL(0, 1, 0, 0, 0, 1, 1, 0, 1, 1); break;
      // case 867: CALL(1, 1, 0, 0, 0, 1, 1, 0, 1, 1); break;
      // case 868: CALL(0, 0, 1, 0, 0, 1, 1, 0, 1, 1); break;
      // case 869: CALL(1, 0, 1, 0, 0, 1, 1, 0, 1, 1); break;
      // case 870: CALL(0, 1, 1, 0, 0, 1, 1, 0, 1, 1); break;
      // case 871: CALL(1, 1, 1, 0, 0, 1, 1, 0, 1, 1); break;
      // case 872: CALL(0, 0, 0, 1, 0, 1, 1, 0, 1, 1); break;
      // case 873: CALL(1, 0, 0, 1, 0, 1, 1, 0, 1, 1); break;
      // case 874: CALL(0, 1, 0, 1, 0, 1, 1, 0, 1, 1); break;
      // case 875: CALL(1, 1, 0, 1, 0, 1, 1, 0, 1, 1); break;
      // case 876: CALL(0, 0, 1, 1, 0, 1, 1, 0, 1, 1); break;
      // case 877: CALL(1, 0, 1, 1, 0, 1, 1, 0, 1, 1); break;
      // case 878: CALL(0, 1, 1, 1, 0, 1, 1, 0, 1, 1); break;
      // case 879: CALL(1, 1, 1, 1, 0, 1, 1, 0, 1, 1); break;
      // case 880: CALL(0, 0, 0, 0, 1, 1, 1, 0, 1, 1); break;
      // case 881: CALL(1, 0, 0, 0, 1, 1, 1, 0, 1, 1); break;
      // case 882: CALL(0, 1, 0, 0, 1, 1, 1, 0, 1, 1); break;
      // case 883: CALL(1, 1, 0, 0, 1, 1, 1, 0, 1, 1); break;
      // case 884: CALL(0, 0, 1, 0, 1, 1, 1, 0, 1, 1); break;
      // case 885: CALL(1, 0, 1, 0, 1, 1, 1, 0, 1, 1); break;
      // case 886: CALL(0, 1, 1, 0, 1, 1, 1, 0, 1, 1); break;
      // case 887: CALL(1, 1, 1, 0, 1, 1, 1, 0, 1, 1); break;
      // case 888: CALL(0, 0, 0, 1, 1, 1, 1, 0, 1, 1); break;
      // case 889: CALL(1, 0, 0, 1, 1, 1, 1, 0, 1, 1); break;
      // case 890: CALL(0, 1, 0, 1, 1, 1, 1, 0, 1, 1); break;
      // case 891: CALL(1, 1, 0, 1, 1, 1, 1, 0, 1, 1); break;
      // case 892: CALL(0, 0, 1, 1, 1, 1, 1, 0, 1, 1); break;
      // case 893: CALL(1, 0, 1, 1, 1, 1, 1, 0, 1, 1); break;
      // case 894: CALL(0, 1, 1, 1, 1, 1, 1, 0, 1, 1); break;
      // case 895: CALL(1, 1, 1, 1, 1, 1, 1, 0, 1, 1); break;
      // case 896: CALL(0, 0, 0, 0, 0, 0, 0, 1, 1, 1); break;
      // case 897: CALL(1, 0, 0, 0, 0, 0, 0, 1, 1, 1); break;
      // case 898: CALL(0, 1, 0, 0, 0, 0, 0, 1, 1, 1); break;
      // case 899: CALL(1, 1, 0, 0, 0, 0, 0, 1, 1, 1); break;
      // case 900: CALL(0, 0, 1, 0, 0, 0, 0, 1, 1, 1); break;
      // case 901: CALL(1, 0, 1, 0, 0, 0, 0, 1, 1, 1); break;
      // case 902: CALL(0, 1, 1, 0, 0, 0, 0, 1, 1, 1); break;
      // case 903: CALL(1, 1, 1, 0, 0, 0, 0, 1, 1, 1); break;
      // case 904: CALL(0, 0, 0, 1, 0, 0, 0, 1, 1, 1); break;
      // case 905: CALL(1, 0, 0, 1, 0, 0, 0, 1, 1, 1); break;
      // case 906: CALL(0, 1, 0, 1, 0, 0, 0, 1, 1, 1); break;
      // case 907: CALL(1, 1, 0, 1, 0, 0, 0, 1, 1, 1); break;
      // case 908: CALL(0, 0, 1, 1, 0, 0, 0, 1, 1, 1); break;
      // case 909: CALL(1, 0, 1, 1, 0, 0, 0, 1, 1, 1); break;
      // case 910: CALL(0, 1, 1, 1, 0, 0, 0, 1, 1, 1); break;
      // case 911: CALL(1, 1, 1, 1, 0, 0, 0, 1, 1, 1); break;
      // case 912: CALL(0, 0, 0, 0, 1, 0, 0, 1, 1, 1); break;
      // case 913: CALL(1, 0, 0, 0, 1, 0, 0, 1, 1, 1); break;
      // case 914: CALL(0, 1, 0, 0, 1, 0, 0, 1, 1, 1); break;
      // case 915: CALL(1, 1, 0, 0, 1, 0, 0, 1, 1, 1); break;
      // case 916: CALL(0, 0, 1, 0, 1, 0, 0, 1, 1, 1); break;
      // case 917: CALL(1, 0, 1, 0, 1, 0, 0, 1, 1, 1); break;
      // case 918: CALL(0, 1, 1, 0, 1, 0, 0, 1, 1, 1); break;
      // case 919: CALL(1, 1, 1, 0, 1, 0, 0, 1, 1, 1); break;
      // case 920: CALL(0, 0, 0, 1, 1, 0, 0, 1, 1, 1); break;
      // case 921: CALL(1, 0, 0, 1, 1, 0, 0, 1, 1, 1); break;
      // case 922: CALL(0, 1, 0, 1, 1, 0, 0, 1, 1, 1); break;
      // case 923: CALL(1, 1, 0, 1, 1, 0, 0, 1, 1, 1); break;
      // case 924: CALL(0, 0, 1, 1, 1, 0, 0, 1, 1, 1); break;
      // case 925: CALL(1, 0, 1, 1, 1, 0, 0, 1, 1, 1); break;
      // case 926: CALL(0, 1, 1, 1, 1, 0, 0, 1, 1, 1); break;
      // case 927: CALL(1, 1, 1, 1, 1, 0, 0, 1, 1, 1); break;
      // case 928: CALL(0, 0, 0, 0, 0, 1, 0, 1, 1, 1); break;
      // case 929: CALL(1, 0, 0, 0, 0, 1, 0, 1, 1, 1); break;
      // case 930: CALL(0, 1, 0, 0, 0, 1, 0, 1, 1, 1); break;
      // case 931: CALL(1, 1, 0, 0, 0, 1, 0, 1, 1, 1); break;
      // case 932: CALL(0, 0, 1, 0, 0, 1, 0, 1, 1, 1); break;
      // case 933: CALL(1, 0, 1, 0, 0, 1, 0, 1, 1, 1); break;
      // case 934: CALL(0, 1, 1, 0, 0, 1, 0, 1, 1, 1); break;
      // case 935: CALL(1, 1, 1, 0, 0, 1, 0, 1, 1, 1); break;
      // case 936: CALL(0, 0, 0, 1, 0, 1, 0, 1, 1, 1); break;
      // case 937: CALL(1, 0, 0, 1, 0, 1, 0, 1, 1, 1); break;
      // case 938: CALL(0, 1, 0, 1, 0, 1, 0, 1, 1, 1); break;
      // case 939: CALL(1, 1, 0, 1, 0, 1, 0, 1, 1, 1); break;
      // case 940: CALL(0, 0, 1, 1, 0, 1, 0, 1, 1, 1); break;
      // case 941: CALL(1, 0, 1, 1, 0, 1, 0, 1, 1, 1); break;
      // case 942: CALL(0, 1, 1, 1, 0, 1, 0, 1, 1, 1); break;
      // case 943: CALL(1, 1, 1, 1, 0, 1, 0, 1, 1, 1); break;
      case 944: CALL(0, 0, 0, 0, 1, 1, 0, 1, 1, 1); break;
      case 945: CALL(1, 0, 0, 0, 1, 1, 0, 1, 1, 1); break;
      case 946: CALL(0, 1, 0, 0, 1, 1, 0, 1, 1, 1); break;
      case 947: CALL(1, 1, 0, 0, 1, 1, 0, 1, 1, 1); break;
      case 948: CALL(0, 0, 1, 0, 1, 1, 0, 1, 1, 1); break;
      case 949: CALL(1, 0, 1, 0, 1, 1, 0, 1, 1, 1); break;
      case 950: CALL(0, 1, 1, 0, 1, 1, 0, 1, 1, 1); break;
      case 951: CALL(1, 1, 1, 0, 1, 1, 0, 1, 1, 1); break;
      case 952: CALL(0, 0, 0, 1, 1, 1, 0, 1, 1, 1); break;
      case 953: CALL(1, 0, 0, 1, 1, 1, 0, 1, 1, 1); break;
      case 954: CALL(0, 1, 0, 1, 1, 1, 0, 1, 1, 1); break;
      case 955: CALL(1, 1, 0, 1, 1, 1, 0, 1, 1, 1); break;
      case 956: CALL(0, 0, 1, 1, 1, 1, 0, 1, 1, 1); break;
      case 957: CALL(1, 0, 1, 1, 1, 1, 0, 1, 1, 1); break;
      case 958: CALL(0, 1, 1, 1, 1, 1, 0, 1, 1, 1); break;
      case 959: CALL(1, 1, 1, 1, 1, 1, 0, 1, 1, 1); break;
      // case 960: CALL(0, 0, 0, 0, 0, 0, 1, 1, 1, 1); break;
      // case 961: CALL(1, 0, 0, 0, 0, 0, 1, 1, 1, 1); break;
      // case 962: CALL(0, 1, 0, 0, 0, 0, 1, 1, 1, 1); break;
      // case 963: CALL(1, 1, 0, 0, 0, 0, 1, 1, 1, 1); break;
      // case 964: CALL(0, 0, 1, 0, 0, 0, 1, 1, 1, 1); break;
      // case 965: CALL(1, 0, 1, 0, 0, 0, 1, 1, 1, 1); break;
      // case 966: CALL(0, 1, 1, 0, 0, 0, 1, 1, 1, 1); break;
      // case 967: CALL(1, 1, 1, 0, 0, 0, 1, 1, 1, 1); break;
      // case 968: CALL(0, 0, 0, 1, 0, 0, 1, 1, 1, 1); break;
      // case 969: CALL(1, 0, 0, 1, 0, 0, 1, 1, 1, 1); break;
      // case 970: CALL(0, 1, 0, 1, 0, 0, 1, 1, 1, 1); break;
      // case 971: CALL(1, 1, 0, 1, 0, 0, 1, 1, 1, 1); break;
      // case 972: CALL(0, 0, 1, 1, 0, 0, 1, 1, 1, 1); break;
      // case 973: CALL(1, 0, 1, 1, 0, 0, 1, 1, 1, 1); break;
      // case 974: CALL(0, 1, 1, 1, 0, 0, 1, 1, 1, 1); break;
      // case 975: CALL(1, 1, 1, 1, 0, 0, 1, 1, 1, 1); break;
      case 976: CALL(0, 0, 0, 0, 1, 0, 1, 1, 1, 1); break;
      case 977: CALL(1, 0, 0, 0, 1, 0, 1, 1, 1, 1); break;
      case 978: CALL(0, 1, 0, 0, 1, 0, 1, 1, 1, 1); break;
      case 979: CALL(1, 1, 0, 0, 1, 0, 1, 1, 1, 1); break;
      case 980: CALL(0, 0, 1, 0, 1, 0, 1, 1, 1, 1); break;
      case 981: CALL(1, 0, 1, 0, 1, 0, 1, 1, 1, 1); break;
      case 982: CALL(0, 1, 1, 0, 1, 0, 1, 1, 1, 1); break;
      case 983: CALL(1, 1, 1, 0, 1, 0, 1, 1, 1, 1); break;
      case 984: CALL(0, 0, 0, 1, 1, 0, 1, 1, 1, 1); break;
      case 985: CALL(1, 0, 0, 1, 1, 0, 1, 1, 1, 1); break;
      case 986: CALL(0, 1, 0, 1, 1, 0, 1, 1, 1, 1); break;
      case 987: CALL(1, 1, 0, 1, 1, 0, 1, 1, 1, 1); break;
      case 988: CALL(0, 0, 1, 1, 1, 0, 1, 1, 1, 1); break;
      case 989: CALL(1, 0, 1, 1, 1, 0, 1, 1, 1, 1); break;
      case 990: CALL(0, 1, 1, 1, 1, 0, 1, 1, 1, 1); break;
      case 991: CALL(1, 1, 1, 1, 1, 0, 1, 1, 1, 1); break;
      // case 992: CALL(0, 0, 0, 0, 0, 1, 1, 1, 1, 1); break;
      // case 993: CALL(1, 0, 0, 0, 0, 1, 1, 1, 1, 1); break;
      // case 994: CALL(0, 1, 0, 0, 0, 1, 1, 1, 1, 1); break;
      // case 995: CALL(1, 1, 0, 0, 0, 1, 1, 1, 1, 1); break;
      // case 996: CALL(0, 0, 1, 0, 0, 1, 1, 1, 1, 1); break;
      // case 997: CALL(1, 0, 1, 0, 0, 1, 1, 1, 1, 1); break;
      // case 998: CALL(0, 1, 1, 0, 0, 1, 1, 1, 1, 1); break;
      // case 999: CALL(1, 1, 1, 0, 0, 1, 1, 1, 1, 1); break;
      // case 1000: CALL(0, 0, 0, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1001: CALL(1, 0, 0, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1002: CALL(0, 1, 0, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1003: CALL(1, 1, 0, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1004: CALL(0, 0, 1, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1005: CALL(1, 0, 1, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1006: CALL(0, 1, 1, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1007: CALL(1, 1, 1, 1, 0, 1, 1, 1, 1, 1); break;
      // case 1008: CALL(0, 0, 0, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1009: CALL(1, 0, 0, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1010: CALL(0, 1, 0, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1011: CALL(1, 1, 0, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1012: CALL(0, 0, 1, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1013: CALL(1, 0, 1, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1014: CALL(0, 1, 1, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1015: CALL(1, 1, 1, 0, 1, 1, 1, 1, 1, 1); break;
      // case 1016: CALL(0, 0, 0, 1, 1, 1, 1, 1, 1, 1); break;
      // case 1017: CALL(1, 0, 0, 1, 1, 1, 1, 1, 1, 1); break;
      // case 1018: CALL(0, 1, 0, 1, 1, 1, 1, 1, 1, 1); break;
      // case 1019: CALL(1, 1, 0, 1, 1, 1, 1, 1, 1, 1); break;
      // case 1020: CALL(0, 0, 1, 1, 1, 1, 1, 1, 1, 1); break;
      // case 1021: CALL(1, 0, 1, 1, 1, 1, 1, 1, 1, 1); break;
      // case 1022: CALL(0, 1, 1, 1, 1, 1, 1, 1, 1, 1); break;
      // case 1023: CALL(1, 1, 1, 1, 1, 1, 1, 1, 1, 1); break;

      default: {
#ifdef DEBUG
        sprintf(errmsg,
            "doEnergy=%d  doVirial=%d  doSlow=%d  doPairlist=%d  "
            "doAlch=%d  doFEP=%d  doTI=%d  doStreaming=%d doTable=%d "
            "\noptions = %d\n",
            doEnergy, doVirial, doSlow, doPairlist, doAlch, doFEP, doTI,
            doStreaming, doTable, options);
        NAMD_die(errmsg);
#else
        std::string call_options;
        call_options += "doEnergy = " + std::to_string(int(doEnergy));
        call_options += ", doVirial = " + std::to_string(int(doVirial));
        call_options += ", doSlow = " + std::to_string(int(doSlow));
        call_options += ", doPairlist = " + std::to_string(int(doPairlist));
        call_options += ", doAlch = " + std::to_string(int(doAlch));
        call_options += ", doFEP = " + std::to_string(int(doFEP));
        call_options += ", doTI = " + std::to_string(int(doTI));
        call_options += ", doStreaming = " + std::to_string(int(doStreaming));
        call_options += ", doTable = " + std::to_string(int(doTable));
        call_options += ", doAlchVdwForceSwitching = " + std::to_string(int(doAlchVdwForceSwitching));
        const std::string error = "CudaComputeNonbondedKernel::nonbondedForce, none of the kernels called. Options are:\n" + call_options;
        NAMD_bug(error.c_str());
#endif
      }

    }

#endif

#undef CALL
    cudaCheck(hipGetLastError());

    start += nblock*nwarp;
  }
  if ( doVirial || ! doStreaming ){
    int block = 128;
    int grid = (atomStorageSize + block - 1)/block;
    if (doSlow)
      transposeForcesKernel<1><<<grid, block, 0, stream>>>(d_forces, d_forcesSlow,
                      force_x, force_y, force_z, force_w,
                      forceSlow_x, forceSlow_y, forceSlow_z, forceSlow_w,
                      atomStorageSize);
    else
      transposeForcesKernel<0><<<grid, block, 0, stream>>>(d_forces, d_forcesSlow,
                      force_x, force_y, force_z, force_w,
                      forceSlow_x, forceSlow_y, forceSlow_z, forceSlow_w,
                      atomStorageSize);
  }
#if 0
  cudaCheck(hipStreamSynchronize(stream));

  // XXX TODO: ERASE THIS AFTERWARDS
  // this is not numAtoms, this is something else
  // will print the force inside the compute and afterwards
  float4* h_f;
  allocate_host<float4>(&h_f, atomStorageSize);
  copy_DtoH_sync<float4>(d_forces, h_f, atomStorageSize);

  FILE* pos_nb_f = fopen("compute_nb_dforces.txt", "a+");
  fprintf(pos_nb_f, "forces after kernel\n");
  // I'm gonna copy back the forces and just print them
  
  for(int i = 0; i < atomStorageSize; i++){
  //for(int i = 83000; i < 85000; i++){
    fprintf(pos_nb_f, "%3.10lf %3.10lf %3.10lf\n", h_f[i].x, 
      h_f[i].y, h_f[i].z);
  }
  deallocate_host<float4>(&h_f);
  fclose(pos_nb_f);
#endif
}

//
// Perform virial and energy reductions for non-bonded force calculation
//
void CudaComputeNonbondedKernel::reduceVirialEnergy(CudaTileListKernel& tlKernel,
  const int atomStorageSize, const bool doEnergy, const bool doVirial, const bool doSlow, const bool doGBIS,
  float4* d_forces, float4* d_forcesSlow,
  VirialEnergy* d_virialEnergy, hipStream_t stream) {

  if (doEnergy || doVirial) {
    clear_device_array<VirialEnergy>(d_virialEnergy, 1, stream);
  }

  if (doVirial)
  {
    int nthread = REDUCENONBONDEDVIRIALKERNEL_NUM_WARP*WARPSIZE;
    int nblock = min(deviceCUDA->getMaxNumBlocks(), (atomStorageSize-1)/nthread+1);
    reduceNonbondedVirialKernel <<< nblock, nthread, 0, stream >>>
    (doSlow, atomStorageSize, tlKernel.get_xyzq(), d_forces, d_forcesSlow, d_virialEnergy);
    cudaCheck(hipGetLastError());
  }

  if (doVirial || doEnergy)
  {
    int nthread = REDUCEVIRIALENERGYKERNEL_NUM_WARP*WARPSIZE;
    int nblock = min(deviceCUDA->getMaxNumBlocks(), (tlKernel.getTileListVirialEnergyLength()-1)/nthread+1);
    reduceVirialEnergyKernel <<< nblock, nthread, 0, stream >>>
    (doEnergy, doVirial, doSlow, tlKernel.getTileListVirialEnergyLength(), tlKernel.getTileListVirialEnergy(), d_virialEnergy);
    cudaCheck(hipGetLastError());
  }  

  if (doGBIS && doEnergy)
  {
    int nthread = REDUCEGBISENERGYKERNEL_NUM_WARP*WARPSIZE;
    int nblock = min(deviceCUDA->getMaxNumBlocks(), (tlKernel.getTileListVirialEnergyGBISLength()-1)/nthread+1);
    reduceGBISEnergyKernel <<< nblock, nthread, 0, stream >>>
    (tlKernel.getTileListVirialEnergyGBISLength(), tlKernel.getTileListVirialEnergy(), d_virialEnergy);
    cudaCheck(hipGetLastError());
  }

}

void CudaComputeNonbondedKernel::bindExclusions(int numExclusions, unsigned int* exclusion_bits) {
  int nconst = ( numExclusions < MAX_CONST_EXCLUSIONS ? numExclusions : MAX_CONST_EXCLUSIONS );
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(constExclusions), exclusion_bits, nconst*sizeof(unsigned int), 0));

  reallocate_device<unsigned int>(&overflowExclusions, &overflowExclusionsSize, numExclusions);
  copy_HtoD_sync<unsigned int>(exclusion_bits, overflowExclusions, numExclusions);
}


void CudaComputeNonbondedKernel::setExclusionsByAtom(int2* h_data, const int num_atoms) {
  // Global data structure shouldn't be reallocated
  if (d_exclusionsByAtom == NULL) allocate_device<int2>(&d_exclusionsByAtom, num_atoms);
  copy_HtoD_sync<int2>(h_data, d_exclusionsByAtom, num_atoms);

}


template<bool kDoAlch>
__global__ void updateVdwTypesExclKernel(
  const int numPatches,
  const CudaLocalRecord* localRecords,
  const int* global_vdwTypes,
  const int* global_id,
  const int* patchSortOrder,
  const int2* exclusionsByAtom,
  const int* global_partition,
  int* vdwTypes,
  int* atomIndex,
  int2* exclusions,
  char* part
) {
  __shared__ CudaLocalRecord s_record;
  using AccessType = int32_t;
  AccessType* s_record_buffer = (AccessType*)  &s_record;

  for (int patchIndex = blockIdx.x; patchIndex < numPatches; patchIndex += gridDim.x) {
    // Read in the CudaLocalRecord using multiple threads. This should 
    #pragma unroll 1
    for (int i = threadIdx.x; i < sizeof(CudaLocalRecord)/sizeof(AccessType); i += blockDim.x) {
      s_record_buffer[i] = ((AccessType*) &(localRecords[patchIndex]))[i];
    }
    __syncthreads();

    const int numAtoms = s_record.numAtoms;
    const int offset = s_record.bufferOffset;
    const int offsetNB = s_record.bufferOffsetNBPad;

    for (int i = threadIdx.x; i < numAtoms; i += blockDim.x) {
      const int order = patchSortOrder[offset + i];
      const int id = global_id[offset + order];
      vdwTypes  [offsetNB + i]   = global_vdwTypes[offset + order];
      atomIndex [offsetNB + i]   = id;
      exclusions[offsetNB + i].x = exclusionsByAtom[id].y;
      exclusions[offsetNB + i].y = exclusionsByAtom[id].x;
      if (kDoAlch) {
        part    [offsetNB + i]   = global_partition[offset + order];
      }
    }
    __syncthreads();
  }
}


void CudaComputeNonbondedKernel::updateVdwTypesExclOnGPU(CudaTileListKernel& tlKernel,
  const int numPatches, const int atomStorageSize, const bool alchOn,
  CudaLocalRecord* localRecords,
  const int* d_vdwTypes, const int* d_id, const int* d_sortOrder, 
  const int* d_partition,
  hipStream_t stream
) {
  reallocate_device<int>(&vdwTypes, &vdwTypesSize, atomStorageSize, OVERALLOC);
  reallocate_device<int2>(&exclIndexMaxDiff, &exclIndexMaxDiffSize, atomStorageSize, OVERALLOC);
  reallocate_device<int>(&atomIndex, &atomIndexSize, atomStorageSize, OVERALLOC);
 
  const int numBlocks = numPatches;
  const int numThreads = 512;
  
  if (alchOn) {
    updateVdwTypesExclKernel<true><<<numBlocks, numThreads, 0, stream>>>(
      numPatches, localRecords,
      d_vdwTypes, d_id, d_sortOrder, d_exclusionsByAtom, d_partition,
      vdwTypes, atomIndex, exclIndexMaxDiff, tlKernel.get_part()
    );
  } else {
    updateVdwTypesExclKernel<false><<<numBlocks, numThreads, 0, stream>>>(
      numPatches, localRecords,
      d_vdwTypes, d_id, d_sortOrder, d_exclusionsByAtom, d_partition,
      vdwTypes, atomIndex, exclIndexMaxDiff, tlKernel.get_part()
    );
  }

}

#endif // NAMD_CUDA
